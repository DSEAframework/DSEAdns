#include "hip/hip_runtime.h"
// Data Streaming for Explicit Algorithms - DSEA

#include <dsea.h>
#include <stdio.h>		// printf
#include <hipcub/hipcub.hpp>
#include <climits>       // for INT_MAX
#include <fstream>
#include <boost/filesystem.hpp>

using namespace :: std;

// print variadic template values
// overload
template<typename T>
void myprint(T head)
{
    std::cout << head << std::endl;
}
// base case: used when pack is non-empty
template<typename T, typename... Ts>
void myprint(T head, Ts... tail)
{
    std::cout << head << std::endl;
    myprint(tail...);
}

// Calculate the Z Y Coordinates in the grid from an thread ID
// ID: Array Index/Thread ID
// Y,Z: Grid coordinates in part, Y row, Z column
// NC: Number of Columns (NZ)

#define COORDS(ID, Y, Z, NC) \
	do { \
    Y = (ID) / (NC); \
    Z = (ID) % (NC); \
  } while(0)

// Calculate the array index from the grid coordinates
// ID: Array Index/Thread ID
// Y,Z: Grid coordinates in part, Z column, Y row
// NC: Number of Columns (NZ)
#define IDX(Y, Z, ID, NC) \ 
	do { \
    ID = (Y) * (NC) + (Z); \
  } while(0)



__device__ __forceinline__ double dns_pDer1(double v_ll, double v_l, double v_r, double v_rr, double DK) {
	return 1./DK * (1./12. * v_ll - 2./3. * v_l + 2./3. * v_r - 1./12. * v_rr);
}

__device__ __forceinline__ double dns_pDer2(double v_ll, double v_l, double v_c, double v_r, double v_rr, double DK) {
	return 1./(DK*DK) * (-1./12. * v_ll + 4./3. * v_l - 5./2. * v_c + 4./3. * v_r - 1./12. * v_rr);
}

__device__ __forceinline__ double calp(double irhoE, double irho, double irhou0, double irhou1, double irhou2) {
	//return ((GAMA - 1) * (rhoE - 0.5 * rhou0 * rhou0 / rho - 0.5 * rhou1 * rhou1 / rho - 0.5 rhou2 * rhou2 / rho));
	double tmp = (irhoE - 0.5 * irhou0 * irhou0 / irho - 0.5 * irhou1 * irhou1 / irho - 0.5 * irhou2 * irhou2 / irho);
	return (GAMA - 1) * tmp;
}

__device__ __forceinline__ double calT(double ip, double irho) {
	return MINF * MINF * GAMA * ip / irho;
}

__device__ __forceinline__ double calp_ui(double irhoE, double irho, double u0, double u1, double u2) {
	double tmp = (irhoE - 0.5 * irho * u0 * u0 - 0.5 * irho * u1 * u1 - 0.5 * irho * u2 * u2);
	return (GAMA - 1) * tmp;
}

__device__ __forceinline__ double calT_ui(double ip, double iinvrho) {
	return MINF * MINF * GAMA * ip * iinvrho;
}



// Calculates the global array index from 
__device__ int32_t thread_to_global_idx(int32_t problemsize, int32_t thread_idx, 
						int32_t block_size_z, int32_t block_size_y, 
						int32_t warp_size_z, int32_t warp_size_y,
						int32_t* c_i_block_out, int32_t* r_i_block_out) {
	int32_t global_idx;


	int32_t block_size = block_size_z * block_size_y;
	int32_t num_blocks = (problemsize*problemsize) / block_size;
	int32_t blocks_per_row = (problemsize) / block_size_z;

	int32_t warp_size = warp_size_z * warp_size_y;
	int32_t num_warps_per_block = block_size / warp_size;
	int32_t warps_per_row = block_size_z / warp_size_z;

	int32_t block_idx = thread_idx / block_size;
	int32_t block_row = block_idx / blocks_per_row;
	int32_t block_col = block_idx % blocks_per_row;

	int32_t idx_in_block = thread_idx % block_size;

	int32_t warp_idx = idx_in_block / warp_size;
	int32_t warp_row = warp_idx / warps_per_row;
	int32_t warp_col = warp_idx % warps_per_row;

	int32_t idx_in_warp = idx_in_block % warp_size;

	int32_t col_in_warp = idx_in_warp % warp_size_z;
	int32_t row_in_warp = idx_in_warp / warp_size_z;

	int32_t col_in_block = warp_col * warp_size_z + col_in_warp;
	int32_t row_in_block = warp_row * warp_size_y + row_in_warp;

	int32_t global_row = block_row * block_size_y + row_in_block;
	int32_t global_col = block_col * block_size_z + col_in_block;

	global_idx = global_row * problemsize + global_col;

	*c_i_block_out = col_in_block;
	*r_i_block_out = row_in_block;

	return global_idx;
	

}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Fused Kernels
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void dns_du0dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
	int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou0_c,
	/*order 1*/ double * __restrict__ irhou0_l, double * __restrict__ irhou0_r,
	/*order 2*/ double * __restrict__ irhou0_ll, double * __restrict__ irhou0_rr,
	int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
	/*order 0*/ double * __restrict__ odu0dx,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		double tmp0 = 0;


		double rhou0_dy_ll, rhou0_dy_l, rhou0_dy_c, rhou0_dy_r, rhou0_dy_rr;
		rhou0_dy_ll = irhou0_c[dy_ll];
		rhou0_dy_l = irhou0_c[dy_l];
		rhou0_dy_c = irhou0_c[gidx];
		rhou0_dy_r = irhou0_c[dy_r];
		rhou0_dy_rr = irhou0_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u0_dy_ll, u0_dy_l, u0_dy_c, u0_dy_r, u0_dy_rr;
		u0_dy_ll = rhou0_dy_ll / rho_dy_ll;
		u0_dy_l = rhou0_dy_l / rho_dy_l;
		u0_dy_c = rhou0_dy_c / rho_dy_c;
		u0_dy_r = rhou0_dy_r / rho_dy_r;
		u0_dy_rr = rhou0_dy_rr / rho_dy_rr;

		tmp0 += dns_pDer2(u0_dy_ll, u0_dy_l, u0_dy_c, u0_dy_r, u0_dy_rr, DY);


		double rhou0_dz_ll, rhou0_dz_l, rhou0_dz_c, rhou0_dz_r, rhou0_dz_rr;
		rhou0_dz_ll = irhou0_c[dz_ll];
		rhou0_dz_l = irhou0_c[dz_l];
		rhou0_dz_c = irhou0_c[gidx];
		rhou0_dz_r = irhou0_c[dz_r];
		rhou0_dz_rr = irhou0_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u0_dz_ll, u0_dz_l, u0_dz_c, u0_dz_r, u0_dz_rr;
		u0_dz_ll = rhou0_dz_ll / rho_dz_ll;
		u0_dz_l = rhou0_dz_l / rho_dz_l;
		u0_dz_c = rhou0_dz_c / rho_dz_c;
		u0_dz_r = rhou0_dz_r / rho_dz_r;
		u0_dz_rr = rhou0_dz_rr / rho_dz_rr;

		tmp0 += dns_pDer2(u0_dz_ll, u0_dz_l, u0_dz_c, u0_dz_r, u0_dz_rr, DZ);


		double rhou0_dx_ll, rhou0_dx_l, rhou0_dx_c, rhou0_dx_r, rhou0_dx_rr;
		rhou0_dx_ll = irhou0_ll[gidx];
		rhou0_dx_l = irhou0_l[gidx];
		rhou0_dx_c = irhou0_c[gidx];
		rhou0_dx_r = irhou0_r[gidx];
		rhou0_dx_rr = irhou0_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u0_dx_ll, u0_dx_l, u0_dx_c, u0_dx_r, u0_dx_rr;
		u0_dx_ll = rhou0_dx_ll / rho_dx_ll;
		u0_dx_l = rhou0_dx_l / rho_dx_l;
		u0_dx_c = rhou0_dx_c / rho_dx_c;
		u0_dx_r = rhou0_dx_r / rho_dx_r;
		u0_dx_rr = rhou0_dx_rr / rho_dx_rr;

		odu0dx[gidx] = dns_pDer1(sy_bc_ll *  u0_dx_ll, sy_bc_l * u0_dx_l, sy_bc_r * u0_dx_r, sy_bc_rr * u0_dx_rr, DX);
		tmp0 += 4./3. * dns_pDer2(sy_bc_ll * u0_dx_ll, sy_bc_l * u0_dx_l, u0_dx_c, sy_bc_r * u0_dx_r, sy_bc_rr * u0_dx_rr, DX);

		tmp_du0d2xi[gidx] = tmp0;

		double u0_dy_ll_dx_ll = irhou0_ll[dy_ll] / irho_ll[dy_ll];
		double u0_dy_l_dx_ll = irhou0_ll[dy_l] / irho_ll[dy_l];
		double u0_dy_r_dx_ll = irhou0_ll[dy_r] / irho_ll[dy_r];
		double u0_dy_rr_dx_ll = irhou0_ll[dy_rr] / irho_ll[dy_rr];

		double u0_dz_ll_dx_ll = irhou0_ll[dz_ll] / irho_ll[dz_ll];
		double u0_dz_l_dx_ll = irhou0_ll[dz_l] / irho_ll[dz_l];
		double u0_dz_r_dx_ll = irhou0_ll[dz_r] / irho_ll[dz_r];
		double u0_dz_rr_dx_ll = irhou0_ll[dz_rr] / irho_ll[dz_rr];

		double du0dy_dx_ll = dns_pDer1(u0_dy_ll_dx_ll, u0_dy_l_dx_ll, u0_dy_r_dx_ll, u0_dy_rr_dx_ll, DY);
		double du0dz_dx_ll = dns_pDer1(u0_dz_ll_dx_ll, u0_dz_l_dx_ll, u0_dz_r_dx_ll, u0_dz_rr_dx_ll, DZ);

		double u0_dy_ll_dx_l = irhou0_l[dy_ll] / irho_l[dy_ll];
		double u0_dy_l_dx_l = irhou0_l[dy_l] / irho_l[dy_l];
		double u0_dy_r_dx_l = irhou0_l[dy_r] / irho_l[dy_r];
		double u0_dy_rr_dx_l = irhou0_l[dy_rr] / irho_l[dy_rr];

		double u0_dz_ll_dx_l = irhou0_l[dz_ll] / irho_l[dz_ll];
		double u0_dz_l_dx_l = irhou0_l[dz_l] / irho_l[dz_l];
		double u0_dz_r_dx_l = irhou0_l[dz_r] / irho_l[dz_r];
		double u0_dz_rr_dx_l = irhou0_l[dz_rr] / irho_l[dz_rr];

		double du0dy_dx_l = dns_pDer1(u0_dy_ll_dx_l, u0_dy_l_dx_l, u0_dy_r_dx_l, u0_dy_rr_dx_l, DY);
		double du0dz_dx_l = dns_pDer1(u0_dz_ll_dx_l, u0_dz_l_dx_l, u0_dz_r_dx_l, u0_dz_rr_dx_l, DZ);

		double u0_dy_ll_dx_r = irhou0_r[dy_ll] / irho_r[dy_ll];
		double u0_dy_l_dx_r = irhou0_r[dy_l] / irho_r[dy_l];
		double u0_dy_r_dx_r = irhou0_r[dy_r] / irho_r[dy_r];
		double u0_dy_rr_dx_r = irhou0_r[dy_rr] / irho_r[dy_rr];

		double u0_dz_ll_dx_r = irhou0_r[dz_ll] / irho_r[dz_ll];
		double u0_dz_l_dx_r = irhou0_r[dz_l] / irho_r[dz_l];
		double u0_dz_r_dx_r = irhou0_r[dz_r] / irho_r[dz_r];
		double u0_dz_rr_dx_r = irhou0_r[dz_rr] / irho_r[dz_rr];

		double du0dy_dx_r = dns_pDer1(u0_dy_ll_dx_r, u0_dy_l_dx_r, u0_dy_r_dx_r, u0_dy_rr_dx_r, DY);
		double du0dz_dx_r = dns_pDer1(u0_dz_ll_dx_r, u0_dz_l_dx_r, u0_dz_r_dx_r, u0_dz_rr_dx_r, DZ);

		double u0_dy_ll_dx_rr = irhou0_rr[dy_ll] / irho_rr[dy_ll];
		double u0_dy_l_dx_rr = irhou0_rr[dy_l] / irho_rr[dy_l];
		double u0_dy_r_dx_rr = irhou0_rr[dy_r] / irho_rr[dy_r];
		double u0_dy_rr_dx_rr = irhou0_rr[dy_rr] / irho_rr[dy_rr];

		double u0_dz_ll_dx_rr = irhou0_rr[dz_ll] / irho_rr[dz_ll];
		double u0_dz_l_dx_rr = irhou0_rr[dz_l] / irho_rr[dz_l];
		double u0_dz_r_dx_rr = irhou0_rr[dz_r] / irho_rr[dz_r];
		double u0_dz_rr_dx_rr = irhou0_rr[dz_rr] / irho_rr[dz_rr];

		double du0dy_dx_rr = dns_pDer1(u0_dy_ll_dx_rr, u0_dy_l_dx_rr, u0_dy_r_dx_rr, u0_dy_rr_dx_rr, DY);
		double du0dz_dx_rr = dns_pDer1(u0_dz_ll_dx_rr, u0_dz_l_dx_rr, u0_dz_r_dx_rr, u0_dz_rr_dx_rr, DZ);

		tmp_du1d2xi[gidx] = 1./3. * dns_pDer1(sy_bc_ll *  du0dy_dx_ll, sy_bc_l * du0dy_dx_l, sy_bc_r * du0dy_dx_r, sy_bc_rr * du0dy_dx_rr, DX);
		tmp_du2d2xi[gidx] = 1./3. * dns_pDer1(sy_bc_ll *  du0dz_dx_ll, sy_bc_l * du0dz_dx_l, sy_bc_r * du0dz_dx_r, sy_bc_rr * du0dz_dx_rr, DX);

	}
}

__global__ void dns_du1dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
	int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou1_c,
	/*order 1*/ double * __restrict__ irhou1_l, double * __restrict__ irhou1_r,
	/*order 2*/ double * __restrict__ irhou1_ll, double * __restrict__ irhou1_rr,
	/*order 0*/ double * __restrict__ odu1dx,
  /*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		double tmp0 = 0;


		double rhou1_dy_ll, rhou1_dy_l, rhou1_dy_c, rhou1_dy_r, rhou1_dy_rr;
		rhou1_dy_ll = irhou1_c[dy_ll];
		rhou1_dy_l = irhou1_c[dy_l];
		rhou1_dy_c = irhou1_c[gidx];
		rhou1_dy_r = irhou1_c[dy_r];
		rhou1_dy_rr = irhou1_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u1_dy_ll, u1_dy_l, u1_dy_c, u1_dy_r, u1_dy_rr;
		u1_dy_ll = rhou1_dy_ll / rho_dy_ll;
		u1_dy_l = rhou1_dy_l / rho_dy_l;
		u1_dy_c = rhou1_dy_c / rho_dy_c;
		u1_dy_r = rhou1_dy_r / rho_dy_r;
		u1_dy_rr = rhou1_dy_rr / rho_dy_rr;

		tmp0 += 4./3. * dns_pDer2(u1_dy_ll, u1_dy_l, u1_dy_c, u1_dy_r, u1_dy_rr, DY);

		double rhou1_dz_ll, rhou1_dz_l, rhou1_dz_c, rhou1_dz_r, rhou1_dz_rr;
		rhou1_dz_ll = irhou1_c[dz_ll];
		rhou1_dz_l = irhou1_c[dz_l];
		rhou1_dz_c = irhou1_c[gidx];
		rhou1_dz_r = irhou1_c[dz_r];
		rhou1_dz_rr = irhou1_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u1_dz_ll, u1_dz_l, u1_dz_c, u1_dz_r, u1_dz_rr;
		u1_dz_ll = rhou1_dz_ll / rho_dz_ll;
		u1_dz_l = rhou1_dz_l / rho_dz_l;
		u1_dz_c = rhou1_dz_c / rho_dz_c;
		u1_dz_r = rhou1_dz_r / rho_dz_r;
		u1_dz_rr = rhou1_dz_rr / rho_dz_rr;

		tmp0 += dns_pDer2(u1_dz_ll, u1_dz_l, u1_dz_c, u1_dz_r, u1_dz_rr, DZ);


		double rhou1_dx_ll, rhou1_dx_l, rhou1_dx_c, rhou1_dx_r, rhou1_dx_rr;
		rhou1_dx_ll = irhou1_ll[gidx];
		rhou1_dx_l = irhou1_l[gidx];
		rhou1_dx_c = irhou1_c[gidx];
		rhou1_dx_r = irhou1_r[gidx];
		rhou1_dx_rr = irhou1_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u1_dx_ll, u1_dx_l, u1_dx_c, u1_dx_r, u1_dx_rr;
		u1_dx_ll = rhou1_dx_ll / rho_dx_ll;
		u1_dx_l = rhou1_dx_l / rho_dx_l;
		u1_dx_c = rhou1_dx_c / rho_dx_c;
		u1_dx_r = rhou1_dx_r / rho_dx_r;
		u1_dx_rr = rhou1_dx_rr / rho_dx_rr;

		odu1dx[gidx] = dns_pDer1(u1_dx_ll, u1_dx_l, u1_dx_r, u1_dx_rr, DX);
		tmp0 += dns_pDer2(u1_dx_ll, u1_dx_l, u1_dx_c, u1_dx_r, u1_dx_rr, DX);

		tmp_du1d2xi[gidx] += tmp0;


		double u1_dx_ll_dy_ll = irhou1_ll[dy_ll] / irho_ll[dy_ll];
		double u1_dx_l_dy_ll = irhou1_l[dy_ll] / irho_l[dy_ll];
		double u1_dx_r_dy_ll = irhou1_r[dy_ll] / irho_r[dy_ll];
		double u1_dx_rr_dy_ll = irhou1_rr[dy_ll] / irho_rr[dy_ll];

		double du1dx_dy_ll = dns_pDer1(u1_dx_ll_dy_ll, u1_dx_l_dy_ll, u1_dx_r_dy_ll, u1_dx_rr_dy_ll, DX);

		double u1_dx_ll_dy_l = irhou1_ll[dy_l] / irho_ll[dy_l];
		double u1_dx_l_dy_l = irhou1_l[dy_l] / irho_l[dy_l];
		double u1_dx_r_dy_l = irhou1_r[dy_l] / irho_r[dy_l];
		double u1_dx_rr_dy_l = irhou1_rr[dy_l] / irho_rr[dy_l];

		double du1dx_dy_l = dns_pDer1(u1_dx_ll_dy_l, u1_dx_l_dy_l, u1_dx_r_dy_l, u1_dx_rr_dy_l, DX);

		double u1_dx_ll_dy_r = irhou1_ll[dy_r] / irho_ll[dy_r];
		double u1_dx_l_dy_r = irhou1_l[dy_r] / irho_l[dy_r];
		double u1_dx_r_dy_r = irhou1_r[dy_r] / irho_r[dy_r];
		double u1_dx_rr_dy_r = irhou1_rr[dy_r] / irho_rr[dy_r];

		double du1dx_dy_r = dns_pDer1(u1_dx_ll_dy_r, u1_dx_l_dy_r, u1_dx_r_dy_r, u1_dx_rr_dy_r, DX);

		double u1_dx_ll_dy_rr = irhou1_ll[dy_rr] / irho_ll[dy_rr];
		double u1_dx_l_dy_rr = irhou1_l[dy_rr] / irho_l[dy_rr];
		double u1_dx_r_dy_rr = irhou1_r[dy_rr] / irho_r[dy_rr];
		double u1_dx_rr_dy_rr = irhou1_rr[dy_rr] / irho_rr[dy_rr];

		double du1dx_dy_rr = dns_pDer1(u1_dx_ll_dy_rr, u1_dx_l_dy_rr, u1_dx_r_dy_rr, u1_dx_rr_dy_rr, DX);

		tmp_du0d2xi[gidx] += 1./3. * dns_pDer1(du1dx_dy_ll, du1dx_dy_l, du1dx_dy_r, du1dx_dy_rr, DY);

		// Calculate idx with periodic boundary condition
		int32_t dy_ll_dz_ll, dy_ll_dz_l, dy_ll_dz_r, dy_ll_dz_rr;
		IDX((NY+Y-2)%NY, (NZ+Z-2)%NZ, dy_ll_dz_ll, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z-1)%NZ, dy_ll_dz_l, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+1)%NZ, dy_ll_dz_r, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+2)%NZ, dy_ll_dz_rr, NZ);

		double u1_dz_ll_dy_ll = irhou1_c[dy_ll_dz_ll] / irho_c[dy_ll_dz_ll];
		double u1_dz_l_dy_ll = irhou1_c[dy_ll_dz_l] / irho_c[dy_ll_dz_l];
		double u1_dz_r_dy_ll = irhou1_c[dy_ll_dz_r] / irho_c[dy_ll_dz_r];
		double u1_dz_rr_dy_ll = irhou1_c[dy_ll_dz_rr] / irho_c[dy_ll_dz_rr];

		double du1dz_dy_ll = dns_pDer1(u1_dz_ll_dy_ll, u1_dz_l_dy_ll, u1_dz_r_dy_ll, u1_dz_rr_dy_ll, DZ);


		int32_t dy_l_dz_ll, dy_l_dz_l, dy_l_dz_r, dy_l_dz_rr;
		IDX((NY+Y-1)%NY, (NZ+Z-2)%NZ, dy_l_dz_ll, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z-1)%NZ, dy_l_dz_l, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+1)%NZ, dy_l_dz_r, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+2)%NZ, dy_l_dz_rr, NZ);

		double u1_dz_ll_dy_l = irhou1_c[dy_l_dz_ll] / irho_c[dy_l_dz_ll];
		double u1_dz_l_dy_l = irhou1_c[dy_l_dz_l] / irho_c[dy_l_dz_l];
		double u1_dz_r_dy_l = irhou1_c[dy_l_dz_r] / irho_c[dy_l_dz_r];
		double u1_dz_rr_dy_l = irhou1_c[dy_l_dz_rr] / irho_c[dy_l_dz_rr];

		double du1dz_dy_l = dns_pDer1(u1_dz_ll_dy_l, u1_dz_l_dy_l, u1_dz_r_dy_l, u1_dz_rr_dy_l, DZ);


		int32_t dy_r_dz_ll, dy_r_dz_l, dy_r_dz_r, dy_r_dz_rr;
		IDX((NY+Y+1)%NY, (NZ+Z-2)%NZ, dy_r_dz_ll, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z-1)%NZ, dy_r_dz_l, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+1)%NZ, dy_r_dz_r, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+2)%NZ, dy_r_dz_rr, NZ);

		double u1_dz_ll_dy_r = irhou1_c[dy_r_dz_ll] / irho_c[dy_r_dz_ll];
		double u1_dz_l_dy_r = irhou1_c[dy_r_dz_l] / irho_c[dy_r_dz_l];
		double u1_dz_r_dy_r = irhou1_c[dy_r_dz_r] / irho_c[dy_r_dz_r];
		double u1_dz_rr_dy_r = irhou1_c[dy_r_dz_rr] / irho_c[dy_r_dz_rr];

		double du1dz_dy_r = dns_pDer1(u1_dz_ll_dy_r, u1_dz_l_dy_r, u1_dz_r_dy_r, u1_dz_rr_dy_r, DZ);


		int32_t dy_rr_dz_ll, dy_rr_dz_l, dy_rr_dz_r, dy_rr_dz_rr;
		IDX((NY+Y+2)%NY, (NZ+Z-2)%NZ, dy_rr_dz_ll, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z-1)%NZ, dy_rr_dz_l, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+1)%NZ, dy_rr_dz_r, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+2)%NZ, dy_rr_dz_rr, NZ);

		double u1_dz_ll_dy_rr = irhou1_c[dy_rr_dz_ll] / irho_c[dy_rr_dz_ll];
		double u1_dz_l_dy_rr = irhou1_c[dy_rr_dz_l] / irho_c[dy_rr_dz_l];
		double u1_dz_r_dy_rr = irhou1_c[dy_rr_dz_r] / irho_c[dy_rr_dz_r];
		double u1_dz_rr_dy_rr = irhou1_c[dy_rr_dz_rr] / irho_c[dy_rr_dz_rr];

		double du1dz_dy_rr = dns_pDer1(u1_dz_ll_dy_rr, u1_dz_l_dy_rr, u1_dz_r_dy_rr, u1_dz_rr_dy_rr, DZ);

		tmp_du2d2xi[gidx] += 1./3. * dns_pDer1(du1dz_dy_ll,  du1dz_dy_l, du1dz_dy_r, du1dz_dy_rr, DY);
	}
}

__global__ void dns_du2dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
	int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou2_c,
	/*order 1*/ double * __restrict__ irhou2_l, double * __restrict__ irhou2_r,
	/*order 2*/ double * __restrict__ irhou2_ll, double * __restrict__ irhou2_rr,
	/*order 0*/ double * __restrict__ odu2dx,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		double tmp0 = 0;

		double rhou2_dy_ll, rhou2_dy_l, rhou2_dy_c, rhou2_dy_r, rhou2_dy_rr;
		rhou2_dy_ll = irhou2_c[dy_ll];
		rhou2_dy_l = irhou2_c[dy_l];
		rhou2_dy_c = irhou2_c[gidx];
		rhou2_dy_r = irhou2_c[dy_r];
		rhou2_dy_rr = irhou2_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u2_dy_ll, u2_dy_l, u2_dy_c, u2_dy_r, u2_dy_rr;
		u2_dy_ll = rhou2_dy_ll / rho_dy_ll;
		u2_dy_l = rhou2_dy_l / rho_dy_l;
		u2_dy_c = rhou2_dy_c / rho_dy_c;
		u2_dy_r = rhou2_dy_r / rho_dy_r;
		u2_dy_rr = rhou2_dy_rr / rho_dy_rr;

		tmp0 += dns_pDer2(u2_dy_ll, u2_dy_l, u2_dy_c, u2_dy_r, u2_dy_rr, DY);

		double rhou2_dz_ll, rhou2_dz_l, rhou2_dz_c, rhou2_dz_r, rhou2_dz_rr;
		rhou2_dz_ll = irhou2_c[dz_ll];
		rhou2_dz_l = irhou2_c[dz_l];
		rhou2_dz_c = irhou2_c[gidx];
		rhou2_dz_r = irhou2_c[dz_r];
		rhou2_dz_rr = irhou2_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u2_dz_ll, u2_dz_l, u2_dz_c, u2_dz_r, u2_dz_rr;
		u2_dz_ll = rhou2_dz_ll / rho_dz_ll;
		u2_dz_l = rhou2_dz_l / rho_dz_l;
		u2_dz_c = rhou2_dz_c / rho_dz_c;
		u2_dz_r = rhou2_dz_r / rho_dz_r;
		u2_dz_rr = rhou2_dz_rr / rho_dz_rr;

		tmp0 += 4./3. * dns_pDer2(u2_dz_ll, u2_dz_l, u2_dz_c, u2_dz_r, u2_dz_rr, DZ);

		double rhou2_dx_ll, rhou2_dx_l, rhou2_dx_c, rhou2_dx_r, rhou2_dx_rr;
		rhou2_dx_ll = irhou2_ll[gidx];
		rhou2_dx_l = irhou2_l[gidx];
		rhou2_dx_c = irhou2_c[gidx];
		rhou2_dx_r = irhou2_r[gidx];
		rhou2_dx_rr = irhou2_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u2_dx_ll, u2_dx_l, u2_dx_c, u2_dx_r, u2_dx_rr;
		u2_dx_ll = rhou2_dx_ll / rho_dx_ll;
		u2_dx_l = rhou2_dx_l / rho_dx_l;
		u2_dx_c = rhou2_dx_c / rho_dx_c;
		u2_dx_r = rhou2_dx_r / rho_dx_r;
		u2_dx_rr = rhou2_dx_rr / rho_dx_rr;

		//odrhou2dx[gidx] = dns_pDer1(rhou2_dx_ll, rhou2_dx_l, rhou2_dx_r, rhou2_dx_rr, DX);
		odu2dx[gidx]= dns_pDer1(u2_dx_ll, u2_dx_l, u2_dx_r, u2_dx_rr, DX);
		tmp0 += dns_pDer2(u2_dx_ll, u2_dx_l, u2_dx_c, u2_dx_r, u2_dx_rr, DX);

		tmp_du2d2xi[gidx] += tmp0;


		double u2_dx_ll_dz_ll = irhou2_ll[dz_ll] / irho_ll[dz_ll];
		double u2_dx_l_dz_ll = irhou2_l[dz_ll] / irho_l[dz_ll];
		double u2_dx_r_dz_ll = irhou2_r[dz_ll] / irho_r[dz_ll];
		double u2_dx_rr_dz_ll = irhou2_rr[dz_ll] / irho_rr[dz_ll];

		double du2dx_dz_ll = dns_pDer1(u2_dx_ll_dz_ll, u2_dx_l_dz_ll, u2_dx_r_dz_ll, u2_dx_rr_dz_ll, DX);

		double u2_dx_ll_dz_l = irhou2_ll[dz_l] / irho_ll[dz_l];
		double u2_dx_l_dz_l = irhou2_l[dz_l] / irho_l[dz_l];
		double u2_dx_r_dz_l = irhou2_r[dz_l] / irho_r[dz_l];
		double u2_dx_rr_dz_l = irhou2_rr[dz_l] / irho_rr[dz_l];

		double du2dx_dz_l = dns_pDer1(u2_dx_ll_dz_l, u2_dx_l_dz_l, u2_dx_r_dz_l, u2_dx_rr_dz_l, DX);

		double u2_dx_ll_dz_r = irhou2_ll[dz_r] / irho_ll[dz_r];
		double u2_dx_l_dz_r = irhou2_l[dz_r] / irho_l[dz_r];
		double u2_dx_r_dz_r = irhou2_r[dz_r] / irho_r[dz_r];
		double u2_dx_rr_dz_r = irhou2_rr[dz_r] / irho_rr[dz_r];

		double du2dx_dz_r = dns_pDer1(u2_dx_ll_dz_r, u2_dx_l_dz_r, u2_dx_r_dz_r, u2_dx_rr_dz_r, DX);

		double u2_dx_ll_dz_rr = irhou2_ll[dz_rr] / irho_ll[dz_rr];
		double u2_dx_l_dz_rr = irhou2_l[dz_rr] / irho_l[dz_rr];
		double u2_dx_r_dz_rr = irhou2_r[dz_rr] / irho_r[dz_rr];
		double u2_dx_rr_dz_rr = irhou2_rr[dz_rr] / irho_rr[dz_rr];

		double du2dx_dz_rr = dns_pDer1(u2_dx_ll_dz_rr, u2_dx_l_dz_rr, u2_dx_r_dz_rr, u2_dx_rr_dz_rr, DX);

		tmp_du0d2xi[gidx] += 1./3. * dns_pDer1(du2dx_dz_ll, du2dx_dz_l, du2dx_dz_r, du2dx_dz_rr, DZ);


		// Calculate idx with periodic boundary condition
		int32_t dy_ll_dz_ll, dy_ll_dz_l, dy_ll_dz_r, dy_ll_dz_rr;
		IDX((NY+Y-2)%NY, (NZ+Z-2)%NZ, dy_ll_dz_ll, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z-1)%NZ, dy_ll_dz_l, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+1)%NZ, dy_ll_dz_r, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+2)%NZ, dy_ll_dz_rr, NZ);

		double u2_dz_ll_dy_ll = irhou2_c[dy_ll_dz_ll] / irho_c[dy_ll_dz_ll];
		double u2_dz_l_dy_ll = irhou2_c[dy_ll_dz_l] / irho_c[dy_ll_dz_l];
		double u2_dz_r_dy_ll = irhou2_c[dy_ll_dz_r] / irho_c[dy_ll_dz_r];
		double u2_dz_rr_dy_ll = irhou2_c[dy_ll_dz_rr] / irho_c[dy_ll_dz_rr];

		double du2dz_dy_ll = dns_pDer1(u2_dz_ll_dy_ll, u2_dz_l_dy_ll, u2_dz_r_dy_ll, u2_dz_rr_dy_ll, DZ);


		int32_t dy_l_dz_ll, dy_l_dz_l, dy_l_dz_r, dy_l_dz_rr;
		IDX((NY+Y-1)%NY, (NZ+Z-2)%NZ, dy_l_dz_ll, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z-1)%NZ, dy_l_dz_l, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+1)%NZ, dy_l_dz_r, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+2)%NZ, dy_l_dz_rr, NZ);

		double u2_dz_ll_dy_l = irhou2_c[dy_l_dz_ll] / irho_c[dy_l_dz_ll];
		double u2_dz_l_dy_l = irhou2_c[dy_l_dz_l] / irho_c[dy_l_dz_l];
		double u2_dz_r_dy_l = irhou2_c[dy_l_dz_r] / irho_c[dy_l_dz_r];
		double u2_dz_rr_dy_l = irhou2_c[dy_l_dz_rr] / irho_c[dy_l_dz_rr];

		double du2dz_dy_l = dns_pDer1(u2_dz_ll_dy_l, u2_dz_l_dy_l, u2_dz_r_dy_l, u2_dz_rr_dy_l, DZ);


		int32_t dy_r_dz_ll, dy_r_dz_l, dy_r_dz_r, dy_r_dz_rr;
		IDX((NY+Y+1)%NY, (NZ+Z-2)%NZ, dy_r_dz_ll, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z-1)%NZ, dy_r_dz_l, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+1)%NZ, dy_r_dz_r, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+2)%NZ, dy_r_dz_rr, NZ);

		double u2_dz_ll_dy_r = irhou2_c[dy_r_dz_ll] / irho_c[dy_r_dz_ll];
		double u2_dz_l_dy_r = irhou2_c[dy_r_dz_l] / irho_c[dy_r_dz_l];
		double u2_dz_r_dy_r = irhou2_c[dy_r_dz_r] / irho_c[dy_r_dz_r];
		double u2_dz_rr_dy_r = irhou2_c[dy_r_dz_rr] / irho_c[dy_r_dz_rr];

		double du2dz_dy_r = dns_pDer1(u2_dz_ll_dy_r, u2_dz_l_dy_r, u2_dz_r_dy_r, u2_dz_rr_dy_r, DZ);


		int32_t dy_rr_dz_ll, dy_rr_dz_l, dy_rr_dz_r, dy_rr_dz_rr;
		IDX((NY+Y+2)%NY, (NZ+Z-2)%NZ, dy_rr_dz_ll, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z-1)%NZ, dy_rr_dz_l, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+1)%NZ, dy_rr_dz_r, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+2)%NZ, dy_rr_dz_rr, NZ);

		double u2_dz_ll_dy_rr = irhou2_c[dy_rr_dz_ll] / irho_c[dy_rr_dz_ll];
		double u2_dz_l_dy_rr = irhou2_c[dy_rr_dz_l] / irho_c[dy_rr_dz_l];
		double u2_dz_r_dy_rr = irhou2_c[dy_rr_dz_r] / irho_c[dy_rr_dz_r];
		double u2_dz_rr_dy_rr = irhou2_c[dy_rr_dz_rr] / irho_c[dy_rr_dz_rr];

		double du2dz_dy_rr = dns_pDer1(u2_dz_ll_dy_rr, u2_dz_l_dy_rr, u2_dz_r_dy_rr, u2_dz_rr_dy_rr, DZ);

		tmp_du1d2xi[gidx] += 1./3. * dns_pDer1(du2dz_dy_ll, du2dz_dy_l, du2dz_dy_r, du2dz_dy_rr, DY);
	}
}


__global__ void dns_drhoETpdyz(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 0*/ double * __restrict__ irhou0_c,
	/*order 0*/ double * __restrict__ irhou1_c,
	/*order 0*/ double * __restrict__ irhou2_c,
	/*order 0*/ double * __restrict__ irhoE_c,
	/*order 0*/ double * __restrict__ idu0dx,
	/*order 0*/ double * __restrict__ idu1dx,
	/*order 0*/ double * __restrict__ idu2dx,
	int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
	/*order 0*/ double * __restrict__ Res_rho,
	/*order 0*/ double * __restrict__ Res_rhou0,
	/*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ Res_rhou2,
	/*order 0*/ double * __restrict__ Res_rhoE) {

	__shared__ double s_rho_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_invrho_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rhoE_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo

	__shared__ double s_u0_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_u1_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_u2_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo

	__shared__ double s_p_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo


	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t cb, rb;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &cb, &rb);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &cb, &rb);

	double tmp0 = 1. / (MINF * MINF * PR * RE * (GAMA - 1));


	if (gidx<block_ncc) {
		rb+=2;
		cb+=2;

		s_rho_c[rb][cb] = irho_c[gidx];
		s_invrho_c[rb][cb] = 1./s_rho_c[rb][cb];
		s_rhoE_c[rb][cb] = irhoE_c[gidx];

		s_u0_c[rb][cb] = irhou0_c[gidx] / s_rho_c[rb][cb];
		s_u1_c[rb][cb] = irhou1_c[gidx] / s_rho_c[rb][cb];
		s_u2_c[rb][cb] = irhou2_c[gidx] / s_rho_c[rb][cb];

		s_p_c[rb][cb] = calp_ui(s_rhoE_c[rb][cb], s_rho_c[rb][cb], s_u0_c[rb][cb], s_u1_c[rb][cb], s_u2_c[rb][cb]);


		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// get halos
		if (rb < 4) {
			IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
			s_rho_c[rb-2][cb] = irho_c[dy_ll];
			s_invrho_c[rb-2][cb] = 1./s_rho_c[rb-2][cb];
			s_rhoE_c[rb-2][cb] = irhoE_c[dy_ll];

			s_u0_c[rb-2][cb] = irhou0_c[dy_ll] / s_rho_c[rb-2][cb];
			s_u1_c[rb-2][cb] = irhou1_c[dy_ll] / s_rho_c[rb-2][cb];
			s_u2_c[rb-2][cb] = irhou2_c[dy_ll] / s_rho_c[rb-2][cb];

			s_p_c[rb-2][cb] = calp_ui(s_rhoE_c[rb-2][cb], s_rho_c[rb-2][cb], s_u0_c[rb-2][cb], s_u1_c[rb-2][cb], s_u2_c[rb-2][cb]);

		}
		if (rb >= BLOCKSIZE_Y) {
			IDX((NY+Y+2)%NY, Z, dy_rr, NZ);
			s_rho_c[rb+2][cb] = irho_c[dy_rr];
			s_invrho_c[rb+2][cb] = 1./s_rho_c[rb+2][cb];
			s_rhoE_c[rb+2][cb] = irhoE_c[dy_rr];

			s_u0_c[rb+2][cb] = irhou0_c[dy_rr] / s_rho_c[rb+2][cb];
			s_u1_c[rb+2][cb] = irhou1_c[dy_rr] / s_rho_c[rb+2][cb];
			s_u2_c[rb+2][cb] = irhou2_c[dy_rr] / s_rho_c[rb+2][cb];

			s_p_c[rb+2][cb] = calp_ui(s_rhoE_c[rb+2][cb], s_rho_c[rb+2][cb], s_u0_c[rb+2][cb], s_u1_c[rb+2][cb], s_u2_c[rb+2][cb]);

		}

		// get halos
		if (cb < 4) {
			IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
			s_rho_c[rb][cb-2] = irho_c[dz_ll];
			s_invrho_c[rb][cb-2] = 1./s_rho_c[rb][cb-2];
			s_rhoE_c[rb][cb-2] = irhoE_c[dz_ll];

			s_u0_c[rb][cb-2] = irhou0_c[dz_ll] / s_rho_c[rb][cb-2];
			s_u1_c[rb][cb-2] = irhou1_c[dz_ll] / s_rho_c[rb][cb-2];
			s_u2_c[rb][cb-2] = irhou2_c[dz_ll] / s_rho_c[rb][cb-2];

			s_p_c[rb][cb-2] = calp_ui(s_rhoE_c[rb][cb-2], s_rho_c[rb][cb-2], s_u0_c[rb][cb-2], s_u1_c[rb][cb-2], s_u2_c[rb][cb-2]);

		}
		if (cb >= BLOCKSIZE_Z) {
			IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);
			s_rho_c[rb][cb+2] = irho_c[dz_rr];
			s_invrho_c[rb][cb+2] = 1./s_rho_c[rb][cb+2];
			s_rhoE_c[rb][cb+2] = irhoE_c[dz_rr];

			s_u0_c[rb][cb+2] = irhou0_c[dz_rr] / s_rho_c[rb][cb+2];
			s_u1_c[rb][cb+2] = irhou1_c[dz_rr] / s_rho_c[rb][cb+2];
			s_u2_c[rb][cb+2] = irhou2_c[dz_rr] / s_rho_c[rb][cb+2];

			s_p_c[rb][cb+2] = calp_ui(s_rhoE_c[rb][cb+2], s_rho_c[rb][cb+2], s_u0_c[rb][cb+2], s_u1_c[rb][cb+2], s_u2_c[rb][cb+2]);


		}

		__syncthreads();

		double tmp_Res_rho = 0;
		double tmp_Res_rhou0 = 0;
		double tmp_Res_rhou1 = 0;
		double tmp_Res_rhou2 = 0;
		double tmp_Res_rhoE = 0;
		
		double tmp_dTd2xi = 0;
		double tmp1;
		double tmp2;
		double tmp3;
		double tmp4;

		tmp1 = idu0dx[gidx];
		tmp2 = dns_pDer1(s_u1_c[rb-2][cb], s_u1_c[rb-1][cb], s_u1_c[rb+1][cb], s_u1_c[rb+2][cb], DY);
		tmp3 = dns_pDer1(s_u2_c[rb][cb-2], s_u2_c[rb][cb-1], s_u2_c[rb][cb+1], s_u2_c[rb][cb+2], DZ);

		tmp4 = -0.5 * (tmp1 + tmp2 + tmp3);
		tmp_Res_rho += tmp4 * s_rho_c[rb][cb];
		tmp_Res_rhou0 += tmp4 * s_rho_c[rb][cb] * s_u0_c[rb][cb];
		tmp_Res_rhou1 += tmp4 * s_rho_c[rb][cb] * s_u1_c[rb][cb];
		tmp_Res_rhou2 += tmp4 * s_rho_c[rb][cb] * s_u2_c[rb][cb];
		tmp_Res_rhoE += tmp4 * s_rhoE_c[rb][cb];

		tmp_Res_rhoE += 1./RE * (-2./3. * tmp1 - 2./3. * tmp2 + 4./3. * tmp3) * tmp3;
		tmp_Res_rhoE += 1./RE * (-2./3. * tmp1 + 4./3. * tmp2 - 2./3. * tmp3) * tmp2;
		tmp_Res_rhoE += 1./RE * ( 4./3. * tmp1 - 2./3. * tmp2 - 2./3. * tmp3) * tmp1;

		tmp1 = idu1dx[gidx];
		tmp2 = dns_pDer1(s_u0_c[rb-2][cb], s_u0_c[rb-1][cb], s_u0_c[rb+1][cb], s_u0_c[rb+2][cb], DY);
		tmp_Res_rhoE += 1./RE * (tmp2 + tmp1) * tmp2 + 1./RE * (tmp2 + tmp1) * tmp1;

		tmp3 = idu2dx[gidx];
		tmp4 = dns_pDer1(s_u0_c[rb][cb-2], s_u0_c[rb][cb-1], s_u0_c[rb][cb+1], s_u0_c[rb][cb+2], DZ);
		tmp_Res_rhoE += 1./RE * (tmp4 + tmp3) * tmp4 + 1./RE * (tmp4 + tmp3) * tmp3;

		
		tmp_Res_rho += -0.5 * dns_pDer1(s_rho_c[rb-2][cb], s_rho_c[rb-1][cb], s_rho_c[rb+1][cb], s_rho_c[rb+2][cb], DY) * s_u1_c[rb][cb];
		tmp_Res_rhou0 += -0.5 * dns_pDer1(s_rho_c[rb-2][cb] * s_u0_c[rb-2][cb] * s_u1_c[rb-2][cb], s_rho_c[rb-1][cb] * s_u0_c[rb-1][cb] * s_u1_c[rb-1][cb], s_rho_c[rb+1][cb] * s_u0_c[rb+1][cb] * s_u1_c[rb+1][cb], s_rho_c[rb+2][cb] * s_u0_c[rb+2][cb] * s_u1_c[rb+2][cb], DY);
		tmp_Res_rhou2 += -0.5 * dns_pDer1(s_rho_c[rb-2][cb] * s_u2_c[rb-2][cb] * s_u1_c[rb-2][cb], s_rho_c[rb-1][cb] * s_u2_c[rb-1][cb] * s_u1_c[rb-1][cb], s_rho_c[rb+1][cb] * s_u2_c[rb+1][cb] * s_u1_c[rb+1][cb], s_rho_c[rb+2][cb] * s_u2_c[rb+2][cb] * s_u1_c[rb+2][cb], DY);
		tmp_Res_rhou0 += -0.5 * dns_pDer1(s_rho_c[rb-2][cb] * s_u0_c[rb-2][cb], s_rho_c[rb-1][cb] * s_u0_c[rb-1][cb], s_rho_c[rb+1][cb] * s_u0_c[rb+1][cb], s_rho_c[rb+2][cb] * s_u0_c[rb+2][cb], DY) * s_u1_c[rb][cb];
		tmp_Res_rhou2 += -0.5 * dns_pDer1(s_rho_c[rb-2][cb] * s_u2_c[rb-2][cb], s_rho_c[rb-1][cb] * s_u2_c[rb-1][cb], s_rho_c[rb+1][cb] * s_u2_c[rb+1][cb], s_rho_c[rb+2][cb] * s_u2_c[rb+2][cb], DY) * s_u1_c[rb][cb];
		tmp_Res_rhou1 += -0.5 * dns_pDer1(s_rho_c[rb-2][cb] * s_u1_c[rb-2][cb] * s_u1_c[rb-2][cb], s_rho_c[rb-1][cb] * s_u1_c[rb-1][cb] * s_u1_c[rb-1][cb], s_rho_c[rb+1][cb] * s_u1_c[rb+1][cb] * s_u1_c[rb+1][cb], s_rho_c[rb+2][cb] * s_u1_c[rb+2][cb] * s_u1_c[rb+2][cb], DY);
		tmp1 = dns_pDer1(s_rho_c[rb-2][cb] * s_u1_c[rb-2][cb], s_rho_c[rb-1][cb] * s_u1_c[rb-1][cb], s_rho_c[rb+1][cb] * s_u1_c[rb+1][cb], s_rho_c[rb+2][cb] * s_u1_c[rb+2][cb], DY);
		tmp_Res_rhou1 += -0.5 * tmp1 * s_u1_c[rb][cb];
		tmp_Res_rho += -0.5 * tmp1;

	
		tmp_Res_rho += -0.5 * dns_pDer1(s_rho_c[rb][cb-2], s_rho_c[rb][cb-1], s_rho_c[rb][cb+1], s_rho_c[rb][cb+2], DZ) * s_u2_c[rb][cb];
		tmp_Res_rhou0 += -0.5 * dns_pDer1(s_rho_c[rb][cb-2] * s_u0_c[rb][cb-2] * s_u2_c[rb][cb-2], s_rho_c[rb][cb-1] * s_u0_c[rb][cb-1] * s_u2_c[rb][cb-1], s_rho_c[rb][cb+1] * s_u0_c[rb][cb+1] * s_u2_c[rb][cb+1], s_rho_c[rb][cb+2] * s_u0_c[rb][cb+2] * s_u2_c[rb][cb+2], DZ);
		tmp_Res_rhou1 += -0.5 * dns_pDer1(s_rho_c[rb][cb-2] * s_u1_c[rb][cb-2] * s_u2_c[rb][cb-2], s_rho_c[rb][cb-1] * s_u1_c[rb][cb-1] * s_u2_c[rb][cb-1], s_rho_c[rb][cb+1] * s_u1_c[rb][cb+1] * s_u2_c[rb][cb+1], s_rho_c[rb][cb+2] * s_u1_c[rb][cb+2] * s_u2_c[rb][cb+2], DZ);
		tmp_Res_rhou0 += -0.5 * dns_pDer1(s_rho_c[rb][cb-2] * s_u0_c[rb][cb-2], s_rho_c[rb][cb-1] * s_u0_c[rb][cb-1], s_rho_c[rb][cb+1] * s_u0_c[rb][cb+1], s_rho_c[rb][cb+2] * s_u0_c[rb][cb+2], DZ) * s_u2_c[rb][cb];
		tmp_Res_rhou1 += -0.5 * dns_pDer1(s_rho_c[rb][cb-2] * s_u1_c[rb][cb-2], s_rho_c[rb][cb-1] * s_u1_c[rb][cb-1], s_rho_c[rb][cb+1] * s_u1_c[rb][cb+1], s_rho_c[rb][cb+2] * s_u1_c[rb][cb+2], DZ) * s_u2_c[rb][cb];
		tmp_Res_rhou2 += -0.5 * dns_pDer1(s_rho_c[rb][cb-2] * s_u2_c[rb][cb-2] * s_u2_c[rb][cb-2], s_rho_c[rb][cb-1] * s_u2_c[rb][cb-1] * s_u2_c[rb][cb-1], s_rho_c[rb][cb+1] * s_u2_c[rb][cb+1] * s_u2_c[rb][cb+1], s_rho_c[rb][cb+2] * s_u2_c[rb][cb+2] * s_u2_c[rb][cb+2], DZ);
		tmp1 = dns_pDer1(s_rho_c[rb][cb-2] * s_u2_c[rb][cb-2], s_rho_c[rb][cb-1] * s_u2_c[rb][cb-1], s_rho_c[rb][cb+1] * s_u2_c[rb][cb+1], s_rho_c[rb][cb+2] * s_u2_c[rb][cb+2], DZ);
		tmp_Res_rhou2 += -0.5 * tmp1 * s_u2_c[rb][cb];
		tmp_Res_rho += -0.5 * tmp1;
		

		tmp1 = dns_pDer1(s_u1_c[rb][cb-2], s_u1_c[rb][cb-1], s_u1_c[rb][cb+1], s_u1_c[rb][cb+2], DZ);
		tmp2 = dns_pDer1(s_u2_c[rb-2][cb], s_u2_c[rb-1][cb], s_u2_c[rb+1][cb], s_u2_c[rb+2][cb], DY);
		tmp_Res_rhoE += 1./RE * (tmp1 + tmp2) * tmp1 + 1./RE * (tmp1 + tmp2) * tmp2;

		tmp_Res_rhou1 += -dns_pDer1(s_p_c[rb-2][cb], s_p_c[rb-1][cb], s_p_c[rb+1][cb], s_p_c[rb+2][cb], DY);
		tmp_Res_rhoE -= dns_pDer1(s_p_c[rb-2][cb] * s_u1_c[rb-2][cb], s_p_c[rb-1][cb] * s_u1_c[rb-1][cb], s_p_c[rb+1][cb] * s_u1_c[rb+1][cb], s_p_c[rb+2][cb] * s_u1_c[rb+2][cb], DY);
		tmp_dTd2xi  += dns_pDer2(calT_ui(s_p_c[rb-2][cb], s_invrho_c[rb-2][cb]), calT_ui(s_p_c[rb-1][cb], s_invrho_c[rb-1][cb]), calT_ui(s_p_c[rb][cb], s_invrho_c[rb][cb]), calT_ui(s_p_c[rb+1][cb], s_invrho_c[rb+1][cb]), calT_ui(s_p_c[rb+2][cb], s_invrho_c[rb+2][cb]), DY);
		
		tmp_Res_rhou2 += -dns_pDer1(s_p_c[rb][cb-2], s_p_c[rb][cb-1], s_p_c[rb][cb+1], s_p_c[rb][cb+2], DZ);
		tmp_Res_rhoE -= dns_pDer1(s_p_c[rb][cb-2]* s_u2_c[rb][cb-2], s_p_c[rb][cb-1] * s_u2_c[rb][cb-1], s_p_c[rb][cb+1] * s_u2_c[rb][cb+1], s_p_c[rb][cb+2] * s_u2_c[rb][cb+2], DZ);
		tmp_dTd2xi  +=  dns_pDer2(calT_ui(s_p_c[rb][cb-2], s_invrho_c[rb][cb-2]), calT_ui(s_p_c[rb][cb-1], s_invrho_c[rb][cb-1]), calT_ui(s_p_c[rb][cb], s_invrho_c[rb][cb]), calT_ui(s_p_c[rb][cb+1], s_invrho_c[rb][cb+1]), calT_ui(s_p_c[rb][cb+2], s_invrho_c[rb][cb+2]), DZ);	

		tmp_Res_rhoE += -0.5 * dns_pDer1(s_rhoE_c[rb-2][cb], s_rhoE_c[rb-1][cb], s_rhoE_c[rb+1][cb], s_rhoE_c[rb+2][cb], DY) * s_u1_c[rb][cb];
		tmp_Res_rhoE += -0.5 * dns_pDer1(s_rhoE_c[rb-2][cb] * s_u1_c[rb-2][cb], s_rhoE_c[rb-1][cb] * s_u1_c[rb-1][cb], s_rhoE_c[rb+1][cb] * s_u1_c[rb+1][cb], s_rhoE_c[rb+2][cb] * s_u1_c[rb+2][cb], DY);
		tmp_Res_rhoE += -0.5 * dns_pDer1(s_rhoE_c[rb][cb-2], s_rhoE_c[rb][cb-1], s_rhoE_c[rb][cb+1], s_rhoE_c[rb][cb+2], DZ) * s_u2_c[rb][cb];
		tmp_Res_rhoE += -0.5 * dns_pDer1(s_rhoE_c[rb][cb-2] * s_u2_c[rb][cb-2], s_rhoE_c[rb][cb-1] * s_u2_c[rb][cb-1], s_rhoE_c[rb][cb+1] * s_u2_c[rb][cb+1], s_rhoE_c[rb][cb+2] * s_u2_c[rb][cb+2], DZ);
		
		tmp_Res_rhoE += tmp_dTd2xi * tmp0;

		Res_rho[gidx] = tmp_Res_rho;
		Res_rhou0[gidx] = tmp_Res_rhou0;
		Res_rhou1[gidx] = tmp_Res_rhou1;
		Res_rhou2[gidx] = tmp_Res_rhou2;
		Res_rhoE[gidx] = tmp_Res_rhoE;
			
	}

}

__global__ void dns_drhoETpdx(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou0_c,
	/*order 1*/ double * __restrict__ irhou0_l, double * __restrict__ irhou0_r,
	/*order 2*/ double * __restrict__ irhou0_ll, double * __restrict__ irhou0_rr,
	/*order 0*/ double * __restrict__ irhou1_c,
	/*order 1*/ double * __restrict__ irhou1_l, double * __restrict__ irhou1_r,
	/*order 2*/ double * __restrict__ irhou1_ll, double * __restrict__ irhou1_rr,
	/*order 0*/ double * __restrict__ irhou2_c,
	/*order 1*/ double * __restrict__ irhou2_l, double * __restrict__ irhou2_r,
	/*order 2*/ double * __restrict__ irhou2_ll, double * __restrict__ irhou2_rr,
	/*order 0*/ double * __restrict__ irhoE_c,
	/*order 1*/ double * __restrict__ irhoE_l, double * __restrict__ irhoE_r,
	/*order 2*/ double * __restrict__ irhoE_ll, double * __restrict__ irhoE_rr,
	int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
	/*order 0*/ double * __restrict__ Res_rho,
	/*order 0*/ double * __restrict__ Res_rhou0,
	/*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ Res_rhou2,
	/*order 0*/ double * __restrict__ Res_rhoE) {

	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t cb, rb;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &cb, &rb);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &cb, &rb);

	double tmp0 = 1. / (MINF * MINF * PR * RE * (GAMA - 1));

	double fac_ll = 1./12.;
	double fac_l = -2./3.;
	double fac_r = 2./3.;
	double fac_rr = -1./12.;

	double fac_dk = 1./DX;

	if (gidx<block_ncc) {

		
		double rho, invrho, rhou0, u0, rhou1, rhou2, rhoE, p;
		double drhodx;
		double dpdx, dpu0dx;
		double drhoEdx, drhoEu0dx;
		double drhou0u0dx, drhou1u0dx, drhou2u0dx;
		double dTd2x;
		double drhou0dx, drhou1dx, drhou2dx;

		// dx_ll
		rho = irho_ll[gidx];
		invrho = 1./rho;
		rhou0 = irhou0_ll[gidx];
		u0 = rhou0 * invrho;
		rhou1 = irhou1_ll[gidx];
		rhou2 = irhou2_ll[gidx];
		rhoE = irhoE_ll[gidx];
		p = calp_ui(rhoE, rho, u0 , rhou1 * invrho, rhou2 * invrho);

		drhodx = fac_ll * rho;
		dpdx = fac_ll * p;
		dpu0dx = fac_ll * p * sy_bc_ll * u0;
		drhoEdx = fac_ll * rhoE;
		drhoEu0dx = fac_ll * rhoE * sy_bc_ll * u0;
		drhou1u0dx = fac_ll * rhou1 * sy_bc_ll * u0;
		drhou2u0dx = fac_ll * rhou2 * sy_bc_ll * u0;
		drhou0u0dx = fac_ll * sy_bc_ll * rhou0 * sy_bc_ll * u0;

		drhou0dx = fac_ll * sy_bc_ll * rhou0;
		drhou1dx = fac_ll * rhou1;
		drhou2dx = fac_ll * rhou2;

		dTd2x = -fac_ll * calT_ui(p, invrho);

		// dx_l
		rho = irho_l[gidx];
		invrho = 1./rho;
		rhou0 = irhou0_l[gidx];
		u0 = rhou0 * invrho;
		rhou1 = irhou1_l[gidx];
		rhou2 = irhou2_l[gidx];
		rhoE = irhoE_l[gidx];
		p = calp_ui(rhoE, rho, u0 , rhou1 * invrho, rhou2 * invrho);

		drhodx += fac_l * rho;
		dpdx += fac_l * p;
		dpu0dx += fac_l* p * sy_bc_l * u0;
		drhoEdx += fac_l * rhoE;
		drhoEu0dx += fac_l * rhoE * sy_bc_l * u0;
		drhou1u0dx += fac_l * rhou1 * sy_bc_l * u0;
		drhou2u0dx += fac_l * rhou2 * sy_bc_l * u0;
		drhou0u0dx += fac_l * sy_bc_l * rhou0 * sy_bc_l * u0;

		drhou0dx += fac_l * sy_bc_l * rhou0;
		drhou1dx += fac_l * rhou1;
		drhou2dx += fac_l * rhou2;

		dTd2x += -2. * fac_l * calT_ui(p, invrho);

		// dx_r
		rho = irho_r[gidx];
		invrho = 1./rho;
		rhou0 = irhou0_r[gidx];
		u0 = rhou0 * invrho;
		rhou1 = irhou1_r[gidx];
		rhou2 = irhou2_r[gidx];
		rhoE = irhoE_r[gidx];
		p = calp_ui(rhoE, rho, u0 , rhou1 * invrho, rhou2 * invrho);

		drhodx += fac_r * rho;
		dpdx += fac_r * p;
		dpu0dx += fac_r* p * sy_bc_r * u0;
		drhoEdx += fac_r * rhoE;
		drhoEu0dx += fac_r * rhoE * sy_bc_r * u0;
		drhou1u0dx += fac_r * rhou1 * sy_bc_r * u0;
		drhou2u0dx += fac_r * rhou2 * sy_bc_r * u0;
		drhou0u0dx += fac_r * sy_bc_r * rhou0 * sy_bc_r * u0;

		drhou0dx += fac_r * sy_bc_r * rhou0;
		drhou1dx += fac_r * rhou1;
		drhou2dx += fac_r * rhou2;

		dTd2x += 2. * fac_r * calT_ui(p, invrho);

		// dx_rr
		rho = irho_rr[gidx];
		invrho = 1./rho;
		rhou0 = irhou0_rr[gidx];
		u0 = rhou0 * invrho;
		rhou1 = irhou1_rr[gidx];
		rhou2 = irhou2_rr[gidx];
		rhoE = irhoE_rr[gidx];
		p = calp_ui(rhoE, rho, u0 , rhou1 * invrho, rhou2 * invrho);

		drhodx += fac_rr * rho;
		dpdx += fac_rr * p;
		dpu0dx += fac_rr* p * sy_bc_rr * u0;
		drhoEdx += fac_rr * rhoE;
		drhoEu0dx += fac_rr * rhoE * sy_bc_rr * u0;
		drhou1u0dx += fac_rr * rhou1 * sy_bc_rr * u0;
		drhou2u0dx += fac_rr * rhou2 * sy_bc_rr * u0;
		drhou0u0dx += fac_rr * sy_bc_rr * rhou0 * sy_bc_rr * u0;

		drhou0dx += fac_rr * sy_bc_rr * rhou0;
		drhou1dx += fac_rr * rhou1;
		drhou2dx += fac_rr * rhou2;

		dTd2x += fac_rr * calT_ui(p, invrho);

		// dx_c
		rho = irho_c[gidx];
		invrho = 1./rho;
		rhou0 = irhou0_c[gidx];
		u0 = rhou0 * invrho;
		rhou1 = irhou1_c[gidx];
		rhou2 = irhou2_c[gidx];
		rhoE = irhoE_c[gidx];
		p = calp_ui(rhoE, rho, u0 , rhou1 * invrho, rhou2 * invrho);

		dTd2x += -5./2. * calT_ui(p, invrho);

		double tmp_Res_rho = 0;
		tmp_Res_rho += -0.5 * fac_dk * drhodx * u0;
		tmp_Res_rho += -0.5 * fac_dk * drhou0dx;

		double tmp_Res_rhou0 = 0;
		tmp_Res_rhou0 += - 0.5 * fac_dk * drhou0u0dx;
		tmp_Res_rhou0 += - fac_dk * dpdx;
		tmp_Res_rhou0 += -0.5 * fac_dk * drhou0dx * u0;

		double tmp_Res_rhou1 = 0;
		tmp_Res_rhou1 += -0.5 * fac_dk * drhou1u0dx;
		tmp_Res_rhou1 += -0.5 * fac_dk * drhou1dx * u0;

		double tmp_Res_rhou2 = 0;
		tmp_Res_rhou2 += -0.5 * fac_dk * drhou2u0dx;
		tmp_Res_rhou2 += -0.5 * fac_dk * drhou2dx * u0;

		double tmp_Res_rhoE = 0;
		tmp_Res_rhoE -= fac_dk * dpu0dx;
		tmp_Res_rhoE += -0.5 * fac_dk * drhoEdx * u0;
		tmp_Res_rhoE += -0.5 * fac_dk * drhoEu0dx;
		tmp_Res_rhoE += fac_dk * fac_dk * dTd2x * tmp0 ;

		Res_rho[gidx] += tmp_Res_rho;
		Res_rhou0[gidx] += tmp_Res_rhou0;
		Res_rhou1[gidx] += tmp_Res_rhou1;
		Res_rhou2[gidx] += tmp_Res_rhou2;
		Res_rhoE[gidx] += tmp_Res_rhoE;
	}

}

__global__ void dns_Res_StageAdvance(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ rho,
	/*order 0*/ double * __restrict__ rhou0,
	/*order 0*/ double * __restrict__ rhou1,
	/*order 0*/ double * __restrict__ rhou2,
	/*order 0*/ double * __restrict__ rhoE,
	
	/*order 0*/ double * __restrict__ irho_old,
	/*order 0*/ double * __restrict__ irhou0_old,
	/*order 0*/ double * __restrict__ irhou1_old,
	/*order 0*/ double * __restrict__ irhou2_old,
	/*order 0*/ double * __restrict__ irhoE_old,
	
	/*order 0*/ double * __restrict__ orho,
	/*order 0*/ double * __restrict__ orhou0,
	/*order 0*/ double * __restrict__ orhou1,
	/*order 0*/ double * __restrict__ orhou2,
	/*order 0*/ double * __restrict__ orhoE,

	/*order 0*/ double * __restrict__ orho_old,
	/*order 0*/ double * __restrict__ orhou0_old,
	/*order 0*/ double * __restrict__ orhou1_old,
	/*order 0*/ double * __restrict__ orhou2_old,
	/*order 0*/ double * __restrict__ orhoE_old,
	
	double rknew, double rkold,
	/*order 0*/ double * __restrict__ Res_rho,
	/*order 0*/ double * __restrict__ Res_rhou0,
	/*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ Res_rhou2,
	/*order 0*/ double * __restrict__ Res_rhoE,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {


	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t idx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	idx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);

	if (idx<block_ncc) {
		double lRes_rho = 0;
		double lRes_rhou0 = 0;
		double lRes_rhou1 = 0;
		double lRes_rhou2 = 0;
		double lRes_rhoE = 0;

		double tmp0;

		lRes_rho += Res_rho[idx];
		orho[idx] = DT * rknew * lRes_rho + irho_old[idx];
		orho_old[idx] = DT * rkold * lRes_rho + irho_old[idx];

		double frac0 = 1./RE;

		double lu0 = rhou0[idx] / rho[idx];
		tmp0 = frac0 * tmp_du0d2xi[idx];
		lRes_rhou0 += tmp0;
		lRes_rhoE += lu0 * tmp0;

		lRes_rhou0 += Res_rhou0[idx];
		orhou0[idx] = DT * rknew * lRes_rhou0 + irhou0_old[idx];
		orhou0_old[idx] = DT * rkold * lRes_rhou0 + irhou0_old[idx];


		double lu1 = rhou1[idx] / rho[idx];
		tmp0 = frac0 * tmp_du1d2xi[idx];
		lRes_rhou1 += tmp0;
		lRes_rhoE += lu1 * tmp0;

		lRes_rhou1 += Res_rhou1[idx];
		orhou1[idx] = DT * rknew * lRes_rhou1 + irhou1_old[idx];
		orhou1_old[idx] = DT * rkold * lRes_rhou1 + irhou1_old[idx];


		double lu2 = rhou2[idx] / rho[idx];
		tmp0 = frac0 * tmp_du2d2xi[idx];
		lRes_rhou2 += tmp0;
		lRes_rhoE += lu2 * tmp0;

		lRes_rhou2 += Res_rhou2[idx];
		orhou2[idx] = DT * rknew * lRes_rhou2 + irhou2_old[idx];
		orhou2_old[idx] = DT * rkold * lRes_rhou2 + irhou2_old[idx];

		lRes_rhoE += Res_rhoE[idx];

		orhoE[idx] = DT * rknew * lRes_rhoE + irhoE_old[idx];
		orhoE_old[idx] = DT * rkold * lRes_rhoE + irhoE_old[idx];

	}

}



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Utility
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void dns_copy(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ f,
	/*order 0*/ double * __restrict__ g) {
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		g[idx] = f[idx];
	}

}


__global__ void dns_init(const double * __restrict__ p_in, double * __restrict__ p_out) {
	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_header_size) {
		p_out[idx]=p_in[idx];
	}


}


__global__ void dns_DebugAdvance(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ f0_out,
	/*order 0*/ double * __restrict__ f0_in) {

	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		f0_out[idx] = f0_in[idx] + 1.0;

	}
}

__global__ void dns_Debug(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ f0_in, /*order 0*/ double * __restrict__ f0_out) {

	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<2) {
		printf("In: f0[%d] = %lf    ", idx, f0_in[idx]);
		if (idx == 0) {
			f0_out[idx] = f0_in[idx];
		}
		if (idx == 1) {
			printf("Out old: f0[%d] = %lf    ", idx, f0_out[idx]);
			f0_out[idx] = f0_in[idx] + 1;
			printf("Out new: f0[%d] = %lf    ", idx, f0_out[idx]);
		}

	}
}



void DS::caller_worker (double ** p_in, double ** p_out, int32_t i_part, int32_t i_super_cycle,
						int32_t order_in, int32_t order_out, int32_t iworker, int32_t nworker,
						hipStream_t * stream, int32_t threads_per_block, int32_t blockSize, int32_t myID) {

	//cout << "in:" << p_in[0] << " " << p_in[1] << " " << p_in[2] << " " << p_in[3] << " " << p_in[4] << " " << p_in[5] << endl;
	//cout << "out:" << p_out[0] << " " << p_out[1] << " " << p_out[2] << " " << p_out[3] << " " << p_out[4] << " " << p_out[5] << endl;

	// the order of arrays in p_in and p_out is:
	// center, left, right, left-left, right-right, left-left-left, right-right-right, and so on
	// entries can be NULL when invalid

	//cout << NX << ", " << NY << ", " << NZ << ", " << DX << ", " << DY << ", " << DZ <<  ", " <<  DT << ", " <<  GAMA <<  ", " <<  MINF <<  ", " <<  RE <<  ", " <<  PR << endl;


	int32_t global_worker_id = nworker * myID + iworker;
	int32_t n_global_worker = n_procs * n_worker;
	int32_t stage = (global_worker_id + n_global_worker * i_super_cycle) % 3;

	//cout << "Working on stage " << stage << endl;

	double rkold = RKOLD[stage];
	double rknew = RKNEW[stage];

	//cout << "rkold " << rkold << " rknew " << rknew << endl;

	
	
	// Sort out parts
	double* p_c = p_in[0];
	double* p_l = p_in[1];
	double* p_r = p_in[2];
	double* p_ll = p_in[3];
	double* p_rr = p_in[4];

	// Sort out parts
	double* p_c_out = p_out[0];

	// Symmetry BC u0 = -u0
	int sy_bc_ll = 1;
	int sy_bc_l  = 1;
	int sy_bc_r  = 1;
	int sy_bc_rr = 1;

	if (i_part == 0) {
		p_ll = p_rr;
		p_l = p_r;
		sy_bc_ll = -1;
		sy_bc_l  = -1;
	} else if(i_part == 1) {
		p_ll = p_c;
		sy_bc_ll = -1;
	} else if (i_part == my_n_part - 2) {
		p_rr = p_c;
		sy_bc_rr = -1;
	} else if (i_part == my_n_part - 1) {
		p_rr = p_ll;
		p_r = p_l;
		sy_bc_rr = -1;
		sy_bc_r  = -1;
	}

	// Offsets in pages for each field
	size_t offset_rho   = block_header_size + 0*block_ncc;
	size_t offset_rhou0 = block_header_size + 1*block_ncc;
	size_t offset_rhou1 = block_header_size + 2*block_ncc;
	size_t offset_rhou2 = block_header_size + 3*block_ncc;
	size_t offset_rhoE = block_header_size + 4*block_ncc;
	size_t offset_rho_old   = block_header_size + 5*block_ncc;
	size_t offset_rhou0_old = block_header_size + 6*block_ncc;
	size_t offset_rhou1_old = block_header_size + 7*block_ncc;
	size_t offset_rhou2_old = block_header_size + 8*block_ncc;
	size_t offset_rhoE_old = block_header_size + 9*block_ncc;
	size_t offset_tmp0 = block_header_size + 10*block_ncc;
	size_t offset_tmp1 = block_header_size + 11*block_ncc;
	size_t offset_tmp2 = block_header_size + 12*block_ncc;
	size_t offset_tmp3 = block_header_size + 13*block_ncc;
	size_t offset_tmp4 = block_header_size + 14*block_ncc;



	//cout << "Blocksize: " << blockSize << ", threads_per_block: " << threads_per_block << endl;

	threads_per_block = BLOCKSIZE_Z * BLOCKSIZE_Y;
	int32_t gridSize = (blockSize + threads_per_block - 1) / threads_per_block;

	//cout << "Slice Size: " << blockSize << ", gridSize: " << gridSize << ", started Threads: " << gridSize * threads_per_block << endl;


	if (stage == 0) {
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], &p_c[offset_rho_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], &p_c[offset_rhou0_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], &p_c[offset_rhou1_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], &p_c[offset_rhou2_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhoE], &p_c[offset_rhoE_old]);
	}



	dns_du0dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou0], &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
		sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
		(double*) d_du0dx,
		(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	dns_du1dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou1], &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
		(double*) d_du1dx,
		(double*) d_Res_rhou1,
		(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	dns_du2dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou2], &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
		(double*) d_du2dx,
		(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	dns_drhoETpdyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
		&p_c[offset_rho], &p_c[offset_rhou0], &p_c[offset_rhou1], &p_c[offset_rhou2], &p_c[offset_rhoE],
		(double*) d_du0dx, (double*) d_du1dx, (double*) d_du2dx,
		sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
		(double*) d_Res_rho,
		(double*) d_Res_rhou0,
		(double*) d_Res_rhou1,
		(double*) d_Res_rhou2,
		(double*) d_Res_rhoE);

	dns_drhoETpdx <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou0], &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
		&p_c[offset_rhou1], &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
		&p_c[offset_rhou2], &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
		&p_c[offset_rhoE], &p_l[offset_rhoE], &p_r[offset_rhoE], &p_ll[offset_rhoE], &p_rr[offset_rhoE],
		sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
		(double*) d_Res_rho,
		(double*) d_Res_rhou0,
		(double*) d_Res_rhou1,
		(double*) d_Res_rhou2,
		(double*) d_Res_rhoE);

	dns_Res_StageAdvance <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
			(double*) &p_c[offset_rho],
			&p_c[offset_rhou0], &p_c[offset_rhou1], &p_c[offset_rhou2], &p_c[offset_rhoE],
			&p_c[offset_rho_old],
			&p_c[offset_rhou0_old], &p_c[offset_rhou1_old], &p_c[offset_rhou2_old],
			&p_c[offset_rhoE_old],
			&p_c_out[offset_rho], 
			&p_c_out[offset_rhou0], &p_c_out[offset_rhou1], &p_c_out[offset_rhou2],
			&p_c_out[offset_rhoE],
			&p_c_out[offset_rho_old],
			&p_c_out[offset_rhou0_old], &p_c_out[offset_rhou1_old], &p_c_out[offset_rhou2_old],
			&p_c_out[offset_rhoE_old],
			rknew, rkold,
			(double*) d_Res_rho,
			(double*) d_Res_rhou0,
			(double*) d_Res_rhou1,
			(double*) d_Res_rhou2,
			(double*) d_Res_rhoE,
			(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	//// Copy Header
	dns_init <<<gridSize,threads_per_block,0,*stream>>>((const double*)p_c, (double*)p_c_out);
	//hipMemcpy((void*)p_c_out,(const void*)p_c,block_header_size * sizeof(double),hipMemcpyDeviceToDevice); cudaCheckError(__LINE__,__FILE__);
	// 4

}


__global__ void prepare_visual_rectilinear(double * __restrict__ p_in, double * __restrict__ p_out) {

	int32_t global_id = blockIdx.x*blockDim.x+threadIdx.x;
	// int32_t n_threads = blockDim.x*gridDim.x;

	// int32_t * p_in_i32 = (int32_t *)p_in;
	int64_t * p_in_i64 = (int64_t*)p_in;
	double * p_in_d = (double*)p_in;


	int64_t i_part=p_in_i64[0];

	// if (global_id==0) {
	// 	printf("part:%i\n",i_part);
	// }

	double * p_out_double=(double*)p_out;


	// if (global_id==0) {
	// 	p_out_i32[0]=n_mol;
	// 	p_out_i32[1]=i_part;
	// }
	if (global_id<block_ncc) {
		int32_t i_cell=global_id;
		int32_t i_x=i_part;
		int32_t i_y=i_cell/my_n_part;
		int32_t i_z=i_cell-i_y*my_n_part;

		//printf("i_part_%i_%i_%i_%i_\n",i_x,i_y,i_z,i_part);

		for (int32_t i_field=0;i_field<block_n_fields;i_field++) {
			double dtmp=p_in_d[block_header_size+i_field*block_ncc+i_cell];

			/*
			if (i_field == 0) {
			if (i_cell < 2) {
				printf(" \n i_cell = %d, dtmp: %lf", i_cell, dtmp);
			}
			if (i_cell == 0) {
				printf("\nI write my part index to %d", i_field*my_n_part*block_ncc+i_z*block_ncc+i_y*my_n_part+i_x);
				printf("\nCalculated from: i_field %d, my_n_part %d, block_ncc %d, i_z %d, i_y %d, i_x %d\n", i_field, my_n_part, block_ncc, i_z, i_y, i_x);
			}
			}
			*/
			

			
			//double dtmp = 5.0;

			p_out_double[i_field*my_n_part*block_ncc+i_z*block_ncc+i_y*my_n_part+i_x]=dtmp;
		}
	}

}

void DS::write_vtr (double * p_data, int32_t i_part, int32_t i_cycle) {
	string FileName;
	FileName.append("/direc/visual_");
	FileName+=to_string(my_n_part);
	FileName.append("_");
	FileName+=to_string(i_cycle);
	// FileName.append("/visual_");
	// FileName+=to_string(i_part);
	FileName.append(".vtr");

	/*
	printf("\np_data: ");
	
	for (int i = 0; i < 27; ++i) {
		if (i % 3 == 0) printf("\n");
		if (i % 9 == 0) printf("\n");
		printf("%f, ", p_data[i]);
		
	}
	printf("\n");
	*/
	
	

	ofstream ofs;
	ofs.open(FileName, ios::out | ios::binary);
	if (ofs) {
		int64_t append_offset=0;
		ofs << "<VTKFile type=\"RectilinearGrid\" version=\"1.0\" byte_order=\"LittleEndian\" header_type=\"UInt64\">" << endl;
		ofs << "<RectilinearGrid WholeExtent=\"" << "0 " << my_n_part-1 << " 0 " << my_n_part-1 << " 0 " << my_n_part-1 << "\">" << endl;
		ofs << "<Piece Extent=\"" << "0 " << my_n_part-1 << " 0 " << my_n_part-1 << " 0 " << my_n_part-1 << "\">" << endl;

		ofs << "<PointData Scalars=\"\" Name=\"a\">" << endl;
		ofs << "<DataArray type=\"Float64\" Name=\"rho\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou0\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou1\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou2\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhoE\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rho_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou0_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou1_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
		
		ofs << "<DataArray type=\"Float64\" Name=\"rhou2_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhoE_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		// ============================ TMP Output ============================
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp0\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp1\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp2\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp3\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp4\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		// ============================ TMP Output ============================

		ofs << "</PointData>" << endl;

		ofs << "<Coordinates>" << endl;
		ofs << "<DataArray type=\"Float64\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(double)+sizeof(int64_t);

		ofs << "</Coordinates>" << endl;

		// ofs << "\" NumberOfCells=\"0\">" << endl;
		// ofs << "<PointData Scalars=\"species\">" << endl;
		// ofs << "<DataArray type=\"Float32\" Name=\"species\" format=\"appended\" offset=\"0\" RangeMin=\"0\" RangeMax=\"6\">" << endl;
		// ofs << "</DataArray>" << endl;
		// ofs << "</PointData>" << endl;
		// ofs << "<Points>" << endl;
		// ofs << "<DataArray type=\"Float32\" Name=\"Points\" NumberOfComponents=\"3\" format=\"appended\" offset=\"";
		// ofs << n_mol*sizeof(double)+8;
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		// ofs << "</DataArray>" << endl;
		// ofs << "</Points>" << endl;
		// ofs << "<Cells>" << endl;
		// ofs << "<DataArray type=\"Int32\" Name=\"connectivity\"></DataArray>" << endl;
		// ofs << "<DataArray type=\"Int32\" Name=\"offsets\"></DataArray>" << endl;
		// ofs << "<DataArray type=\"UInt8\" Name=\"types\"></DataArray>" << endl;
		// ofs << "</Cells>" << endl;
		ofs << "</Piece>" << endl;
		ofs << "</RectilinearGrid>" << endl;
		ofs << "<AppendedData encoding=\"raw\">" << endl;
		ofs << "_";	// mark start of appended data
		ofs.close();
	}

	// write appended data
	int64_t size_append=0;

	// cell data
	for (int32_t i_field=0;i_field<block_n_fields;i_field++) {
		size_append=(my_n_part*block_ncc)*sizeof(double);
		MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
		MemToFile((int64_t*)&p_data[i_field*my_n_part*block_ncc],size_append,(char*)FileName.c_str(),0);
	}

	// coordinates - same for x,y,z
	double * x_coordinates=new double [my_n_part+1];
	for (int i=0;i<my_n_part+1;i++) {
		x_coordinates[i]=i;
	}
	size_append=(my_n_part+1)*sizeof(double);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);

	size_append=(my_n_part+1)*sizeof(double);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);

	size_append=(my_n_part+1)*sizeof(double);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);
	delete [] x_coordinates;

	// write closing tags
	ofs.open(FileName, ios::out | ios::binary | ios_base::app);
	if (ofs) {
		ofs << "</AppendedData>" << endl;
		ofs << "</VTKFile>" << endl;
		ofs.close();
	}
}

void DS::caller_output_vtk_rectilinear (double * p_in, double * p_out, hipStream_t * stream, int32_t threads_per_block, int32_t blockSize, int32_t myID, int32_t i_cycle, int32_t i_part) {

	int32_t n_blocks=block_ncc/threads_per_block;
	n_blocks++;

	prepare_visual_rectilinear <<<n_blocks,threads_per_block,0,*stream>>> (p_in,p_out);
	// int32_t * p_my_vis_i32=(int32_t*)p_my_vis;
	// float * p_my_vis_float=(float*)p_my_vis;
	if (i_part==(my_n_part-1)) {
		// last part
		double * p_my_vis_double=new double[block_n_fields*my_n_part*block_ncc];

		hipDeviceSynchronize();        cudaCheckError(__LINE__,__FILE__);

		size_t copy_size=1;
		copy_size*=block_n_fields;
		copy_size*=my_n_part;
		copy_size*=block_ncc;
		copy_size*=sizeof(double);
		// cout << copy_size << endl;
		hipError_t cer=hipMemcpy((void*)p_my_vis_double,(const void*)p_out,copy_size,hipMemcpyDeviceToHost); //cudaCheckError(__LINE__,__FILE__);
		cout << cer << endl;
		// for (int i=0;i<block_n_fields*my_n_part*block_ncc;i++) cout << p_my_vis_float[i] << endl;

		// string new_dir;
		// new_dir.append("visual/visual_");
		// new_dir+=to_string(i_cycle);

		// boost::filesystem::create_directory(new_dir.c_str());
		write_vtr(p_my_vis_double,0,i_cycle);
		delete [] p_my_vis_double;
	}
}
