#include "hip/hip_runtime.h"
// Data Streaming for Explicit Algorithms - DSEA

#include <dsea.h>
#include <stdio.h>		// printf
#include <hipcub/hipcub.hpp>
#include <climits>       // for INT_MAX
#include <fstream>
#include <boost/filesystem.hpp>

#include <cuda_pipeline.h>


using namespace :: std;

// print variadic template values
// overload
template<typename T>
void myprint(T head)
{
    std::cout << head << std::endl;
}
// base case: used when pack is non-empty
template<typename T, typename... Ts>
void myprint(T head, Ts... tail)
{
    std::cout << head << std::endl;
    myprint(tail...);
}

// Calculate the Z Y Coordinates in the grid from an thread ID
// ID: Array Index/Thread ID
// Y,Z: Grid coordinates in part, Y row, Z column
// NC: Number of Columns (NZ)

#define COORDS(ID, Y, Z, NC) \
	do { \
    Y = (ID) / (NC); \
    Z = (ID) % (NC); \
  } while(0)

// Calculate the array index from the grid coordinates
// ID: Array Index/Thread ID
// Y,Z: Grid coordinates in part, Z column, Y row
// NC: Number of Columns (NZ)
#define IDX(Y, Z, ID, NC) \ 
	do { \
    ID = (Y) * (NC) + (Z); \
  } while(0)


__device__ double dns_pDer1(double v_ll, double v_l, double v_r, double v_rr, double DK) {
	return 1./DK * (1./12. * v_ll - 2./3. * v_l + 2./3. * v_r - 1./12. * v_rr);
}

__device__ double dns_pDer2(double v_ll, double v_l, double v_c, double v_r, double v_rr, double DK) {
	return 1./(DK*DK) * (-1./12. * v_ll + 4./3. * v_l - 5./2. * v_c + 4./3. * v_r - 1./12. * v_rr);
}

__device__ double calp(double irhoE, double irho, double irhou0, double irhou1, double irhou2) {
	//return ((GAMA - 1) * (rhoE - 0.5 * rhou0 * rhou0 / rho - 0.5 * rhou1 * rhou1 / rho - 0.5 rhou2 * rhou2 / rho));
	double tmp = (irhoE - 0.5 * irhou0 * irhou0 / irho - 0.5 * irhou1 * irhou1 / irho - 0.5 * irhou2 * irhou2 / irho);
	return (GAMA - 1) * tmp;
}

__device__ double calT(double ip, double irho) {
	return MINF * MINF * GAMA * ip / irho;
}


// Calculates the global array index from 
__device__ int32_t thread_to_global_idx(int32_t problemsize, int32_t thread_idx, 
						int32_t block_size_z, int32_t block_size_y, 
						int32_t warp_size_z, int32_t warp_size_y,
						int32_t* c_i_block_out, int32_t* r_i_block_out) {
	int32_t global_idx;


	int32_t block_size = block_size_z * block_size_y;
	int32_t num_blocks = (problemsize*problemsize) / block_size;
	int32_t blocks_per_row = (problemsize) / block_size_z;

	int32_t warp_size = warp_size_z * warp_size_y;
	int32_t num_warps_per_block = block_size / warp_size;
	int32_t warps_per_row = block_size_z / warp_size_z;

	int32_t block_idx = thread_idx / block_size;
	int32_t block_row = block_idx / blocks_per_row;
	int32_t block_col = block_idx % blocks_per_row;

	int32_t idx_in_block = thread_idx % block_size;

	int32_t warp_idx = idx_in_block / warp_size;
	int32_t warp_row = warp_idx / warps_per_row;
	int32_t warp_col = warp_idx % warps_per_row;

	int32_t idx_in_warp = idx_in_block % warp_size;

	int32_t col_in_warp = idx_in_warp % warp_size_z;
	int32_t row_in_warp = idx_in_warp / warp_size_z;

	int32_t col_in_block = warp_col * warp_size_z + col_in_warp;
	int32_t row_in_block = warp_row * warp_size_y + row_in_warp;

	int32_t global_row = block_row * block_size_y + row_in_block;
	int32_t global_col = block_col * block_size_z + col_in_block;

	global_idx = global_row * problemsize + global_col;

	*c_i_block_out = col_in_block;
	*r_i_block_out = row_in_block;

	return global_idx;
	

}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Fused Kernels
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void dns_du0dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
	int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou0_c,
	/*order 1*/ double * __restrict__ irhou0_l, double * __restrict__ irhou0_r,
	/*order 2*/ double * __restrict__ irhou0_ll, double * __restrict__ irhou0_rr,
	int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
	/*order 0*/ double * __restrict__ odrhou0dx,
	/*order 0*/ double * __restrict__ odrhou0dy,
	/*order 0*/ double * __restrict__ odrhou0dz,
	/*order 0*/ double * __restrict__ odu0dx,
	/*order 0*/ double * __restrict__ odu0dy,
	/*order 0*/ double * __restrict__ odu0dz,
	/*order 0*/ double * __restrict__ Res_rhou0,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {

	__shared__ double s_rhou0_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rhou0_ll[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rhou0_l[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rhou0_r[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rhou0_rr[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	
	__shared__ double s_rho_c[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rho_ll[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rho_l[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rho_r[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	__shared__ double s_rho_rr[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo

	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// column in block, row in block
	int32_t cb, rb;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &cb, &rb);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &cb, &rb);


	if (gidx<block_ncc) {
		rb+=2;
		cb+=2;

//		s_rhou0_c[rb][cb] = irhou0_c[gidx];
//		s_rho_c[rb][cb] = irho_c[gidx];
//
//		s_rhou0_ll[rb][cb] = irhou0_ll[gidx];
//		s_rhou0_l[rb][cb] = irhou0_l[gidx];
//		s_rhou0_r[rb][cb] = irhou0_r[gidx];
//		s_rhou0_rr[rb][cb] = irhou0_rr[gidx];
//
//		s_rho_ll[rb][cb] = irho_ll[gidx];
//		s_rho_l[rb][cb] = irho_l[gidx];
//		s_rho_r[rb][cb] = irho_r[gidx];
//		s_rho_rr[rb][cb] = irho_rr[gidx];

		__pipeline_memcpy_async(&s_rhou0_c[rb][cb], &irhou0_c[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rho_c[rb][cb], &irho_c[gidx], sizeof(double));

		__pipeline_memcpy_async(&s_rhou0_ll[rb][cb], &irhou0_ll[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rhou0_l[rb][cb], &irhou0_l[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rhou0_r[rb][cb], &irhou0_r[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rhou0_rr[rb][cb], &irhou0_rr[gidx], sizeof(double));

		__pipeline_memcpy_async(&s_rho_ll[rb][cb], &irho_ll[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rho_l[rb][cb], &irho_l[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rho_r[rb][cb], &irho_r[gidx], sizeof(double));
		__pipeline_memcpy_async(&s_rho_rr[rb][cb], &irho_rr[gidx], sizeof(double));

		__pipeline_commit();


		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// get halos
		if (rb < 4) {
			IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
			__pipeline_memcpy_async(&s_rhou0_c[rb-2][cb], &irhou0_c[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_c[rb-2][cb], &irho_c[dy_ll], sizeof(double));

			__pipeline_memcpy_async(&s_rhou0_ll[rb-2][cb], &irhou0_ll[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_l[rb-2][cb], &irhou0_l[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_r[rb-2][cb], &irhou0_r[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_rr[rb-2][cb], &irhou0_rr[dy_ll], sizeof(double));

			__pipeline_memcpy_async(&s_rho_ll[rb-2][cb], &irho_ll[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_l[rb-2][cb], &irho_l[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_r[rb-2][cb], &irho_r[dy_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_rr[rb-2][cb], &irho_rr[dy_ll], sizeof(double));
		}
		if (rb >= BLOCKSIZE_Y) {
			IDX((NY+Y+2)%NY, Z, dy_rr, NZ);
			__pipeline_memcpy_async(&s_rhou0_c[rb+2][cb], &irhou0_c[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_c[rb+2][cb], &irho_c[dy_rr], sizeof(double));

			__pipeline_memcpy_async(&s_rhou0_ll[rb+2][cb], &irhou0_ll[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_l[rb+2][cb], &irhou0_l[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_r[rb+2][cb], &irhou0_r[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_rr[rb+2][cb], &irhou0_rr[dy_rr], sizeof(double));

			__pipeline_memcpy_async(&s_rho_ll[rb+2][cb], &irho_ll[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_l[rb+2][cb], &irho_l[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_r[rb+2][cb], &irho_r[dy_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_rr[rb+2][cb], &irho_rr[dy_rr], sizeof(double));
		}

		// get halos
		if (cb < 4) {
			IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
			__pipeline_memcpy_async(&s_rhou0_c[rb][cb-2], &irhou0_c[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_c[rb][cb-2], &irho_c[dz_ll], sizeof(double));

			__pipeline_memcpy_async(&s_rhou0_ll[rb][cb-2], &irhou0_ll[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_l[rb][cb-2], &irhou0_l[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_r[rb][cb-2], &irhou0_r[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_rr[rb][cb-2], &irhou0_rr[dz_ll], sizeof(double));

			__pipeline_memcpy_async(&s_rho_ll[rb][cb-2], &irho_ll[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_l[rb][cb-2], &irho_l[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_r[rb][cb-2], &irho_r[dz_ll], sizeof(double));
			__pipeline_memcpy_async(&s_rho_rr[rb][cb-2], &irho_rr[dz_ll], sizeof(double));
		}
		if (cb >= BLOCKSIZE_Z) {
			IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);
			__pipeline_memcpy_async(&s_rhou0_c[rb][cb+2], &irhou0_c[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_c[rb][cb+2], &irho_c[dz_rr], sizeof(double));

			__pipeline_memcpy_async(&s_rhou0_ll[rb][cb+2], &irhou0_ll[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_l[rb][cb+2], &irhou0_l[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_r[rb][cb+2], &irhou0_r[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rhou0_rr[rb][cb+2], &irhou0_rr[dz_rr], sizeof(double));

			__pipeline_memcpy_async(&s_rho_ll[rb][cb+2], &irho_ll[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_l[rb][cb+2], &irho_l[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_r[rb][cb+2], &irho_r[dz_rr], sizeof(double));
			__pipeline_memcpy_async(&s_rho_rr[rb][cb+2], &irho_rr[dz_rr], sizeof(double));
		}

		__pipeline_wait_prior(0);

		double tmp0 = 0;

		double u0_dy_ll, u0_dy_l, u0_dy_c, u0_dy_r, u0_dy_rr;
		u0_dy_ll = s_rhou0_c[rb-2][cb] / s_rho_c[rb-2][cb];
		u0_dy_l = s_rhou0_c[rb-1][cb] / s_rho_c[rb-1][cb];
		u0_dy_c = s_rhou0_c[rb][cb] / s_rho_c[rb][cb];
		u0_dy_r = s_rhou0_c[rb+1][cb] / s_rho_c[rb+1][cb];
		u0_dy_rr = s_rhou0_c[rb+2][cb] / s_rho_c[rb+2][cb];

		odrhou0dy[gidx] = dns_pDer1(s_rhou0_c[rb-2][cb], s_rhou0_c[rb-1][cb], s_rhou0_c[rb+1][cb], s_rhou0_c[rb+2][cb], DY);
		odu0dy[gidx] = dns_pDer1(u0_dy_ll, u0_dy_l, u0_dy_r, u0_dy_rr, DY);
		tmp0 += dns_pDer2(u0_dy_ll, u0_dy_l, u0_dy_c, u0_dy_r, u0_dy_rr, DY);

		
		double u0_dz_ll, u0_dz_l, u0_dz_c, u0_dz_r, u0_dz_rr;
		u0_dz_ll = s_rhou0_c[rb][cb-2] / s_rho_c[rb][cb-2];
		u0_dz_l = s_rhou0_c[rb][cb-1] / s_rho_c[rb][cb-1];
		u0_dz_c = s_rhou0_c[rb][cb] / s_rho_c[rb][cb];
		u0_dz_r = s_rhou0_c[rb][cb+1] / s_rho_c[rb][cb+1];
		u0_dz_rr = s_rhou0_c[rb][cb+2] / s_rho_c[rb][cb+2];

		odrhou0dz[gidx] = dns_pDer1(s_rhou0_c[rb][cb-2], s_rhou0_c[rb][cb-1], s_rhou0_c[rb][cb+1], s_rhou0_c[rb][cb+2], DZ);
		odu0dz[gidx] = dns_pDer1(u0_dz_ll, u0_dz_l, u0_dz_r, u0_dz_rr, DZ);
		tmp0 += dns_pDer2(u0_dz_ll, u0_dz_l, u0_dz_c, u0_dz_r, u0_dz_rr, DZ);


		double u0_dx_ll, u0_dx_l, u0_dx_c, u0_dx_r, u0_dx_rr;
		u0_dx_ll = s_rhou0_ll[rb][cb] / s_rho_ll[rb][cb];
		u0_dx_l =s_rhou0_l[rb][cb] / s_rho_l[rb][cb];
		u0_dx_c = s_rhou0_c[rb][cb] / s_rho_c[rb][cb];
		u0_dx_r = s_rhou0_r[rb][cb] / s_rho_r[rb][cb];
		u0_dx_rr = s_rhou0_rr[rb][cb] / s_rho_rr[rb][cb];

		odrhou0dx[gidx] = dns_pDer1(sy_bc_ll *  s_rhou0_ll[rb][cb], sy_bc_l * s_rhou0_l[rb][cb], sy_bc_r * s_rhou0_r[rb][cb], sy_bc_rr * s_rhou0_rr[rb][cb], DX);
		odu0dx[gidx] = dns_pDer1(sy_bc_ll *  u0_dx_ll, sy_bc_l * u0_dx_l, sy_bc_r * u0_dx_r, sy_bc_rr * u0_dx_rr, DX);
		tmp0 += 4./3. * dns_pDer2(sy_bc_ll * u0_dx_ll, sy_bc_l * u0_dx_l, u0_dx_c, sy_bc_r * u0_dx_r, sy_bc_rr * u0_dx_rr, DX);
	
		Res_rhou0[gidx] = -0.5 * dns_pDer1(sy_bc_ll * s_rhou0_ll[rb][cb] * sy_bc_ll * u0_dx_ll,
													sy_bc_l * s_rhou0_l[rb][cb] * sy_bc_l * u0_dx_l, 
													sy_bc_r * s_rhou0_r[rb][cb] * sy_bc_r * u0_dx_r, 
													sy_bc_rr * s_rhou0_rr[rb][cb] * sy_bc_rr * u0_dx_rr, DX);

		tmp_du0d2xi[gidx] = tmp0;

		double u0_dy_ll_dx_ll = s_rhou0_ll[rb-2][cb] / s_rho_ll[rb-2][cb];
		double u0_dy_l_dx_ll = s_rhou0_ll[rb-1][cb] / s_rho_ll[rb-1][cb];
		double u0_dy_r_dx_ll = s_rhou0_ll[rb+1][cb] / s_rho_ll[rb+1][cb];
		double u0_dy_rr_dx_ll = s_rhou0_ll[rb+2][cb] / s_rho_ll[rb+2][cb];

		double du0dy_dx_ll = dns_pDer1(u0_dy_ll_dx_ll, u0_dy_l_dx_ll, u0_dy_r_dx_ll, u0_dy_rr_dx_ll, DY);

		double u0_dz_ll_dx_ll =s_rhou0_ll[rb][cb-2] / s_rho_ll[rb][cb-2];
		double u0_dz_l_dx_ll = s_rhou0_ll[rb][cb-1] / s_rho_ll[rb][cb-1];
		double u0_dz_r_dx_ll = s_rhou0_ll[rb][cb+1] / s_rho_ll[rb][cb+1];
		double u0_dz_rr_dx_ll = s_rhou0_ll[rb][cb+2] / s_rho_ll[rb][cb+2];

		double du0dz_dx_ll = dns_pDer1(u0_dz_ll_dx_ll, u0_dz_l_dx_ll, u0_dz_r_dx_ll, u0_dz_rr_dx_ll, DZ);

		double u0_dy_ll_dx_l = s_rhou0_l[rb-2][cb] / s_rho_l[rb-2][cb];
		double u0_dy_l_dx_l = s_rhou0_l[rb-1][cb] / s_rho_l[rb-1][cb];
		double u0_dy_r_dx_l = s_rhou0_l[rb+1][cb] / s_rho_l[rb+1][cb];
		double u0_dy_rr_dx_l = s_rhou0_l[rb+2][cb] / s_rho_l[rb+2][cb];

		double du0dy_dx_l = dns_pDer1(u0_dy_ll_dx_l, u0_dy_l_dx_l, u0_dy_r_dx_l, u0_dy_rr_dx_l, DY);

		double u0_dz_ll_dx_l =s_rhou0_l[rb][cb-2] / s_rho_l[rb][cb-2];
		double u0_dz_l_dx_l = s_rhou0_l[rb][cb-1] / s_rho_l[rb][cb-1];
		double u0_dz_r_dx_l = s_rhou0_l[rb][cb+1] / s_rho_l[rb][cb+1];
		double u0_dz_rr_dx_l = s_rhou0_l[rb][cb+2] / s_rho_l[rb][cb+2];

		double du0dz_dx_l = dns_pDer1(u0_dz_ll_dx_l, u0_dz_l_dx_l, u0_dz_r_dx_l, u0_dz_rr_dx_l, DZ);

		double u0_dy_ll_dx_r = s_rhou0_r[rb-2][cb] / s_rho_r[rb-2][cb];
		double u0_dy_l_dx_r = s_rhou0_r[rb-1][cb] / s_rho_r[rb-1][cb];
		double u0_dy_r_dx_r = s_rhou0_r[rb+1][cb] / s_rho_r[rb+1][cb];
		double u0_dy_rr_dx_r = s_rhou0_r[rb+2][cb] / s_rho_r[rb+2][cb];

		double du0dy_dx_r = dns_pDer1(u0_dy_ll_dx_r, u0_dy_l_dx_r, u0_dy_r_dx_r, u0_dy_rr_dx_r, DY);

		double u0_dz_ll_dx_r =s_rhou0_r[rb][cb-2] / s_rho_r[rb][cb-2];
		double u0_dz_l_dx_r = s_rhou0_r[rb][cb-1] / s_rho_r[rb][cb-1];
		double u0_dz_r_dx_r = s_rhou0_r[rb][cb+1] / s_rho_r[rb][cb+1];
		double u0_dz_rr_dx_r = s_rhou0_r[rb][cb+2] / s_rho_r[rb][cb+2];

		double du0dz_dx_r = dns_pDer1(u0_dz_ll_dx_r, u0_dz_l_dx_r, u0_dz_r_dx_r, u0_dz_rr_dx_r, DZ);

		double u0_dy_ll_dx_rr = s_rhou0_rr[rb-2][cb] / s_rho_rr[rb-2][cb];
		double u0_dy_l_dx_rr = s_rhou0_rr[rb-1][cb] / s_rho_rr[rb-1][cb];
		double u0_dy_r_dx_rr = s_rhou0_rr[rb+1][cb] / s_rho_rr[rb+1][cb];
		double u0_dy_rr_dx_rr = s_rhou0_rr[rb+2][cb] / s_rho_rr[rb+2][cb];

		double du0dy_dx_rr = dns_pDer1(u0_dy_ll_dx_rr, u0_dy_l_dx_rr, u0_dy_r_dx_rr, u0_dy_rr_dx_rr, DY);

		double u0_dz_ll_dx_rr =s_rhou0_rr[rb][cb-2] / s_rho_rr[rb][cb-2];
		double u0_dz_l_dx_rr = s_rhou0_rr[rb][cb-1] / s_rho_rr[rb][cb-1];
		double u0_dz_r_dx_rr = s_rhou0_rr[rb][cb+1] / s_rho_rr[rb][cb+1];
		double u0_dz_rr_dx_rr = s_rhou0_rr[rb][cb+2] / s_rho_rr[rb][cb+2];

		double du0dz_dx_rr = dns_pDer1(u0_dz_ll_dx_rr, u0_dz_l_dx_rr, u0_dz_r_dx_rr, u0_dz_rr_dx_rr, DZ);

		tmp_du1d2xi[gidx] = 1./3. * dns_pDer1(sy_bc_ll *  du0dy_dx_ll, sy_bc_l * du0dy_dx_l, sy_bc_r * du0dy_dx_r, sy_bc_rr * du0dy_dx_rr, DX);
		tmp_du2d2xi[gidx] = 1./3. * dns_pDer1(sy_bc_ll *  du0dz_dx_ll, sy_bc_l * du0dz_dx_l, sy_bc_r * du0dz_dx_r, sy_bc_rr * du0dz_dx_rr, DX);

	}
}

__global__ void dns_du1dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
	int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou1_c,
	/*order 1*/ double * __restrict__ irhou1_l, double * __restrict__ irhou1_r,
	/*order 2*/ double * __restrict__ irhou1_ll, double * __restrict__ irhou1_rr,
	/*order 0*/ double * __restrict__ odrhou1dx,
	/*order 0*/ double * __restrict__ odrhou1dy,
	/*order 0*/ double * __restrict__ odrhou1dz,
	/*order 0*/ double * __restrict__ odu1dx,
	/*order 0*/ double * __restrict__ odu1dy,
	/*order 0*/ double * __restrict__ odu1dz,
  /*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		double tmp0 = 0;


		double rhou1_dy_ll, rhou1_dy_l, rhou1_dy_c, rhou1_dy_r, rhou1_dy_rr;
		rhou1_dy_ll = irhou1_c[dy_ll];
		rhou1_dy_l = irhou1_c[dy_l];
		rhou1_dy_c = irhou1_c[gidx];
		rhou1_dy_r = irhou1_c[dy_r];
		rhou1_dy_rr = irhou1_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u1_dy_ll, u1_dy_l, u1_dy_c, u1_dy_r, u1_dy_rr;
		u1_dy_ll = rhou1_dy_ll / rho_dy_ll;
		u1_dy_l = rhou1_dy_l / rho_dy_l;
		u1_dy_c = rhou1_dy_c / rho_dy_c;
		u1_dy_r = rhou1_dy_r / rho_dy_r;
		u1_dy_rr = rhou1_dy_rr / rho_dy_rr;

		odrhou1dy[gidx] = dns_pDer1(rhou1_dy_ll, rhou1_dy_l, rhou1_dy_r, rhou1_dy_rr, DY);
		odu1dy[gidx] = dns_pDer1(u1_dy_ll, u1_dy_l, u1_dy_r, u1_dy_rr, DY);
		tmp0 += 4./3. * dns_pDer2(u1_dy_ll, u1_dy_l, u1_dy_c, u1_dy_r, u1_dy_rr, DY);
		Res_rhou1[gidx] = -0.5 * dns_pDer1(rhou1_dy_ll * u1_dy_ll, rhou1_dy_l * u1_dy_l, rhou1_dy_r * u1_dy_r, rhou1_dy_rr * u1_dy_rr, DY);


		double rhou1_dz_ll, rhou1_dz_l, rhou1_dz_c, rhou1_dz_r, rhou1_dz_rr;
		rhou1_dz_ll = irhou1_c[dz_ll];
		rhou1_dz_l = irhou1_c[dz_l];
		rhou1_dz_c = irhou1_c[gidx];
		rhou1_dz_r = irhou1_c[dz_r];
		rhou1_dz_rr = irhou1_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u1_dz_ll, u1_dz_l, u1_dz_c, u1_dz_r, u1_dz_rr;
		u1_dz_ll = rhou1_dz_ll / rho_dz_ll;
		u1_dz_l = rhou1_dz_l / rho_dz_l;
		u1_dz_c = rhou1_dz_c / rho_dz_c;
		u1_dz_r = rhou1_dz_r / rho_dz_r;
		u1_dz_rr = rhou1_dz_rr / rho_dz_rr;

		odrhou1dz[gidx] = dns_pDer1(rhou1_dz_ll, rhou1_dz_l, rhou1_dz_r, rhou1_dz_rr, DZ);
		odu1dz[gidx] = dns_pDer1(u1_dz_ll, u1_dz_l, u1_dz_r, u1_dz_rr, DZ);
		tmp0 += dns_pDer2(u1_dz_ll, u1_dz_l, u1_dz_c, u1_dz_r, u1_dz_rr, DZ);


		double rhou1_dx_ll, rhou1_dx_l, rhou1_dx_c, rhou1_dx_r, rhou1_dx_rr;
		rhou1_dx_ll = irhou1_ll[gidx];
		rhou1_dx_l = irhou1_l[gidx];
		rhou1_dx_c = irhou1_c[gidx];
		rhou1_dx_r = irhou1_r[gidx];
		rhou1_dx_rr = irhou1_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u1_dx_ll, u1_dx_l, u1_dx_c, u1_dx_r, u1_dx_rr;
		u1_dx_ll = rhou1_dx_ll / rho_dx_ll;
		u1_dx_l = rhou1_dx_l / rho_dx_l;
		u1_dx_c = rhou1_dx_c / rho_dx_c;
		u1_dx_r = rhou1_dx_r / rho_dx_r;
		u1_dx_rr = rhou1_dx_rr / rho_dx_rr;

		odrhou1dx[gidx] = dns_pDer1(rhou1_dx_ll, rhou1_dx_l, rhou1_dx_r, rhou1_dx_rr, DX);
		odu1dx[gidx] = dns_pDer1(u1_dx_ll, u1_dx_l, u1_dx_r, u1_dx_rr, DX);
		tmp0 += dns_pDer2(u1_dx_ll, u1_dx_l, u1_dx_c, u1_dx_r, u1_dx_rr, DX);

		tmp_du1d2xi[gidx] += tmp0;


		double u1_dx_ll_dy_ll = irhou1_ll[dy_ll] / irho_ll[dy_ll];
		double u1_dx_l_dy_ll = irhou1_l[dy_ll] / irho_l[dy_ll];
		double u1_dx_r_dy_ll = irhou1_r[dy_ll] / irho_r[dy_ll];
		double u1_dx_rr_dy_ll = irhou1_rr[dy_ll] / irho_rr[dy_ll];

		double du1dx_dy_ll = dns_pDer1(u1_dx_ll_dy_ll, u1_dx_l_dy_ll, u1_dx_r_dy_ll, u1_dx_rr_dy_ll, DX);

		double u1_dx_ll_dy_l = irhou1_ll[dy_l] / irho_ll[dy_l];
		double u1_dx_l_dy_l = irhou1_l[dy_l] / irho_l[dy_l];
		double u1_dx_r_dy_l = irhou1_r[dy_l] / irho_r[dy_l];
		double u1_dx_rr_dy_l = irhou1_rr[dy_l] / irho_rr[dy_l];

		double du1dx_dy_l = dns_pDer1(u1_dx_ll_dy_l, u1_dx_l_dy_l, u1_dx_r_dy_l, u1_dx_rr_dy_l, DX);

		double u1_dx_ll_dy_r = irhou1_ll[dy_r] / irho_ll[dy_r];
		double u1_dx_l_dy_r = irhou1_l[dy_r] / irho_l[dy_r];
		double u1_dx_r_dy_r = irhou1_r[dy_r] / irho_r[dy_r];
		double u1_dx_rr_dy_r = irhou1_rr[dy_r] / irho_rr[dy_r];

		double du1dx_dy_r = dns_pDer1(u1_dx_ll_dy_r, u1_dx_l_dy_r, u1_dx_r_dy_r, u1_dx_rr_dy_r, DX);

		double u1_dx_ll_dy_rr = irhou1_ll[dy_rr] / irho_ll[dy_rr];
		double u1_dx_l_dy_rr = irhou1_l[dy_rr] / irho_l[dy_rr];
		double u1_dx_r_dy_rr = irhou1_r[dy_rr] / irho_r[dy_rr];
		double u1_dx_rr_dy_rr = irhou1_rr[dy_rr] / irho_rr[dy_rr];

		double du1dx_dy_rr = dns_pDer1(u1_dx_ll_dy_rr, u1_dx_l_dy_rr, u1_dx_r_dy_rr, u1_dx_rr_dy_rr, DX);

		tmp_du0d2xi[gidx] += 1./3. * dns_pDer1(du1dx_dy_ll, du1dx_dy_l, du1dx_dy_r, du1dx_dy_rr, DY);

		// Calculate idx with periodic boundary condition
		int32_t dy_ll_dz_ll, dy_ll_dz_l, dy_ll_dz_r, dy_ll_dz_rr;
		IDX((NY+Y-2)%NY, (NZ+Z-2)%NZ, dy_ll_dz_ll, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z-1)%NZ, dy_ll_dz_l, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+1)%NZ, dy_ll_dz_r, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+2)%NZ, dy_ll_dz_rr, NZ);

		double u1_dz_ll_dy_ll = irhou1_c[dy_ll_dz_ll] / irho_c[dy_ll_dz_ll];
		double u1_dz_l_dy_ll = irhou1_c[dy_ll_dz_l] / irho_c[dy_ll_dz_l];
		double u1_dz_r_dy_ll = irhou1_c[dy_ll_dz_r] / irho_c[dy_ll_dz_r];
		double u1_dz_rr_dy_ll = irhou1_c[dy_ll_dz_rr] / irho_c[dy_ll_dz_rr];

		double du1dz_dy_ll = dns_pDer1(u1_dz_ll_dy_ll, u1_dz_l_dy_ll, u1_dz_r_dy_ll, u1_dz_rr_dy_ll, DZ);


		int32_t dy_l_dz_ll, dy_l_dz_l, dy_l_dz_r, dy_l_dz_rr;
		IDX((NY+Y-1)%NY, (NZ+Z-2)%NZ, dy_l_dz_ll, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z-1)%NZ, dy_l_dz_l, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+1)%NZ, dy_l_dz_r, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+2)%NZ, dy_l_dz_rr, NZ);

		double u1_dz_ll_dy_l = irhou1_c[dy_l_dz_ll] / irho_c[dy_l_dz_ll];
		double u1_dz_l_dy_l = irhou1_c[dy_l_dz_l] / irho_c[dy_l_dz_l];
		double u1_dz_r_dy_l = irhou1_c[dy_l_dz_r] / irho_c[dy_l_dz_r];
		double u1_dz_rr_dy_l = irhou1_c[dy_l_dz_rr] / irho_c[dy_l_dz_rr];

		double du1dz_dy_l = dns_pDer1(u1_dz_ll_dy_l, u1_dz_l_dy_l, u1_dz_r_dy_l, u1_dz_rr_dy_l, DZ);


		int32_t dy_r_dz_ll, dy_r_dz_l, dy_r_dz_r, dy_r_dz_rr;
		IDX((NY+Y+1)%NY, (NZ+Z-2)%NZ, dy_r_dz_ll, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z-1)%NZ, dy_r_dz_l, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+1)%NZ, dy_r_dz_r, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+2)%NZ, dy_r_dz_rr, NZ);

		double u1_dz_ll_dy_r = irhou1_c[dy_r_dz_ll] / irho_c[dy_r_dz_ll];
		double u1_dz_l_dy_r = irhou1_c[dy_r_dz_l] / irho_c[dy_r_dz_l];
		double u1_dz_r_dy_r = irhou1_c[dy_r_dz_r] / irho_c[dy_r_dz_r];
		double u1_dz_rr_dy_r = irhou1_c[dy_r_dz_rr] / irho_c[dy_r_dz_rr];

		double du1dz_dy_r = dns_pDer1(u1_dz_ll_dy_r, u1_dz_l_dy_r, u1_dz_r_dy_r, u1_dz_rr_dy_r, DZ);


		int32_t dy_rr_dz_ll, dy_rr_dz_l, dy_rr_dz_r, dy_rr_dz_rr;
		IDX((NY+Y+2)%NY, (NZ+Z-2)%NZ, dy_rr_dz_ll, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z-1)%NZ, dy_rr_dz_l, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+1)%NZ, dy_rr_dz_r, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+2)%NZ, dy_rr_dz_rr, NZ);

		double u1_dz_ll_dy_rr = irhou1_c[dy_rr_dz_ll] / irho_c[dy_rr_dz_ll];
		double u1_dz_l_dy_rr = irhou1_c[dy_rr_dz_l] / irho_c[dy_rr_dz_l];
		double u1_dz_r_dy_rr = irhou1_c[dy_rr_dz_r] / irho_c[dy_rr_dz_r];
		double u1_dz_rr_dy_rr = irhou1_c[dy_rr_dz_rr] / irho_c[dy_rr_dz_rr];

		double du1dz_dy_rr = dns_pDer1(u1_dz_ll_dy_rr, u1_dz_l_dy_rr, u1_dz_r_dy_rr, u1_dz_rr_dy_rr, DZ);

		tmp_du2d2xi[gidx] += 1./3. * dns_pDer1(du1dz_dy_ll,  du1dz_dy_l, du1dz_dy_r, du1dz_dy_rr, DY);
	}
}

__global__ void dns_du2dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
	int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou2_c,
	/*order 1*/ double * __restrict__ irhou2_l, double * __restrict__ irhou2_r,
	/*order 2*/ double * __restrict__ irhou2_ll, double * __restrict__ irhou2_rr,
	/*order 0*/ double * __restrict__ odrhou2dx,
	/*order 0*/ double * __restrict__ odrhou2dy,
	/*order 0*/ double * __restrict__ odrhou2dz,
	/*order 0*/ double * __restrict__ odu2dx,
	/*order 0*/ double * __restrict__ odu2dy,
	/*order 0*/ double * __restrict__ odu2dz,
	/*order 0*/ double * __restrict__ Res_rhou2,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		double tmp0 = 0;

		double rhou2_dy_ll, rhou2_dy_l, rhou2_dy_c, rhou2_dy_r, rhou2_dy_rr;
		rhou2_dy_ll = irhou2_c[dy_ll];
		rhou2_dy_l = irhou2_c[dy_l];
		rhou2_dy_c = irhou2_c[gidx];
		rhou2_dy_r = irhou2_c[dy_r];
		rhou2_dy_rr = irhou2_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u2_dy_ll, u2_dy_l, u2_dy_c, u2_dy_r, u2_dy_rr;
		u2_dy_ll = rhou2_dy_ll / rho_dy_ll;
		u2_dy_l = rhou2_dy_l / rho_dy_l;
		u2_dy_c = rhou2_dy_c / rho_dy_c;
		u2_dy_r = rhou2_dy_r / rho_dy_r;
		u2_dy_rr = rhou2_dy_rr / rho_dy_rr;

		odrhou2dy[gidx] = dns_pDer1(rhou2_dy_ll, rhou2_dy_l, rhou2_dy_r, rhou2_dy_rr, DY);
		odu2dy[gidx] = dns_pDer1(u2_dy_ll, u2_dy_l, u2_dy_r, u2_dy_rr, DY);
		tmp0 += dns_pDer2(u2_dy_ll, u2_dy_l, u2_dy_c, u2_dy_r, u2_dy_rr, DY);

		double rhou2_dz_ll, rhou2_dz_l, rhou2_dz_c, rhou2_dz_r, rhou2_dz_rr;
		rhou2_dz_ll = irhou2_c[dz_ll];
		rhou2_dz_l = irhou2_c[dz_l];
		rhou2_dz_c = irhou2_c[gidx];
		rhou2_dz_r = irhou2_c[dz_r];
		rhou2_dz_rr = irhou2_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u2_dz_ll, u2_dz_l, u2_dz_c, u2_dz_r, u2_dz_rr;
		u2_dz_ll = rhou2_dz_ll / rho_dz_ll;
		u2_dz_l = rhou2_dz_l / rho_dz_l;
		u2_dz_c = rhou2_dz_c / rho_dz_c;
		u2_dz_r = rhou2_dz_r / rho_dz_r;
		u2_dz_rr = rhou2_dz_rr / rho_dz_rr;

		odrhou2dz[gidx] = dns_pDer1(rhou2_dz_ll, rhou2_dz_l, rhou2_dz_r, rhou2_dz_rr, DZ);
		odu2dz[gidx] = dns_pDer1(u2_dz_ll, u2_dz_l, u2_dz_r, u2_dz_rr, DZ);
		tmp0 += 4./3. * dns_pDer2(u2_dz_ll, u2_dz_l, u2_dz_c, u2_dz_r, u2_dz_rr, DZ);
		Res_rhou2[gidx] = -0.5 * dns_pDer1(rhou2_dz_ll * u2_dz_ll, rhou2_dz_l * u2_dz_l, rhou2_dz_r * u2_dz_r, rhou2_dz_rr * u2_dz_rr, DZ);


		double rhou2_dx_ll, rhou2_dx_l, rhou2_dx_c, rhou2_dx_r, rhou2_dx_rr;
		rhou2_dx_ll = irhou2_ll[gidx];
		rhou2_dx_l = irhou2_l[gidx];
		rhou2_dx_c = irhou2_c[gidx];
		rhou2_dx_r = irhou2_r[gidx];
		rhou2_dx_rr = irhou2_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u2_dx_ll, u2_dx_l, u2_dx_c, u2_dx_r, u2_dx_rr;
		u2_dx_ll = rhou2_dx_ll / rho_dx_ll;
		u2_dx_l = rhou2_dx_l / rho_dx_l;
		u2_dx_c = rhou2_dx_c / rho_dx_c;
		u2_dx_r = rhou2_dx_r / rho_dx_r;
		u2_dx_rr = rhou2_dx_rr / rho_dx_rr;

		odrhou2dx[gidx] = dns_pDer1(rhou2_dx_ll, rhou2_dx_l, rhou2_dx_r, rhou2_dx_rr, DX);
		odu2dx[gidx]= dns_pDer1(u2_dx_ll, u2_dx_l, u2_dx_r, u2_dx_rr, DX);
		tmp0 += dns_pDer2(u2_dx_ll, u2_dx_l, u2_dx_c, u2_dx_r, u2_dx_rr, DX);

		tmp_du2d2xi[gidx] += tmp0;



		double u2_dx_ll_dz_ll = irhou2_ll[dz_ll] / irho_ll[dz_ll];
		double u2_dx_l_dz_ll = irhou2_l[dz_ll] / irho_l[dz_ll];
		double u2_dx_r_dz_ll = irhou2_r[dz_ll] / irho_r[dz_ll];
		double u2_dx_rr_dz_ll = irhou2_rr[dz_ll] / irho_rr[dz_ll];

		double du2dx_dz_ll = dns_pDer1(u2_dx_ll_dz_ll, u2_dx_l_dz_ll, u2_dx_r_dz_ll, u2_dx_rr_dz_ll, DX);

		double u2_dx_ll_dz_l = irhou2_ll[dz_l] / irho_ll[dz_l];
		double u2_dx_l_dz_l = irhou2_l[dz_l] / irho_l[dz_l];
		double u2_dx_r_dz_l = irhou2_r[dz_l] / irho_r[dz_l];
		double u2_dx_rr_dz_l = irhou2_rr[dz_l] / irho_rr[dz_l];

		double du2dx_dz_l = dns_pDer1(u2_dx_ll_dz_l, u2_dx_l_dz_l, u2_dx_r_dz_l, u2_dx_rr_dz_l, DX);

		double u2_dx_ll_dz_r = irhou2_ll[dz_r] / irho_ll[dz_r];
		double u2_dx_l_dz_r = irhou2_l[dz_r] / irho_l[dz_r];
		double u2_dx_r_dz_r = irhou2_r[dz_r] / irho_r[dz_r];
		double u2_dx_rr_dz_r = irhou2_rr[dz_r] / irho_rr[dz_r];

		double du2dx_dz_r = dns_pDer1(u2_dx_ll_dz_r, u2_dx_l_dz_r, u2_dx_r_dz_r, u2_dx_rr_dz_r, DX);

		double u2_dx_ll_dz_rr = irhou2_ll[dz_rr] / irho_ll[dz_rr];
		double u2_dx_l_dz_rr = irhou2_l[dz_rr] / irho_l[dz_rr];
		double u2_dx_r_dz_rr = irhou2_r[dz_rr] / irho_r[dz_rr];
		double u2_dx_rr_dz_rr = irhou2_rr[dz_rr] / irho_rr[dz_rr];

		double du2dx_dz_rr = dns_pDer1(u2_dx_ll_dz_rr, u2_dx_l_dz_rr, u2_dx_r_dz_rr, u2_dx_rr_dz_rr, DX);

		tmp_du0d2xi[gidx] += 1./3. * dns_pDer1(du2dx_dz_ll, du2dx_dz_l, du2dx_dz_r, du2dx_dz_rr, DZ);


		// Calculate idx with periodic boundary condition
		int32_t dy_ll_dz_ll, dy_ll_dz_l, dy_ll_dz_r, dy_ll_dz_rr;
		IDX((NY+Y-2)%NY, (NZ+Z-2)%NZ, dy_ll_dz_ll, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z-1)%NZ, dy_ll_dz_l, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+1)%NZ, dy_ll_dz_r, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+2)%NZ, dy_ll_dz_rr, NZ);

		double u2_dz_ll_dy_ll = irhou2_c[dy_ll_dz_ll] / irho_c[dy_ll_dz_ll];
		double u2_dz_l_dy_ll = irhou2_c[dy_ll_dz_l] / irho_c[dy_ll_dz_l];
		double u2_dz_r_dy_ll = irhou2_c[dy_ll_dz_r] / irho_c[dy_ll_dz_r];
		double u2_dz_rr_dy_ll = irhou2_c[dy_ll_dz_rr] / irho_c[dy_ll_dz_rr];

		double du2dz_dy_ll = dns_pDer1(u2_dz_ll_dy_ll, u2_dz_l_dy_ll, u2_dz_r_dy_ll, u2_dz_rr_dy_ll, DZ);


		int32_t dy_l_dz_ll, dy_l_dz_l, dy_l_dz_r, dy_l_dz_rr;
		IDX((NY+Y-1)%NY, (NZ+Z-2)%NZ, dy_l_dz_ll, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z-1)%NZ, dy_l_dz_l, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+1)%NZ, dy_l_dz_r, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+2)%NZ, dy_l_dz_rr, NZ);

		double u2_dz_ll_dy_l = irhou2_c[dy_l_dz_ll] / irho_c[dy_l_dz_ll];
		double u2_dz_l_dy_l = irhou2_c[dy_l_dz_l] / irho_c[dy_l_dz_l];
		double u2_dz_r_dy_l = irhou2_c[dy_l_dz_r] / irho_c[dy_l_dz_r];
		double u2_dz_rr_dy_l = irhou2_c[dy_l_dz_rr] / irho_c[dy_l_dz_rr];

		double du2dz_dy_l = dns_pDer1(u2_dz_ll_dy_l, u2_dz_l_dy_l, u2_dz_r_dy_l, u2_dz_rr_dy_l, DZ);


		int32_t dy_r_dz_ll, dy_r_dz_l, dy_r_dz_r, dy_r_dz_rr;
		IDX((NY+Y+1)%NY, (NZ+Z-2)%NZ, dy_r_dz_ll, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z-1)%NZ, dy_r_dz_l, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+1)%NZ, dy_r_dz_r, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+2)%NZ, dy_r_dz_rr, NZ);

		double u2_dz_ll_dy_r = irhou2_c[dy_r_dz_ll] / irho_c[dy_r_dz_ll];
		double u2_dz_l_dy_r = irhou2_c[dy_r_dz_l] / irho_c[dy_r_dz_l];
		double u2_dz_r_dy_r = irhou2_c[dy_r_dz_r] / irho_c[dy_r_dz_r];
		double u2_dz_rr_dy_r = irhou2_c[dy_r_dz_rr] / irho_c[dy_r_dz_rr];

		double du2dz_dy_r = dns_pDer1(u2_dz_ll_dy_r, u2_dz_l_dy_r, u2_dz_r_dy_r, u2_dz_rr_dy_r, DZ);


		int32_t dy_rr_dz_ll, dy_rr_dz_l, dy_rr_dz_r, dy_rr_dz_rr;
		IDX((NY+Y+2)%NY, (NZ+Z-2)%NZ, dy_rr_dz_ll, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z-1)%NZ, dy_rr_dz_l, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+1)%NZ, dy_rr_dz_r, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+2)%NZ, dy_rr_dz_rr, NZ);

		double u2_dz_ll_dy_rr = irhou2_c[dy_rr_dz_ll] / irho_c[dy_rr_dz_ll];
		double u2_dz_l_dy_rr = irhou2_c[dy_rr_dz_l] / irho_c[dy_rr_dz_l];
		double u2_dz_r_dy_rr = irhou2_c[dy_rr_dz_r] / irho_c[dy_rr_dz_r];
		double u2_dz_rr_dy_rr = irhou2_c[dy_rr_dz_rr] / irho_c[dy_rr_dz_rr];

		double du2dz_dy_rr = dns_pDer1(u2_dz_ll_dy_rr, u2_dz_l_dy_rr, u2_dz_r_dy_rr, u2_dz_rr_dy_rr, DZ);

		tmp_du1d2xi[gidx] += 1./3. * dns_pDer1(du2dz_dy_ll, du2dz_dy_l, du2dz_dy_r, du2dz_dy_rr, DY);
	}
}


__global__ void dns_drhoETpdxyz(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ irho_c,
	/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
	/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
	/*order 0*/ double * __restrict__ irhou0_c,
	/*order 1*/ double * __restrict__ irhou0_l, double * __restrict__ irhou0_r,
	/*order 2*/ double * __restrict__ irhou0_ll, double * __restrict__ irhou0_rr,
	/*order 0*/ double * __restrict__ irhou1_c,
	/*order 1*/ double * __restrict__ irhou1_l, double * __restrict__ irhou1_r,
	/*order 2*/ double * __restrict__ irhou1_ll, double * __restrict__ irhou1_rr,
	/*order 0*/ double * __restrict__ irhou2_c,
	/*order 1*/ double * __restrict__ irhou2_l, double * __restrict__ irhou2_r,
	/*order 2*/ double * __restrict__ irhou2_ll, double * __restrict__ irhou2_rr,
	/*order 0*/ double * __restrict__ irhoE_c,
	/*order 1*/ double * __restrict__ irhoE_l, double * __restrict__ irhoE_r,
	/*order 2*/ double * __restrict__ irhoE_ll, double * __restrict__ irhoE_rr,
	int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
	/*order 0*/ double * __restrict__ Res_rho,
	/*order 0*/ double * __restrict__ Res_rhou0,
	/*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ Res_rhou2,
	/*order 0*/ double * __restrict__ Res_rhoE) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		double tmp_Res_rho = 0;
		double tmp_Res_rhou0 = 0;
		double tmp_Res_rhou1 = 0;
		double tmp_Res_rhou2 = 0;
		double tmp_Res_rhoE = 0;
		
		double tmp_dTd2xi = 0;
		


		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double rhou0_dy_ll, rhou0_dy_l, rhou0_dy_c, rhou0_dy_r, rhou0_dy_rr;
		rhou0_dy_ll = irhou0_c[dy_ll];
		rhou0_dy_l = irhou0_c[dy_l];
		rhou0_dy_c = irhou0_c[gidx];
		rhou0_dy_r = irhou0_c[dy_r];
		rhou0_dy_rr = irhou0_c[dy_rr];

		double rhou1_dy_ll, rhou1_dy_l, rhou1_dy_c, rhou1_dy_r, rhou1_dy_rr;
		rhou1_dy_ll = irhou1_c[dy_ll];
		rhou1_dy_l = irhou1_c[dy_l];
		rhou1_dy_c = irhou1_c[gidx];
		rhou1_dy_r = irhou1_c[dy_r];
		rhou1_dy_rr = irhou1_c[dy_rr];

		double rhou2_dy_ll, rhou2_dy_l, rhou2_dy_c, rhou2_dy_r, rhou2_dy_rr;
		rhou2_dy_ll = irhou2_c[dy_ll];
		rhou2_dy_l = irhou2_c[dy_l];
		rhou2_dy_c = irhou2_c[gidx];
		rhou2_dy_r = irhou2_c[dy_r];
		rhou2_dy_rr = irhou2_c[dy_rr];

		double rhoE_dy_ll, rhoE_dy_l, rhoE_dy_c, rhoE_dy_r, rhoE_dy_rr;
		rhoE_dy_ll = irhoE_c[dy_ll];
		rhoE_dy_l = irhoE_c[dy_l];
		rhoE_dy_c = irhoE_c[gidx];
		rhoE_dy_r = irhoE_c[dy_r];
		rhoE_dy_rr = irhoE_c[dy_rr];

		double p_dy_ll, p_dy_l, p_dy_c, p_dy_r, p_dy_rr;
		p_dy_ll = calp(rhoE_dy_ll, rho_dy_ll, rhou0_dy_ll, rhou1_dy_ll, rhou2_dy_ll);
		p_dy_l = calp(rhoE_dy_l, rho_dy_l, rhou0_dy_l, rhou1_dy_l, rhou2_dy_l);
		p_dy_c = calp(rhoE_dy_c, rho_dy_c, rhou0_dy_c, rhou1_dy_c, rhou2_dy_c);
		p_dy_r = calp(rhoE_dy_r, rho_dy_r, rhou0_dy_r, rhou1_dy_r, rhou2_dy_r);
		p_dy_rr = calp(rhoE_dy_rr, rho_dy_rr, rhou0_dy_rr, rhou1_dy_rr, rhou2_dy_rr);

		tmp_Res_rho += -0.5 * dns_pDer1(rho_dy_ll, rho_dy_l, rho_dy_r, rho_dy_rr, DY) * rhou1_dy_c / rho_dy_c;
		
		tmp_Res_rhou1 += -dns_pDer1(p_dy_ll, p_dy_l, p_dy_r, p_dy_rr, DY);
		tmp_Res_rhoE -= dns_pDer1(p_dy_ll * rhou1_dy_ll / rho_dy_ll, p_dy_l * rhou1_dy_l / rho_dy_l, p_dy_r * rhou1_dy_r / rho_dy_r, p_dy_rr * rhou1_dy_rr / rho_dy_rr, DY);
		
		tmp_Res_rhoE += -0.5 * dns_pDer1(rhoE_dy_ll, rhoE_dy_l, rhoE_dy_r, rhoE_dy_rr, DY) * rhou1_dy_c / rho_dy_c;
		tmp_Res_rhoE += -0.5 * dns_pDer1(rhoE_dy_ll * rhou1_dy_ll / rho_dy_ll, rhoE_dy_l * rhou1_dy_l / rho_dy_l, rhoE_dy_r * rhou1_dy_r / rho_dy_r, rhoE_dy_rr * rhou1_dy_rr / rho_dy_rr, DY);
		
		tmp_Res_rhou0 += -0.5 * dns_pDer1(rhou0_dy_ll * rhou1_dy_ll / rho_dy_ll, rhou0_dy_l * rhou1_dy_l / rho_dy_l, rhou0_dy_r * rhou1_dy_r / rho_dy_r, rhou0_dy_rr * rhou1_dy_rr / rho_dy_rr, DY);
		tmp_Res_rhou2 += -0.5 * dns_pDer1(rhou2_dy_ll * rhou1_dy_ll / rho_dy_ll, rhou2_dy_l * rhou1_dy_l / rho_dy_l, rhou2_dy_r * rhou1_dy_r / rho_dy_r, rhou2_dy_rr * rhou1_dy_rr / rho_dy_rr, DY);

		tmp_dTd2xi  += dns_pDer2(calT(p_dy_ll, rho_dy_ll), calT(p_dy_l, rho_dy_l), calT(p_dy_c, rho_dy_c), calT(p_dy_r, rho_dy_r), calT(p_dy_rr, rho_dy_rr), DY);
		
		
		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double rhou0_dz_ll, rhou0_dz_l, rhou0_dz_c, rhou0_dz_r, rhou0_dz_rr;
		rhou0_dz_ll = irhou0_c[dz_ll];
		rhou0_dz_l = irhou0_c[dz_l];
		rhou0_dz_c = irhou0_c[gidx];
		rhou0_dz_r = irhou0_c[dz_r];
		rhou0_dz_rr = irhou0_c[dz_rr];

		double rhou1_dz_ll, rhou1_dz_l, rhou1_dz_c, rhou1_dz_r, rhou1_dz_rr;
		rhou1_dz_ll = irhou1_c[dz_ll];
		rhou1_dz_l = irhou1_c[dz_l];
		rhou1_dz_c = irhou1_c[gidx];
		rhou1_dz_r = irhou1_c[dz_r];
		rhou1_dz_rr = irhou1_c[dz_rr];

		double rhou2_dz_ll, rhou2_dz_l, rhou2_dz_c, rhou2_dz_r, rhou2_dz_rr;
		rhou2_dz_ll = irhou2_c[dz_ll];
		rhou2_dz_l = irhou2_c[dz_l];
		rhou2_dz_c = irhou2_c[gidx];
		rhou2_dz_r = irhou2_c[dz_r];
		rhou2_dz_rr = irhou2_c[dz_rr];

		double rhoE_dz_ll, rhoE_dz_l, rhoE_dz_c, rhoE_dz_r, rhoE_dz_rr;
		rhoE_dz_ll = irhoE_c[dz_ll];
		rhoE_dz_l = irhoE_c[dz_l];
		rhoE_dz_c = irhoE_c[gidx];
		rhoE_dz_r = irhoE_c[dz_r];
		rhoE_dz_rr = irhoE_c[dz_rr];

		double p_dz_ll, p_dz_l, p_dz_c, p_dz_r, p_dz_rr;
		p_dz_ll = calp(rhoE_dz_ll, rho_dz_ll, rhou0_dz_ll, rhou1_dz_ll, rhou2_dz_ll);
		p_dz_l = calp(rhoE_dz_l, rho_dz_l, rhou0_dz_l, rhou1_dz_l, rhou2_dz_l);
		p_dz_c = calp(rhoE_dz_c, rho_dz_c, rhou0_dz_c, rhou1_dz_c, rhou2_dz_c);
		p_dz_r = calp(rhoE_dz_r, rho_dz_r, rhou0_dz_r, rhou1_dz_r, rhou2_dz_r);
		p_dz_rr = calp(rhoE_dz_rr, rho_dz_rr, rhou0_dz_rr, rhou1_dz_rr, rhou2_dz_rr);

		tmp_Res_rho += -0.5 * dns_pDer1(rho_dz_ll, rho_dz_l, rho_dz_r, rho_dz_rr, DZ) * rhou2_dz_c / rho_dz_c;
		
		tmp_Res_rhou2 += -dns_pDer1(p_dz_ll, p_dz_l, p_dz_r, p_dz_rr, DZ);
		tmp_Res_rhoE -= dns_pDer1(p_dz_ll * rhou2_dz_ll / rho_dz_ll, p_dz_l * rhou2_dz_l / rho_dz_l, p_dz_r * rhou2_dz_r / rho_dz_r, p_dz_rr * rhou2_dz_rr / rho_dz_rr, DZ);
		
		tmp_Res_rhoE += -0.5 * dns_pDer1(rhoE_dz_ll, rhoE_dz_l, rhoE_dz_r, rhoE_dz_rr, DZ) * rhou2_dz_c / rho_dz_c;
		
		tmp_Res_rhoE += -0.5 * dns_pDer1(rhoE_dz_ll * rhou2_dz_ll / rho_dz_ll, rhoE_dz_l * rhou2_dz_l / rho_dz_l, rhoE_dz_r * rhou2_dz_r / rho_dz_r, rhoE_dz_rr * rhou2_dz_rr / rho_dz_rr, DZ);
		
		tmp_Res_rhou0 += -0.5 * dns_pDer1(rhou0_dz_ll * rhou2_dz_ll / rho_dz_ll, rhou0_dz_l * rhou2_dz_l / rho_dz_l, rhou0_dz_r * rhou2_dz_r / rho_dz_r, rhou0_dz_rr * rhou2_dz_rr / rho_dz_rr, DZ);
		tmp_Res_rhou1 += -0.5 * dns_pDer1(rhou1_dz_ll * rhou2_dz_ll / rho_dz_ll, rhou1_dz_l * rhou2_dz_l / rho_dz_l, rhou1_dz_r * rhou2_dz_r / rho_dz_r, rhou1_dz_rr * rhou2_dz_rr / rho_dz_rr, DZ);
		
		tmp_dTd2xi  +=  dns_pDer2(calT(p_dz_ll, rho_dz_ll), calT(p_dz_l, rho_dz_l), calT(p_dz_c, rho_dz_c), calT(p_dz_r, rho_dz_r), calT(p_dz_rr, rho_dz_rr), DZ);	


		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double rhou0_dx_ll, rhou0_dx_l, rhou0_dx_c, rhou0_dx_r, rhou0_dx_rr;
		rhou0_dx_ll = irhou0_ll[gidx];
		rhou0_dx_l = irhou0_l[gidx];
		rhou0_dx_c = irhou0_c[gidx];
		rhou0_dx_r = irhou0_r[gidx];
		rhou0_dx_rr = irhou0_rr[gidx];

		double rhou1_dx_ll, rhou1_dx_l, rhou1_dx_c, rhou1_dx_r, rhou1_dx_rr;
		rhou1_dx_ll = irhou1_ll[gidx];
		rhou1_dx_l = irhou1_l[gidx];
		rhou1_dx_c = irhou1_c[gidx];
		rhou1_dx_r = irhou1_r[gidx];
		rhou1_dx_rr = irhou1_rr[gidx];

		double rhou2_dx_ll, rhou2_dx_l, rhou2_dx_c, rhou2_dx_r, rhou2_dx_rr;
		rhou2_dx_ll = irhou2_ll[gidx];
		rhou2_dx_l = irhou2_l[gidx];
		rhou2_dx_c = irhou2_c[gidx];
		rhou2_dx_r = irhou2_r[gidx];
		rhou2_dx_rr = irhou2_rr[gidx];

		double rhoE_dx_ll, rhoE_dx_l, rhoE_dx_c, rhoE_dx_r, rhoE_dx_rr;
		rhoE_dx_ll = irhoE_ll[gidx];
		rhoE_dx_l = irhoE_l[gidx];
		rhoE_dx_c = irhoE_c[gidx];
		rhoE_dx_r = irhoE_r[gidx];
		rhoE_dx_rr = irhoE_rr[gidx];

		double p_dx_ll, p_dx_l, p_dx_c, p_dx_r, p_dx_rr;
		p_dx_ll = calp(rhoE_dx_ll, rho_dx_ll, rhou0_dx_ll, rhou1_dx_ll, rhou2_dx_ll);
		p_dx_l = calp(rhoE_dx_l, rho_dx_l, rhou0_dx_l, rhou1_dx_l, rhou2_dx_l);
		p_dx_c = calp(rhoE_dx_c, rho_dx_c, rhou0_dx_c, rhou1_dx_c, rhou2_dx_c);
		p_dx_r = calp(rhoE_dx_r, rho_dx_r, rhou0_dx_r, rhou1_dx_r, rhou2_dx_r);
		p_dx_rr = calp(rhoE_dx_rr, rho_dx_rr, rhou0_dx_rr, rhou1_dx_rr, rhou2_dx_rr);

		tmp_Res_rho += -0.5 * dns_pDer1(rho_dx_ll, rho_dx_l, rho_dx_r, rho_dx_rr, DX) * rhou0_dx_c / rho_dx_c;
		
		tmp_Res_rhou0 += -dns_pDer1(p_dx_ll, p_dx_l, p_dx_r, p_dx_rr, DX);
		tmp_Res_rhoE -= dns_pDer1(p_dx_ll * sy_bc_ll * rhou0_dx_ll / rho_dx_ll, 
												p_dx_l * sy_bc_l * rhou0_dx_l / rho_dx_l, 
												p_dx_r * sy_bc_r * rhou0_dx_r / rho_dx_r, 
												p_dx_rr * sy_bc_rr * rhou0_dx_rr / rho_dx_rr, DX);
		
		tmp_Res_rhoE += -0.5 * dns_pDer1(rhoE_dx_ll, rhoE_dx_l, rhoE_dx_r, rhoE_dx_rr, DX) * rhou0_dx_c / rho_dx_c;
		
		tmp_Res_rhoE += -0.5 * dns_pDer1(rhoE_dx_ll * sy_bc_ll * rhou0_dx_ll / rho_dx_ll, 
														rhoE_dx_l * sy_bc_l * rhou0_dx_l / rho_dx_l, 
														rhoE_dx_r * sy_bc_r * rhou0_dx_r / rho_dx_r, 
														rhoE_dx_rr * sy_bc_rr * rhou0_dx_rr / rho_dx_rr, DX);
		
		tmp_Res_rhou1 += -0.5 * dns_pDer1(rhou1_dx_ll * sy_bc_ll * rhou0_dx_ll / rho_dx_ll, 
										rhou1_dx_l * sy_bc_l * rhou0_dx_l / rho_dx_l, 
										rhou1_dx_r * sy_bc_r * rhou0_dx_r / rho_dx_r, 
										rhou1_dx_rr * sy_bc_rr * rhou0_dx_rr / rho_dx_rr, DX);

		tmp_Res_rhou2 += -0.5 * dns_pDer1(rhou2_dx_ll * sy_bc_ll * rhou0_dx_ll / rho_dx_ll, 
										rhou2_dx_l * sy_bc_l * rhou0_dx_l / rho_dx_l, 
										rhou2_dx_r * sy_bc_r * rhou0_dx_r / rho_dx_r, 
										rhou2_dx_rr * sy_bc_rr * rhou0_dx_rr / rho_dx_rr, DX);
		
		tmp_dTd2xi  +=  dns_pDer2(calT(p_dx_ll, rho_dx_ll), calT(p_dx_l, rho_dx_l), calT(p_dx_c, rho_dx_c), calT(p_dx_r, rho_dx_r), calT(p_dx_rr, rho_dx_rr), DX);	
			
		tmp_Res_rhoE += tmp_dTd2xi / (MINF * MINF * PR * RE * (GAMA - 1));
		
		Res_rho[gidx] = tmp_Res_rho;
		Res_rhou0[gidx] += tmp_Res_rhou0;
		Res_rhou1[gidx] += tmp_Res_rhou1;
		Res_rhou2[gidx] += tmp_Res_rhou2;
		Res_rhoE[gidx] = tmp_Res_rhoE;
		
	}

}

__global__ void dns_Res_StageAdvance(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ rho,
	/*order 0*/ double * __restrict__ rhou0,
	/*order 0*/ double * __restrict__ rhou1,
	/*order 0*/ double * __restrict__ rhou2,
	/*order 0*/ double * __restrict__ rhoE,

	/*order 0*/ double * __restrict__ du0dx,
	/*order 0*/ double * __restrict__ du0dy,
	/*order 0*/ double * __restrict__ du0dz,

	/*order 0*/ double * __restrict__ du1dx,
	/*order 0*/ double * __restrict__ du1dy,
	/*order 0*/ double * __restrict__ du1dz,

	/*order 0*/ double * __restrict__ du2dx,
	/*order 0*/ double * __restrict__ du2dy,
	/*order 0*/ double * __restrict__ du2dz,

	/*order 0*/ double * __restrict__ drhou0dx,
	/*order 0*/ double * __restrict__ drhou0dy,
	/*order 0*/ double * __restrict__ drhou0dz,

	/*order 0*/ double * __restrict__ drhou1dx,
	/*order 0*/ double * __restrict__ drhou1dy,
	/*order 0*/ double * __restrict__ drhou1dz,

	/*order 0*/ double * __restrict__ drhou2dx,
	/*order 0*/ double * __restrict__ drhou2dy,
	/*order 0*/ double * __restrict__ drhou2dz,
	
	/*order 0*/ double * __restrict__ irho_old,
	/*order 0*/ double * __restrict__ irhou0_old,
	/*order 0*/ double * __restrict__ irhou1_old,
	/*order 0*/ double * __restrict__ irhou2_old,
	/*order 0*/ double * __restrict__ irhoE_old,
	
	/*order 0*/ double * __restrict__ orho,
	/*order 0*/ double * __restrict__ orhou0,
	/*order 0*/ double * __restrict__ orhou1,
	/*order 0*/ double * __restrict__ orhou2,
	/*order 0*/ double * __restrict__ orhoE,

	/*order 0*/ double * __restrict__ orho_old,
	/*order 0*/ double * __restrict__ orhou0_old,
	/*order 0*/ double * __restrict__ orhou1_old,
	/*order 0*/ double * __restrict__ orhou2_old,
	/*order 0*/ double * __restrict__ orhoE_old,
	
	double rknew, double rkold,
	/*order 0*/ double * __restrict__ Res_rho,
	/*order 0*/ double * __restrict__ Res_rhou0,
	/*order 0*/ double * __restrict__ Res_rhou1,
	/*order 0*/ double * __restrict__ Res_rhou2,
	/*order 0*/ double * __restrict__ Res_rhoE,
	/*order 0*/ double * __restrict__ tmp_du0d2xi,
	/*order 0*/ double * __restrict__ tmp_du1d2xi,
	/*order 0*/ double * __restrict__ tmp_du2d2xi) {

	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t idx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	idx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);

	if (idx<block_ncc) {

		double lRes_rho = 0;
		double lRes_rhou0 = 0;
		double lRes_rhou1 = 0;
		double lRes_rhou2 = 0;
		double lRes_rhoE = 0;

		double ldu0dx = du0dx[idx];
		double ldu1dy = du1dy[idx];
		double ldu2dz = du2dz[idx];

		double tmp0 = -0.5 * (ldu0dx + ldu1dy + ldu2dz);
		
		lRes_rho += tmp0 * rho[idx];
		lRes_rhou0 += tmp0 * rhou0[idx];
		lRes_rhou1 += tmp0 * rhou1[idx];
		lRes_rhou2 += tmp0 * rhou2[idx];
		lRes_rhoE += tmp0 * rhoE[idx];

		double frac0 = 1./RE;
		double frac1 = 2./3.;
		double frac2 = 4./3.;
		double frac3 = 1./3.;
		lRes_rhoE += frac0 * (-frac1 * ldu0dx - frac1 * ldu1dy + frac2 * ldu2dz) * ldu2dz;
		lRes_rhoE += frac0 * (-frac1 * ldu0dx + frac2 * ldu1dy - frac1 * ldu2dz) * ldu1dy;
		lRes_rhoE += frac0 * ( frac2 * ldu0dx - frac1 * ldu1dy - frac1 * ldu2dz) * ldu0dx;

		double lu0 = rhou0[idx] / rho[idx];
		lRes_rhou1 += -0.5 * lu0 * drhou1dx[idx];
		lRes_rhou2 += -0.5 * lu0 * drhou2dx[idx];

		double ldrhou0dx = drhou0dx[idx];
		lRes_rhou0 += -0.5 * lu0 * ldrhou0dx; 
		lRes_rho +=   -0.5 * ldrhou0dx; 

		
		tmp0 = frac0 * tmp_du0d2xi[idx];

		lRes_rhou0 += tmp0;
		lRes_rhoE += lu0 * tmp0;


		double lu1 = rhou1[idx] / rho[idx];
		lRes_rhou0 += -0.5 * lu1 * drhou0dy[idx];
		lRes_rhou2 += -0.5 * lu1 * drhou2dy[idx];

		double ldrhou1dy = drhou1dy[idx];
		lRes_rhou1 += -0.5 * lu1 * ldrhou1dy;
		lRes_rho +=   -0.5 * ldrhou1dy;

		tmp0 = frac0 * tmp_du1d2xi[idx];

		lRes_rhou1 += tmp0;
		lRes_rhoE += lu1 * tmp0;


		double lu2 = rhou2[idx] / rho[idx];
		lRes_rhou0 += -0.5 * lu2 * drhou0dz[idx];
		lRes_rhou1 += -0.5 * lu2 * drhou1dz[idx];

		double ldrhou2dz = drhou2dz[idx];
		lRes_rhou2 += -0.5 * lu2 * ldrhou2dz;
		lRes_rho +=   -0.5 * ldrhou2dz;


		lRes_rho += Res_rho[idx];

		orho[idx] = DT * rknew * lRes_rho + irho_old[idx];
		orho_old[idx] = DT * rkold * lRes_rho + irho_old[idx];


		tmp0 = frac0 * tmp_du2d2xi[idx];

		lRes_rhou2 += tmp0;
		lRes_rhoE += lu2 * tmp0;


		lRes_rhou0 += Res_rhou0[idx];

		orhou0[idx] = DT * rknew * lRes_rhou0 + irhou0_old[idx];
		orhou0_old[idx] = DT * rkold * lRes_rhou0 + irhou0_old[idx];


		lRes_rhou1 += Res_rhou1[idx];

		orhou1[idx] = DT * rknew * lRes_rhou1 + irhou1_old[idx];
		orhou1_old[idx] = DT * rkold * lRes_rhou1 + irhou1_old[idx];


		lRes_rhou2 += Res_rhou2[idx];

		orhou2[idx] = DT * rknew * lRes_rhou2 + irhou2_old[idx];
		orhou2_old[idx] = DT * rkold * lRes_rhou2 + irhou2_old[idx];


		lRes_rhoE += 1./RE *	(du0dy[idx] + du1dx[idx]) * du0dy[idx]
					+ 1./RE *	(du0dy[idx] + du1dx[idx]) * du1dx[idx];

		lRes_rhoE += 1./RE *	(du0dz[idx] + du2dx[idx]) * du0dz[idx]
					+ 1./RE *	(du0dz[idx] + du2dx[idx]) * du2dx[idx];

		lRes_rhoE += 1./RE *	(du1dz[idx] + du2dy[idx]) * du1dz[idx]
					+ 1./RE *	(du1dz[idx] + du2dy[idx]) * du2dy[idx];

		lRes_rhoE += Res_rhoE[idx];

		orhoE[idx] = DT * rknew * lRes_rhoE + irhoE_old[idx];
		orhoE_old[idx] = DT * rkold * lRes_rhoE + irhoE_old[idx];

	}

}



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Utility
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void dns_copy(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ f,
	/*order 0*/ double * __restrict__ g) {
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		g[idx] = f[idx];
	}

}


__global__ void dns_init(const double * __restrict__ p_in, double * __restrict__ p_out) {
	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_header_size) {
		p_out[idx]=p_in[idx];
	}


}


__global__ void dns_DebugAdvance(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ f0_out,
	/*order 0*/ double * __restrict__ f0_in) {

	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		f0_out[idx] = f0_in[idx] + 1.0;

	}
}

__global__ void dns_Debug(int32_t i_worker, int32_t order_in, int32_t order_out,
	/*order 0*/ double * __restrict__ f0_in, /*order 0*/ double * __restrict__ f0_out) {

	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<2) {
		printf("In: f0[%d] = %lf    ", idx, f0_in[idx]);
		if (idx == 0) {
			f0_out[idx] = f0_in[idx];
		}
		if (idx == 1) {
			printf("Out old: f0[%d] = %lf    ", idx, f0_out[idx]);
			f0_out[idx] = f0_in[idx] + 1;
			printf("Out new: f0[%d] = %lf    ", idx, f0_out[idx]);
		}

	}
}



void DS::caller_worker (double ** p_in, double ** p_out, int32_t i_part, int32_t i_super_cycle,
						int32_t order_in, int32_t order_out, int32_t iworker, int32_t nworker,
						hipStream_t * stream, int32_t threads_per_block, int32_t blockSize, int32_t myID) {

	//cout << "in:" << p_in[0] << " " << p_in[1] << " " << p_in[2] << " " << p_in[3] << " " << p_in[4] << " " << p_in[5] << endl;
	//cout << "out:" << p_out[0] << " " << p_out[1] << " " << p_out[2] << " " << p_out[3] << " " << p_out[4] << " " << p_out[5] << endl;

	// the order of arrays in p_in and p_out is:
	// center, left, right, left-left, right-right, left-left-left, right-right-right, and so on
	// entries can be NULL when invalid

	//cout << NX << ", " << NY << ", " << NZ << ", " << DX << ", " << DY << ", " << DZ <<  ", " <<  DT << ", " <<  GAMA <<  ", " <<  MINF <<  ", " <<  RE <<  ", " <<  PR << endl;


	int32_t global_worker_id = nworker * myID + iworker;
	int32_t n_global_worker = n_procs * n_worker;
	int32_t stage = (global_worker_id + n_global_worker * i_super_cycle) % 3;

	//cout << "Working on stage " << stage << endl;

	double rkold = RKOLD[stage];
	double rknew = RKNEW[stage];

	//cout << "rkold " << rkold << " rknew " << rknew << endl;

	
	
	// Sort out parts
	double* p_c = p_in[0];
	double* p_l = p_in[1];
	double* p_r = p_in[2];
	double* p_ll = p_in[3];
	double* p_rr = p_in[4];

	// Sort out parts
	double* p_c_out = p_out[0];

	// Symmetry BC u0 = -u0
	int sy_bc_ll = 1;
	int sy_bc_l  = 1;
	int sy_bc_r  = 1;
	int sy_bc_rr = 1;

	if (i_part == 0) {
		p_ll = p_rr;
		p_l = p_r;
		sy_bc_ll = -1;
		sy_bc_l  = -1;
	} else if(i_part == 1) {
		p_ll = p_c;
		sy_bc_ll = -1;
	} else if (i_part == my_n_part - 2) {
		p_rr = p_c;
		sy_bc_rr = -1;
	} else if (i_part == my_n_part - 1) {
		p_rr = p_ll;
		p_r = p_l;
		sy_bc_rr = -1;
		sy_bc_r  = -1;
	}

	// Offsets in pages for each field
	size_t offset_rho   = block_header_size + 0*block_ncc;
	size_t offset_rhou0 = block_header_size + 1*block_ncc;
	size_t offset_rhou1 = block_header_size + 2*block_ncc;
	size_t offset_rhou2 = block_header_size + 3*block_ncc;
	size_t offset_rhoE = block_header_size + 4*block_ncc;
	size_t offset_rho_old   = block_header_size + 5*block_ncc;
	size_t offset_rhou0_old = block_header_size + 6*block_ncc;
	size_t offset_rhou1_old = block_header_size + 7*block_ncc;
	size_t offset_rhou2_old = block_header_size + 8*block_ncc;
	size_t offset_rhoE_old = block_header_size + 9*block_ncc;
	size_t offset_tmp0 = block_header_size + 10*block_ncc;
	size_t offset_tmp1 = block_header_size + 11*block_ncc;
	size_t offset_tmp2 = block_header_size + 12*block_ncc;
	size_t offset_tmp3 = block_header_size + 13*block_ncc;
	size_t offset_tmp4 = block_header_size + 14*block_ncc;



	//cout << "Blocksize: " << blockSize << ", threads_per_block: " << threads_per_block << endl;

	threads_per_block = BLOCKSIZE_Z * BLOCKSIZE_Y;
	int32_t gridSize = (blockSize + threads_per_block - 1) / threads_per_block;

	//cout << "Slice Size: " << blockSize << ", gridSize: " << gridSize << ", started Threads: " << gridSize * threads_per_block << endl;


	if (stage == 0) {
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], &p_c[offset_rho_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], &p_c[offset_rhou0_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], &p_c[offset_rhou1_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], &p_c[offset_rhou2_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhoE], &p_c[offset_rhoE_old]);
	}



	dns_du0dxyz <<<gridSize,threads_per_block,0,*stream>>>(myID, i_part, i_super_cycle, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou0], &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
		sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
		(double*) d_drhou0dx, (double*) d_drhou0dy, (double*) d_drhou0dz,
		(double*) d_du0dx, (double*) d_du0dy, (double*) d_du0dz, 
		(double*) d_Res_rhou0,
		(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	dns_du1dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou1], &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
		(double*) d_drhou1dx, (double*) d_drhou1dy, (double*) d_drhou1dz,
		(double*) d_du1dx, (double*) d_du1dy, (double*) d_du1dz, 
		(double*) d_Res_rhou1,
		(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	dns_du2dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou2], &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
		(double*) d_drhou2dx, (double*) d_drhou2dy, (double*) d_drhou2dz,
		(double*) d_du2dx, (double*) d_du2dy, (double*) d_du2dz, 
		(double*) d_Res_rhou2,
		(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	dns_drhoETpdxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
		&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
		&p_c[offset_rhou0], &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
		&p_c[offset_rhou1], &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
		&p_c[offset_rhou2], &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
		&p_c[offset_rhoE], &p_l[offset_rhoE], &p_r[offset_rhoE], &p_ll[offset_rhoE], &p_rr[offset_rhoE],
		sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
		(double*) d_Res_rho,
		(double*) d_Res_rhou0,
		(double*) d_Res_rhou1,
		(double*) d_Res_rhou2,
		(double*) d_Res_rhoE);

	dns_Res_StageAdvance <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
			(double*) &p_c[offset_rho],
			&p_c[offset_rhou0], &p_c[offset_rhou1], &p_c[offset_rhou2], &p_c[offset_rhoE],
			(double*) d_du0dx, (double*) d_du0dy, (double*) d_du0dz,
			(double*) d_du1dx, (double*) d_du1dy, (double*) d_du1dz,
			(double*) d_du2dx, (double*) d_du2dy, (double*) d_du2dz,
			(double*) d_drhou0dx, (double*) d_drhou0dy, (double*) d_drhou0dz,
			(double*) d_drhou1dx, (double*) d_drhou1dy, (double*) d_drhou1dz,
			(double*) d_drhou2dx, (double*) d_drhou2dy, (double*) d_drhou2dz,
			&p_c[offset_rho_old],
			&p_c[offset_rhou0_old], &p_c[offset_rhou1_old], &p_c[offset_rhou2_old],
			&p_c[offset_rhoE_old],
			&p_c_out[offset_rho], 
			&p_c_out[offset_rhou0], &p_c_out[offset_rhou1], &p_c_out[offset_rhou2],
			&p_c_out[offset_rhoE],
			&p_c_out[offset_rho_old],
			&p_c_out[offset_rhou0_old], &p_c_out[offset_rhou1_old], &p_c_out[offset_rhou2_old],
			&p_c_out[offset_rhoE_old],
			rknew, rkold,
			(double*) d_Res_rho,
			(double*) d_Res_rhou0,
			(double*) d_Res_rhou1,
			(double*) d_Res_rhou2,
			(double*) d_Res_rhoE,
			(double*) tmp_du0d2xi, (double*) tmp_du1d2xi, (double*) tmp_du2d2xi);

	//// Copy Header
	dns_init <<<gridSize,threads_per_block,0,*stream>>>((const double*)p_c, (double*)p_c_out);
	//hipMemcpy((void*)p_c_out,(const void*)p_c,block_header_size * sizeof(double),hipMemcpyDeviceToDevice); cudaCheckError(__LINE__,__FILE__);
	// 4

}


__global__ void prepare_visual_rectilinear(double * __restrict__ p_in, double * __restrict__ p_out) {

	int32_t global_id = blockIdx.x*blockDim.x+threadIdx.x;
	// int32_t n_threads = blockDim.x*gridDim.x;

	// int32_t * p_in_i32 = (int32_t *)p_in;
	int64_t * p_in_i64 = (int64_t*)p_in;
	double * p_in_d = (double*)p_in;


	int64_t i_part=p_in_i64[0];

	// if (global_id==0) {
	// 	printf("part:%i\n",i_part);
	// }

	double * p_out_double=(double*)p_out;


	// if (global_id==0) {
	// 	p_out_i32[0]=n_mol;
	// 	p_out_i32[1]=i_part;
	// }
	if (global_id<block_ncc) {
		int32_t i_cell=global_id;
		int32_t i_x=i_part;
		int32_t i_y=i_cell/my_n_part;
		int32_t i_z=i_cell-i_y*my_n_part;

		//printf("i_part_%i_%i_%i_%i_\n",i_x,i_y,i_z,i_part);

		for (int32_t i_field=0;i_field<block_n_fields;i_field++) {
			double dtmp=p_in_d[block_header_size+i_field*block_ncc+i_cell];

			/*
			if (i_field == 0) {
			if (i_cell < 2) {
				printf(" \n i_cell = %d, dtmp: %lf", i_cell, dtmp);
			}
			if (i_cell == 0) {
				printf("\nI write my part index to %d", i_field*my_n_part*block_ncc+i_z*block_ncc+i_y*my_n_part+i_x);
				printf("\nCalculated from: i_field %d, my_n_part %d, block_ncc %d, i_z %d, i_y %d, i_x %d\n", i_field, my_n_part, block_ncc, i_z, i_y, i_x);
			}
			}
			*/
			

			
			//double dtmp = 5.0;

			p_out_double[i_field*my_n_part*block_ncc+i_z*block_ncc+i_y*my_n_part+i_x]=dtmp;
		}
	}

}

void DS::write_vtr (double * p_data, int32_t i_part, int32_t i_cycle) {
	string FileName;
	FileName.append("/direc/visual_");
	FileName+=to_string(my_n_part);
	FileName.append("_");
	FileName+=to_string(i_cycle);
	// FileName.append("/visual_");
	// FileName+=to_string(i_part);
	FileName.append(".vtr");

	/*
	printf("\np_data: ");
	
	for (int i = 0; i < 27; ++i) {
		if (i % 3 == 0) printf("\n");
		if (i % 9 == 0) printf("\n");
		printf("%f, ", p_data[i]);
		
	}
	printf("\n");
	*/
	
	

	ofstream ofs;
	ofs.open(FileName, ios::out | ios::binary);
	if (ofs) {
		int64_t append_offset=0;
		ofs << "<VTKFile type=\"RectilinearGrid\" version=\"1.0\" byte_order=\"LittleEndian\" header_type=\"UInt64\">" << endl;
		ofs << "<RectilinearGrid WholeExtent=\"" << "0 " << my_n_part-1 << " 0 " << my_n_part-1 << " 0 " << my_n_part-1 << "\">" << endl;
		ofs << "<Piece Extent=\"" << "0 " << my_n_part-1 << " 0 " << my_n_part-1 << " 0 " << my_n_part-1 << "\">" << endl;

		ofs << "<PointData Scalars=\"\" Name=\"a\">" << endl;
		ofs << "<DataArray type=\"Float64\" Name=\"rho\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou0\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou1\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou2\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhoE\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rho_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou0_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhou1_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
		
		ofs << "<DataArray type=\"Float64\" Name=\"rhou2_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"rhoE_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);

		// ============================ TMP Output ============================
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp0\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp1\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp2\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp3\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float64\" Name=\"tmp4\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(double)+sizeof(int64_t);
//
		// ============================ TMP Output ============================

		ofs << "</PointData>" << endl;

		ofs << "<Coordinates>" << endl;
		ofs << "<DataArray type=\"Float64\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(double)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float64\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(double)+sizeof(int64_t);

		ofs << "</Coordinates>" << endl;

		// ofs << "\" NumberOfCells=\"0\">" << endl;
		// ofs << "<PointData Scalars=\"species\">" << endl;
		// ofs << "<DataArray type=\"Float32\" Name=\"species\" format=\"appended\" offset=\"0\" RangeMin=\"0\" RangeMax=\"6\">" << endl;
		// ofs << "</DataArray>" << endl;
		// ofs << "</PointData>" << endl;
		// ofs << "<Points>" << endl;
		// ofs << "<DataArray type=\"Float32\" Name=\"Points\" NumberOfComponents=\"3\" format=\"appended\" offset=\"";
		// ofs << n_mol*sizeof(double)+8;
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		// ofs << "</DataArray>" << endl;
		// ofs << "</Points>" << endl;
		// ofs << "<Cells>" << endl;
		// ofs << "<DataArray type=\"Int32\" Name=\"connectivity\"></DataArray>" << endl;
		// ofs << "<DataArray type=\"Int32\" Name=\"offsets\"></DataArray>" << endl;
		// ofs << "<DataArray type=\"UInt8\" Name=\"types\"></DataArray>" << endl;
		// ofs << "</Cells>" << endl;
		ofs << "</Piece>" << endl;
		ofs << "</RectilinearGrid>" << endl;
		ofs << "<AppendedData encoding=\"raw\">" << endl;
		ofs << "_";	// mark start of appended data
		ofs.close();
	}

	// write appended data
	int64_t size_append=0;

	// cell data
	for (int32_t i_field=0;i_field<block_n_fields;i_field++) {
		size_append=(my_n_part*block_ncc)*sizeof(double);
		MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
		MemToFile((int64_t*)&p_data[i_field*my_n_part*block_ncc],size_append,(char*)FileName.c_str(),0);
	}

	// coordinates - same for x,y,z
	double * x_coordinates=new double [my_n_part+1];
	for (int i=0;i<my_n_part+1;i++) {
		x_coordinates[i]=i;
	}
	size_append=(my_n_part+1)*sizeof(double);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);

	size_append=(my_n_part+1)*sizeof(double);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);

	size_append=(my_n_part+1)*sizeof(double);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);
	delete [] x_coordinates;

	// write closing tags
	ofs.open(FileName, ios::out | ios::binary | ios_base::app);
	if (ofs) {
		ofs << "</AppendedData>" << endl;
		ofs << "</VTKFile>" << endl;
		ofs.close();
	}
}

void DS::caller_output_vtk_rectilinear (double * p_in, double * p_out, hipStream_t * stream, int32_t threads_per_block, int32_t blockSize, int32_t myID, int32_t i_cycle, int32_t i_part) {

	int32_t n_blocks=block_ncc/threads_per_block;
	n_blocks++;

	prepare_visual_rectilinear <<<n_blocks,threads_per_block,0,*stream>>> (p_in,p_out);
	// int32_t * p_my_vis_i32=(int32_t*)p_my_vis;
	// float * p_my_vis_float=(float*)p_my_vis;
	if (i_part==(my_n_part-1)) {
		// last part
		double * p_my_vis_double=new double[block_n_fields*my_n_part*block_ncc];

		hipDeviceSynchronize();        cudaCheckError(__LINE__,__FILE__);

		size_t copy_size=1;
		copy_size*=block_n_fields;
		copy_size*=my_n_part;
		copy_size*=block_ncc;
		copy_size*=sizeof(double);
		// cout << copy_size << endl;
		hipError_t cer=hipMemcpy((void*)p_my_vis_double,(const void*)p_out,copy_size,hipMemcpyDeviceToHost); //cudaCheckError(__LINE__,__FILE__);
		cout << cer << endl;
		// for (int i=0;i<block_n_fields*my_n_part*block_ncc;i++) cout << p_my_vis_float[i] << endl;

		// string new_dir;
		// new_dir.append("visual/visual_");
		// new_dir+=to_string(i_cycle);

		// boost::filesystem::create_directory(new_dir.c_str());
		write_vtr(p_my_vis_double,0,i_cycle);
		delete [] p_my_vis_double;
	}
}
