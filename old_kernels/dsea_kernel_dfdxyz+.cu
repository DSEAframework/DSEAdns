#include "hip/hip_runtime.h"
// Data Streaming for Explicit Algorithms - DSEA

#include <dsea.h>
#include <stdio.h>		// printf
#include <hipcub/hipcub.hpp>
#include <climits>       // for INT_MAX
#include <fstream>
#include <boost/filesystem.hpp>

using namespace :: std;

// print variadic template values
// overload
template<typename T>
void myprint(T head)
{
    std::cout << head << std::endl;
}
// base case: used when pack is non-empty
template<typename T, typename... Ts>
void myprint(T head, Ts... tail)
{
    std::cout << head << std::endl;
    myprint(tail...);
}

// Calculate the Z Y Coordinates in the grid from an thread ID
// ID: Array Index/Thread ID
// Y,Z: Grid coordinates in part, Y row, Z column
// NC: Number of Columns (NZ)

#define COORDS(ID, Y, Z, NC) \
	do { \
    Y = (ID) / (NC); \
    Z = (ID) % (NC); \
  } while(0)

// Calculate the array index from the grid coordinates
// ID: Array Index/Thread ID
// Y,Z: Grid coordinates in part, Z column, Y row
// NC: Number of Columns (NZ)
#define IDX(Y, Z, ID, NC) \ 
	do { \
    ID = (Y) * (NC) + (Z); \
  } while(0)


// Calculates the global array index from 
__device__ int32_t thread_to_global_idx(int32_t problemsize, int32_t thread_idx, 
						int32_t block_size_z, int32_t block_size_y, 
						int32_t warp_size_z, int32_t warp_size_y,
						int32_t* c_i_block_out, int32_t* r_i_block_out) {
	int32_t global_idx;


	int32_t block_size = block_size_z * block_size_y;
	int32_t num_blocks = (problemsize*problemsize) / block_size;
	int32_t blocks_per_row = (problemsize) / block_size_z;

	int32_t warp_size = warp_size_z * warp_size_y;
	int32_t num_warps_per_block = block_size / warp_size;
	int32_t warps_per_row = block_size_z / warp_size_z;

	int32_t block_idx = thread_idx / block_size;
	int32_t block_row = block_idx / blocks_per_row;
	int32_t block_col = block_idx % blocks_per_row;

	int32_t idx_in_block = thread_idx % block_size;

	int32_t warp_idx = idx_in_block / warp_size;
	int32_t warp_row = warp_idx / warps_per_row;
	int32_t warp_col = warp_idx % warps_per_row;

	int32_t idx_in_warp = idx_in_block % warp_size;

	int32_t col_in_warp = idx_in_warp % warp_size_z;
	int32_t row_in_warp = idx_in_warp / warp_size_z;

	int32_t col_in_block = warp_col * warp_size_z + col_in_warp;
	int32_t row_in_block = warp_row * warp_size_y + row_in_warp;

	int32_t global_row = block_row * block_size_y + row_in_block;
	int32_t global_col = block_col * block_size_z + col_in_block;

	global_idx = global_row * problemsize + global_col;

	*c_i_block_out = col_in_block;
	*r_i_block_out = row_in_block;

	return global_idx;
	

}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// First Partial Derivatives
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void dns_dfdx(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 1*/ double * __restrict__ f_l, double * __restrict__ f_r,
						/*order 2*/ double * __restrict__ f_ll, double * __restrict__ f_rr,
						/*order 0*/ double * __restrict__ dfdx, int sy_bc_ll = 1, int sy_bc_l = 1, int sy_bc_r = 1, int sy_bc_rr = 1) {

	// Calculate position in part
	//int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;

	int32_t col_in_block, row_in_block;
	int32_t idx = thread_to_global_idx(1024, tidx, 128, 1, 32, 1, &col_in_block, &row_in_block);

	if (idx<block_ncc) {
		dfdx[idx] = 1/DX * (1./12. * sy_bc_ll *  f_ll[idx] - 2./3. * sy_bc_l * f_l[idx] + 2./3. * sy_bc_r * f_r[idx] - 1./12. * sy_bc_rr * f_rr[idx]); 
		//printf("dfdx My Id: %d, values: %lf/%lf/%lf/%lf\n", idx, f_ll[idx], f_l[idx], f_r[idx], f_rr[idx]);
	}

}

__global__ void dns_dfgdx(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 1*/ double * __restrict__ f_l, double * __restrict__ f_r,
						/*order 2*/ double * __restrict__ f_ll, double * __restrict__ f_rr,
						/*order 1*/ double * __restrict__ g_l, double * __restrict__ g_r,
						/*order 2*/ double * __restrict__ g_ll, double * __restrict__ g_rr,
						/*order 0*/ double * __restrict__ dfgdx, 
						int sy_bc_f_ll = 1, int sy_bc_f_l = 1, int sy_bc_f_r = 1, int sy_bc_f_rr = 1,
						int sy_bc_g_ll = 1, int sy_bc_g_l = 1, int sy_bc_g_r = 1, int sy_bc_g_rr = 1) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		dfgdx[idx] = 1/DX * (1./12. * (sy_bc_f_ll * f_ll[idx] * sy_bc_g_ll * g_ll[idx]) - 2./3. * (sy_bc_f_l * f_l[idx] * sy_bc_g_l * g_l[idx]) 
											 + 2./3. * (sy_bc_f_r * f_r[idx] * sy_bc_g_r * g_r[idx]) - 1./12. * (sy_bc_f_rr * f_rr[idx] * sy_bc_g_rr * g_rr[idx])); 
	}

}

__global__ void dns_dfdy(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ dfdy) {

	// Calculate position in part
	//int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;

	int32_t col_in_block, row_in_block;
	int32_t idx = thread_to_global_idx(1024, tidx, 128, 1, 32, 1, &col_in_block, &row_in_block);

	if (idx<block_ncc) {
		int32_t Y, Z;
		int32_t idx_ll, idx_l, idx_r, idx_rr;
		COORDS(idx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, idx_ll, NZ);
		IDX((NY+Y-1)%NY, Z, idx_l, NZ);
		IDX((NY+Y+1)%NY, Z, idx_r, NZ);
		IDX((NY+Y+2)%NY, Z, idx_rr, NZ);

		dfdy[idx] = 1/DY * (1./12. * f[idx_ll] - 2./3. * f[idx_l] + 2./3. * f[idx_r] - 1./12. * f[idx_rr]); 

		//printf("dfdy My Id: %d, my slice coordinates %d,%d, neighbors: %d/%d/%d/%d, values: %lf/%lf/%lf/%lf/%lf, result %lf\n", idx, Y,Z, idx_ll, idx_l, idx_r, idx_rr, f[idx_ll], f[idx_l], f[idx], f[idx_r], f[idx_rr], dfdy[idx]);
	}
}

__global__ void dns_dfgdy(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 1*/ double * __restrict__ f,
						/*order 1*/ double * __restrict__ g,
						/*order 0*/ double * __restrict__ dfgdy) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		int32_t Y, Z;
		int32_t idx_ll, idx_l, idx_r, idx_rr;
		COORDS(idx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, idx_ll, NZ);
		IDX((NY+Y-1)%NY, Z, idx_l, NZ);
		IDX((NY+Y+1)%NY, Z, idx_r, NZ);
		IDX((NY+Y+2)%NY, Z, idx_rr, NZ);

		dfgdy[idx] = 1/DY * (1./12. * (f[idx_ll] * g[idx_ll]) - 2./3. * (f[idx_l] * g[idx_l]) 
											+ 2./3. * (f[idx_r] * g[idx_r]) - 1./12. * (f[idx_rr] * g[idx_rr])); 
	}
}

__global__ void dns_dfdz(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 1*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ dfdz) {

	// Calculate position in part
	//int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;

	int32_t col_in_block, row_in_block;
	int32_t idx = thread_to_global_idx(1024, tidx, 128, 1, 32, 1, &col_in_block, &row_in_block);

	if (idx<block_ncc) {
		int32_t Y, Z;
		int32_t idx_ll, idx_l, idx_r, idx_rr;
		COORDS(idx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, idx_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, idx_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, idx_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, idx_rr, NZ);

		dfdz[idx] = 1/DZ * (1./12. * f[idx_ll] - 2./3. * f[idx_l] + 2./3. * f[idx_r] - 1./12. * f[idx_rr]); 

		//printf("dfdz My Id: %d, my slice coordinates %d,%d, neighbors: %d/%d/%d/%d\n", idx, Y,Z, idx_ll, idx_l, idx_r, idx_rr);
	}

}

__global__ void dns_dfgdz(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 1*/ double * __restrict__ f,
						/*order 1*/ double * __restrict__ g,
						/*order 0*/ double * __restrict__ dfgdz) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		int32_t Y, Z;
		int32_t idx_ll, idx_l, idx_r, idx_rr;
		COORDS(idx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, idx_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, idx_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, idx_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, idx_rr, NZ);

		dfgdz[idx] = 1/DZ * (1./12. * (f[idx_ll] * g[idx_ll]) - 2./3. * (f[idx_l] * g[idx_l]) 
											+ 2./3. * (f[idx_r] * g[idx_r]) - 1./12. * (f[idx_rr] * g[idx_rr])); 
	}

}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Second Partial Derivatives
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void dns_dfd2x(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f_c,
						/*order 1*/ double * __restrict__ f_l, double * __restrict__ f_r,
						/*order 2*/ double * __restrict__ f_ll, double * __restrict__ f_rr,
						/*order 0*/ double * __restrict__ dfd2x, int sy_bc_ll = 1, int sy_bc_l = 1, int sy_bc_r = 1, int sy_bc_rr = 1) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		dfd2x[idx] = 1/(DX*DX) * (-1./12. * sy_bc_ll * f_ll[idx] + 4./3. * sy_bc_l * f_l[idx] - 5./2. * f_c[idx] + 4./3. * sy_bc_r * f_r[idx] - 1./12. * sy_bc_rr * f_rr[idx]); 
	}

}

__global__ void dns_dfd2y(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 1*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ dfd2y) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		int32_t Y, Z;
		int32_t idx_ll, idx_l, idx_r, idx_rr;
		COORDS(idx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, idx_ll, NZ);
		IDX((NY+Y-1)%NY, Z, idx_l, NZ);
		IDX((NY+Y+1)%NY, Z, idx_r, NZ);
		IDX((NY+Y+2)%NY, Z, idx_rr, NZ);

		//printf("dfd2y My Id: %d, my slice coordinates %d,%d, neighbors: %d/%d/%d/%d\n", idx, Y,Z, idx_ll, idx_l, idx_r, idx_rr);

		dfd2y[idx] = 1/(DY*DY) * (-1./12. * f[idx_ll] + 4./3. * f[idx_l] - 5./2. * f[idx] + 4./3. * f[idx_r] - 1./12. * f[idx_rr]); 
	}
}

__global__ void dns_dfd2z(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ dfd2z) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		int32_t Y, Z;
		int32_t idx_ll, idx_l, idx_r, idx_rr;
		COORDS(idx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, idx_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, idx_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, idx_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, idx_rr, NZ);

		//printf("dfd2z My Id: %d, my slice coordinates %d,%d, neighbors: %d/%d/%d/%d\n", idx, Y,Z, idx_ll, idx_l, idx_r, idx_rr);

		dfd2z[idx] = 1/(DZ*DZ) * (-1./12. * f[idx_ll] + 4./3. * f[idx_l] - 5./2. * f[idx] + 4./3. * f[idx_r] - 1./12. * f[idx_rr]); 
	}
}



#ifdef DOPTI
// Version with 1D Kernel Call must be called with nof threads = problemsize
__global__ void dns_dfdyz_shared(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
						int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
						/*order 0*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ dfdy,
						/*order 0*/ double * __restrict__ dfdz) {

	__shared__ float s_d[BLOCKSIZE_Y+4][BLOCKSIZE_Z+4]; // 4-wide halo
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(1024, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		row_in_block+=2;
		col_in_block+=2;
		//dfdx[gidx] = tidx;
		s_d[row_in_block][col_in_block] = f[gidx];

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);

		// get halos
		if (row_in_block < 4) {
			IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
			s_d[row_in_block-2][col_in_block] = f[dy_ll];
		}
		if (row_in_block >= block_size_y) {
			IDX((NY+Y+2)%NY, Z, dy_rr, NZ);
			s_d[row_in_block+2][col_in_block] = f[dy_rr];
		}
		if (col_in_block < 4) {
			IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
			s_d[row_in_block][col_in_block-2] = f[dz_ll];
		}
		if (col_in_block >= block_size_z) {
			IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);
			s_d[row_in_block][col_in_block+2] = f[dz_rr];
		}

		__syncthreads();

		dfdz[gidx] = 1/DZ * (1./12. * s_d[row_in_block][col_in_block-2] - 2./3. * s_d[row_in_block][col_in_block-1] 
												+ 2./3. * s_d[row_in_block][col_in_block+1]  - 1./12. * s_d[row_in_block][col_in_block+2]); 

		dfdy[gidx] = 1/DY * (1./12. * s_d[row_in_block-2][col_in_block] - 2./3. * s_d[row_in_block-1][col_in_block] 
												+ 2./3. * s_d[row_in_block+1][col_in_block]  - 1./12. * s_d[row_in_block+2][col_in_block]); 

	}
}

// Version with 1D Kernel Call must be called with nof threads = problemsize
__global__ void dns_dfdyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
						int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
						/*order 0*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ dfdy,
						/*order 0*/ double * __restrict__ dfdz) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(1024, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);

		dfdz[gidx] = 1/DZ * (1./12. * f[dz_ll] - 2./3. * f[dz_l] + 2./3. * f[dz_r] - 1./12. * f[dz_rr]); 
		dfdy[gidx] = 1/DY * (1./12. * f[dy_ll] - 2./3. * f[dy_l] + 2./3. * f[dy_r] - 1./12. * f[dy_rr]); 

	}
}
#endif


__device__ double dns_pDer1(double v_ll, double v_l, double v_r, double v_rr, double DK) {
	return 1./DK * (1./12. * v_ll - 2./3. * v_l + 2./3. * v_r - 1./12. * v_rr);
}

__device__ double dns_pDer2(double v_ll, double v_l, double v_c, double v_r, double v_rr, double DK) {
	return 1./(DK*DK) * (-1./12. * v_ll + 4./3. * v_l - 5./2. * v_c + 4./3. * v_r - 1./12. * v_rr);
}

__global__ void dns_du0dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
						int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
						/*order 0*/ double * __restrict__ irho_c,
						/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
						/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
						/*order 0*/ double * __restrict__ irhou0_c,
						/*order 1*/ double * __restrict__ irhou0_l, double * __restrict__ irhou0_r,
						/*order 2*/ double * __restrict__ irhou0_ll, double * __restrict__ irhou0_rr,
						int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
						/*order 0*/ double * __restrict__ odrhou0dx,
						/*order 0*/ double * __restrict__ odrhou0dy,
						/*order 0*/ double * __restrict__ odrhou0dz,
						/*order 0*/ double * __restrict__ odu0dx,
						/*order 0*/ double * __restrict__ odu0dy,
						/*order 0*/ double * __restrict__ odu0dz,
						/*order 0*/ double * __restrict__ odu0d2x,
						/*order 0*/ double * __restrict__ odu0d2y,
						/*order 0*/ double * __restrict__ odu0d2z,
						/*order 0*/ double * __restrict__ odrhou0u0dx,
						/*order 0*/ double * __restrict__ odu0dxdy,
						/*order 0*/ double * __restrict__ odu0dxdz) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);


		double rhou0_dy_ll, rhou0_dy_l, rhou0_dy_c, rhou0_dy_r, rhou0_dy_rr;
		rhou0_dy_ll = irhou0_c[dy_ll];
		rhou0_dy_l = irhou0_c[dy_l];
		rhou0_dy_c = irhou0_c[gidx];
		rhou0_dy_r = irhou0_c[dy_r];
		rhou0_dy_rr = irhou0_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u0_dy_ll, u0_dy_l, u0_dy_c, u0_dy_r, u0_dy_rr;
		u0_dy_ll = rhou0_dy_ll / rho_dy_ll;
		u0_dy_l = rhou0_dy_l / rho_dy_l;
		u0_dy_c = rhou0_dy_c / rho_dy_c;
		u0_dy_r = rhou0_dy_r / rho_dy_r;
		u0_dy_rr = rhou0_dy_rr / rho_dy_rr;
		
		odrhou0dy[gidx] = 1./DY * (1./12. * rhou0_dy_ll - 2./3. * rhou0_dy_l + 2./3. * rhou0_dy_r - 1./12. * rhou0_dy_rr);
		odu0dy[gidx] = 1./DY * (1./12. * u0_dy_ll - 2./3. * u0_dy_l + 2./3. * u0_dy_r - 1./12. * u0_dy_rr);
		odu0d2y[gidx] =1./(DY*DY) * (-1./12. * u0_dy_ll + 4./3. * u0_dy_l - 5./2. * u0_dy_c + 4./3. * u0_dy_r - 1./12. * u0_dy_rr);


		double rhou0_dz_ll, rhou0_dz_l, rhou0_dz_c, rhou0_dz_r, rhou0_dz_rr;
		rhou0_dz_ll = irhou0_c[dz_ll];
		rhou0_dz_l = irhou0_c[dz_l];
		rhou0_dz_c = irhou0_c[gidx];
		rhou0_dz_r = irhou0_c[dz_r];
		rhou0_dz_rr = irhou0_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u0_dz_ll, u0_dz_l, u0_dz_c, u0_dz_r, u0_dz_rr;
		u0_dz_ll = rhou0_dz_ll / rho_dz_ll;
		u0_dz_l = rhou0_dz_l / rho_dz_l;
		u0_dz_c = rhou0_dz_c / rho_dz_c;
		u0_dz_r = rhou0_dz_r / rho_dz_r;
		u0_dz_rr = rhou0_dz_rr / rho_dz_rr;
		
		odrhou0dz[gidx] = 1./DZ * (1./12. * rhou0_dz_ll - 2./3. * rhou0_dz_l + 2./3. * rhou0_dz_r - 1./12. * rhou0_dz_rr);
		odu0dz[gidx] = 1./DZ * (1./12. * u0_dz_ll - 2./3. * u0_dz_l + 2./3. * u0_dz_r - 1./12. * u0_dz_rr);
		odu0d2z[gidx] =1./(DZ*DZ) * (-1./12. * u0_dz_ll + 4./3. * u0_dz_l - 5./2. * u0_dz_c + 4./3. * u0_dz_r - 1./12. * u0_dz_rr);


		double rhou0_dx_ll, rhou0_dx_l, rhou0_dx_c, rhou0_dx_r, rhou0_dx_rr;
		rhou0_dx_ll = irhou0_ll[gidx];
		rhou0_dx_l = irhou0_l[gidx];
		rhou0_dx_c = irhou0_c[gidx];
		rhou0_dx_r = irhou0_r[gidx];
		rhou0_dx_rr = irhou0_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u0_dx_ll, u0_dx_l, u0_dx_c, u0_dx_r, u0_dx_rr;
		u0_dx_ll = rhou0_dx_ll / rho_dx_ll;
		u0_dx_l = rhou0_dx_l / rho_dx_l;
		u0_dx_c = rhou0_dx_c / rho_dx_c;
		u0_dx_r = rhou0_dx_r / rho_dx_r;
		u0_dx_rr = rhou0_dx_rr / rho_dx_rr;


		odrhou0dx[gidx] = 1./DX * (1./12. * sy_bc_ll *  rhou0_dx_ll - 2./3. * sy_bc_l * rhou0_dx_l + 2./3. * sy_bc_r * rhou0_dx_r - 1./12. * sy_bc_rr * rhou0_dx_rr);
		odu0dx[gidx] = 1./DX * (1./12. * sy_bc_ll *  u0_dx_ll - 2./3. * sy_bc_l * u0_dx_l + 2./3. * sy_bc_r * u0_dx_r - 1./12. * sy_bc_rr * u0_dx_rr);
		odu0d2x[gidx] = 1./(DX*DX) * (-1./12. * sy_bc_ll * u0_dx_ll + 4./3. * sy_bc_l * u0_dx_l - 5./2. * u0_dx_c + 4./3. * sy_bc_r * u0_dx_r - 1./12. * sy_bc_rr * u0_dx_rr); 
		
		odrhou0u0dx[gidx] = 1./DX * (1./12. * (sy_bc_ll * rhou0_dx_ll * sy_bc_ll * u0_dx_ll) - 2./3. * (sy_bc_l * rhou0_dx_l * sy_bc_l * u0_dx_l) 
											 + 2./3. * (sy_bc_r * rhou0_dx_r * sy_bc_r * u0_dx_r) - 1./12. * (sy_bc_rr * rhou0_dx_rr * sy_bc_rr * u0_dx_rr));

		
		double u0_dy_ll_dx_ll = irhou0_ll[dy_ll] / irho_ll[dy_ll];
		double u0_dy_l_dx_ll = irhou0_ll[dy_l] / irho_ll[dy_l];
		double u0_dy_r_dx_ll = irhou0_ll[dy_r] / irho_ll[dy_r];
		double u0_dy_rr_dx_ll = irhou0_ll[dy_rr] / irho_ll[dy_rr];

		double u0_dz_ll_dx_ll = irhou0_ll[dz_ll] / irho_ll[dz_ll];
		double u0_dz_l_dx_ll = irhou0_ll[dz_l] / irho_ll[dz_l];
		double u0_dz_r_dx_ll = irhou0_ll[dz_r] / irho_ll[dz_r];
		double u0_dz_rr_dx_ll = irhou0_ll[dz_rr] / irho_ll[dz_rr];

		double du0dy_dx_ll = 1./DY * (1./12. * u0_dy_ll_dx_ll - 2./3. * u0_dy_l_dx_ll + 2./3. * u0_dy_r_dx_ll - 1./12. * u0_dy_rr_dx_ll);
		double du0dz_dx_ll = 1./DZ * (1./12. * u0_dz_ll_dx_ll - 2./3. * u0_dz_l_dx_ll + 2./3. * u0_dz_r_dx_ll - 1./12. * u0_dz_rr_dx_ll);

		double u0_dy_ll_dx_l = irhou0_l[dy_ll] / irho_l[dy_ll];
		double u0_dy_l_dx_l = irhou0_l[dy_l] / irho_l[dy_l];
		double u0_dy_r_dx_l = irhou0_l[dy_r] / irho_l[dy_r];
		double u0_dy_rr_dx_l = irhou0_l[dy_rr] / irho_l[dy_rr];

		double u0_dz_ll_dx_l = irhou0_l[dz_ll] / irho_l[dz_ll];
		double u0_dz_l_dx_l = irhou0_l[dz_l] / irho_l[dz_l];
		double u0_dz_r_dx_l = irhou0_l[dz_r] / irho_l[dz_r];
		double u0_dz_rr_dx_l = irhou0_l[dz_rr] / irho_l[dz_rr];

		double du0dy_dx_l = 1./DY * (1./12. * u0_dy_ll_dx_l - 2./3. * u0_dy_l_dx_l + 2./3. * u0_dy_r_dx_l - 1./12. * u0_dy_rr_dx_l);
		double du0dz_dx_l = 1./DZ * (1./12. * u0_dz_ll_dx_l - 2./3. * u0_dz_l_dx_l + 2./3. * u0_dz_r_dx_l - 1./12. * u0_dz_rr_dx_l);

		double u0_dy_ll_dx_r = irhou0_r[dy_ll] / irho_r[dy_ll];
		double u0_dy_l_dx_r = irhou0_r[dy_l] / irho_r[dy_l];
		double u0_dy_r_dx_r = irhou0_r[dy_r] / irho_r[dy_r];
		double u0_dy_rr_dx_r = irhou0_r[dy_rr] / irho_r[dy_rr];

		double u0_dz_ll_dx_r = irhou0_r[dz_ll] / irho_r[dz_ll];
		double u0_dz_l_dx_r = irhou0_r[dz_l] / irho_r[dz_l];
		double u0_dz_r_dx_r = irhou0_r[dz_r] / irho_r[dz_r];
		double u0_dz_rr_dx_r = irhou0_r[dz_rr] / irho_r[dz_rr];

		double du0dy_dx_r = 1./DY * (1./12. * u0_dy_ll_dx_r - 2./3. * u0_dy_l_dx_r + 2./3. * u0_dy_r_dx_r - 1./12. * u0_dy_rr_dx_r);
		double du0dz_dx_r = 1./DZ * (1./12. * u0_dz_ll_dx_r - 2./3. * u0_dz_l_dx_r + 2./3. * u0_dz_r_dx_r - 1./12. * u0_dz_rr_dx_r);

		double u0_dy_ll_dx_rr = irhou0_rr[dy_ll] / irho_rr[dy_ll];
		double u0_dy_l_dx_rr = irhou0_rr[dy_l] / irho_rr[dy_l];
		double u0_dy_r_dx_rr = irhou0_rr[dy_r] / irho_rr[dy_r];
		double u0_dy_rr_dx_rr = irhou0_rr[dy_rr] / irho_rr[dy_rr];

		double u0_dz_ll_dx_rr = irhou0_rr[dz_ll] / irho_rr[dz_ll];
		double u0_dz_l_dx_rr = irhou0_rr[dz_l] / irho_rr[dz_l];
		double u0_dz_r_dx_rr = irhou0_rr[dz_r] / irho_rr[dz_r];
		double u0_dz_rr_dx_rr = irhou0_rr[dz_rr] / irho_rr[dz_rr];

		double du0dy_dx_rr = 1./DY * (1./12. * u0_dy_ll_dx_rr - 2./3. * u0_dy_l_dx_rr + 2./3. * u0_dy_r_dx_rr - 1./12. * u0_dy_rr_dx_rr);
		double du0dz_dx_rr = 1./DZ * (1./12. * u0_dz_ll_dx_rr - 2./3. * u0_dz_l_dx_rr + 2./3. * u0_dz_r_dx_rr - 1./12. * u0_dz_rr_dx_rr);

		odu0dxdy[gidx] = 1./DX * (1./12. * sy_bc_ll *  du0dy_dx_ll - 2./3. * sy_bc_l * du0dy_dx_l + 2./3. * sy_bc_r * du0dy_dx_r - 1./12. * sy_bc_rr * du0dy_dx_rr);
		odu0dxdz[gidx] = 1./DX * (1./12. * sy_bc_ll *  du0dz_dx_ll - 2./3. * sy_bc_l * du0dz_dx_l + 2./3. * sy_bc_r * du0dz_dx_r - 1./12. * sy_bc_rr * du0dz_dx_rr);
	}
}




__global__ void dns_du1dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
						int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
						/*order 0*/ double * __restrict__ irho_c,
						/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
						/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
						/*order 0*/ double * __restrict__ irhou1_c,
						/*order 1*/ double * __restrict__ irhou1_l, double * __restrict__ irhou1_r,
						/*order 2*/ double * __restrict__ irhou1_ll, double * __restrict__ irhou1_rr,
						/*order 0*/ double * __restrict__ odrhou1dx,
						/*order 0*/ double * __restrict__ odrhou1dy,
						/*order 0*/ double * __restrict__ odrhou1dz,
						/*order 0*/ double * __restrict__ odu1dx,
						/*order 0*/ double * __restrict__ odu1dy,
						/*order 0*/ double * __restrict__ odu1dz,
						/*order 0*/ double * __restrict__ odu1d2x,
						/*order 0*/ double * __restrict__ odu1d2y,
						/*order 0*/ double * __restrict__ odu1d2z,
						/*order 0*/ double * __restrict__ odrhou1u1dy,
						/*order 0*/ double * __restrict__ odu1dxdy,
						/*order 0*/ double * __restrict__ odu1dydz) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);


		double rhou1_dy_ll, rhou1_dy_l, rhou1_dy_c, rhou1_dy_r, rhou1_dy_rr;
		rhou1_dy_ll = irhou1_c[dy_ll];
		rhou1_dy_l = irhou1_c[dy_l];
		rhou1_dy_c = irhou1_c[gidx];
		rhou1_dy_r = irhou1_c[dy_r];
		rhou1_dy_rr = irhou1_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u1_dy_ll, u1_dy_l, u1_dy_c, u1_dy_r, u1_dy_rr;
		u1_dy_ll = rhou1_dy_ll / rho_dy_ll;
		u1_dy_l = rhou1_dy_l / rho_dy_l;
		u1_dy_c = rhou1_dy_c / rho_dy_c;
		u1_dy_r = rhou1_dy_r / rho_dy_r;
		u1_dy_rr = rhou1_dy_rr / rho_dy_rr;

		odrhou1dy[gidx] = 1./DY * (1./12. * rhou1_dy_ll - 2./3. * rhou1_dy_l + 2./3. * rhou1_dy_r - 1./12. * rhou1_dy_rr);
		odu1dy[gidx] = 1./DY * (1./12. * u1_dy_ll - 2./3. * u1_dy_l + 2./3. * u1_dy_r - 1./12. * u1_dy_rr);
		odu1d2y[gidx] =1./(DY*DY) * (-1./12. * u1_dy_ll + 4./3. * u1_dy_l - 5./2. * u1_dy_c + 4./3. * u1_dy_r - 1./12. * u1_dy_rr);

				
		odrhou1u1dy[gidx] = 1./DY * (1./12. * (rhou1_dy_ll * u1_dy_ll) - 2./3. * (rhou1_dy_l * u1_dy_l) 
											 + 2./3. * (rhou1_dy_r * u1_dy_r) - 1./12. * (rhou1_dy_rr * u1_dy_rr));


		double rhou1_dz_ll, rhou1_dz_l, rhou1_dz_c, rhou1_dz_r, rhou1_dz_rr;
		rhou1_dz_ll = irhou1_c[dz_ll];
		rhou1_dz_l = irhou1_c[dz_l];
		rhou1_dz_c = irhou1_c[gidx];
		rhou1_dz_r = irhou1_c[dz_r];
		rhou1_dz_rr = irhou1_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u1_dz_ll, u1_dz_l, u1_dz_c, u1_dz_r, u1_dz_rr;
		u1_dz_ll = rhou1_dz_ll / rho_dz_ll;
		u1_dz_l = rhou1_dz_l / rho_dz_l;
		u1_dz_c = rhou1_dz_c / rho_dz_c;
		u1_dz_r = rhou1_dz_r / rho_dz_r;
		u1_dz_rr = rhou1_dz_rr / rho_dz_rr;
		
		odrhou1dz[gidx] = 1./DZ * (1./12. * rhou1_dz_ll - 2./3. * rhou1_dz_l + 2./3. * rhou1_dz_r - 1./12. * rhou1_dz_rr);
		odu1dz[gidx] = 1./DZ * (1./12. * u1_dz_ll - 2./3. * u1_dz_l + 2./3. * u1_dz_r - 1./12. * u1_dz_rr);
		odu1d2z[gidx] =1./(DZ*DZ) * (-1./12. * u1_dz_ll + 4./3. * u1_dz_l - 5./2. * u1_dz_c + 4./3. * u1_dz_r - 1./12. * u1_dz_rr);


		double rhou1_dx_ll, rhou1_dx_l, rhou1_dx_c, rhou1_dx_r, rhou1_dx_rr;
		rhou1_dx_ll = irhou1_ll[gidx];
		rhou1_dx_l = irhou1_l[gidx];
		rhou1_dx_c = irhou1_c[gidx];
		rhou1_dx_r = irhou1_r[gidx];
		rhou1_dx_rr = irhou1_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u1_dx_ll, u1_dx_l, u1_dx_c, u1_dx_r, u1_dx_rr;
		u1_dx_ll = rhou1_dx_ll / rho_dx_ll;
		u1_dx_l = rhou1_dx_l / rho_dx_l;
		u1_dx_c = rhou1_dx_c / rho_dx_c;
		u1_dx_r = rhou1_dx_r / rho_dx_r;
		u1_dx_rr = rhou1_dx_rr / rho_dx_rr;


		odrhou1dx[gidx] = 1./DX * (1./12. *  rhou1_dx_ll - 2./3. * rhou1_dx_l + 2./3. * rhou1_dx_r - 1./12. * rhou1_dx_rr);
		odu1dx[gidx] = 1./DX * (1./12. *  u1_dx_ll - 2./3. * u1_dx_l + 2./3. * u1_dx_r - 1./12. * u1_dx_rr);
		odu1d2x[gidx] = 1./(DX*DX) * (-1./12. * u1_dx_ll + 4./3. * u1_dx_l - 5./2. * u1_dx_c + 4./3. * u1_dx_r - 1./12. * u1_dx_rr); 


		double u1_dx_ll_dy_ll = irhou1_ll[dy_ll] / irho_ll[dy_ll];
		double u1_dx_l_dy_ll = irhou1_l[dy_ll] / irho_l[dy_ll];
		double u1_dx_r_dy_ll = irhou1_r[dy_ll] / irho_r[dy_ll];
		double u1_dx_rr_dy_ll = irhou1_rr[dy_ll] / irho_r[dy_ll];

		double du1dx_dy_ll = 1./DX * (1./12. *  u1_dx_ll_dy_ll - 2./3. * u1_dx_l_dy_ll + 2./3. * u1_dx_r_dy_ll - 1./12. * u1_dx_rr_dy_ll);

		double u1_dx_ll_dy_l = irhou1_ll[dy_l] / irho_ll[dy_l];
		double u1_dx_l_dy_l = irhou1_l[dy_l] / irho_l[dy_l];
		double u1_dx_r_dy_l = irhou1_r[dy_l] / irho_r[dy_l];
		double u1_dx_rr_dy_l = irhou1_rr[dy_l] / irho_r[dy_l];

		double du1dx_dy_l = 1./DX * (1./12. *  u1_dx_ll_dy_l - 2./3. * u1_dx_l_dy_l + 2./3. * u1_dx_r_dy_l - 1./12. * u1_dx_rr_dy_l);

		double u1_dx_ll_dy_r = irhou1_ll[dy_r] / irho_ll[dy_r];
		double u1_dx_l_dy_r = irhou1_l[dy_r] / irho_l[dy_r];
		double u1_dx_r_dy_r = irhou1_r[dy_r] / irho_r[dy_r];
		double u1_dx_rr_dy_r = irhou1_rr[dy_r] / irho_r[dy_r];

		double du1dx_dy_r = 1./DX * (1./12. *  u1_dx_ll_dy_r - 2./3. * u1_dx_l_dy_r + 2./3. * u1_dx_r_dy_r - 1./12. * u1_dx_rr_dy_r);

		double u1_dx_ll_dy_rr = irhou1_ll[dy_rr] / irho_ll[dy_rr];
		double u1_dx_l_dy_rr = irhou1_l[dy_rr] / irho_l[dy_rr];
		double u1_dx_r_dy_rr = irhou1_r[dy_rr] / irho_r[dy_rr];
		double u1_dx_rr_dy_rr = irhou1_rr[dy_rr] / irho_r[dy_rr];

		double du1dx_dy_rr = 1./DX * (1./12. *  u1_dx_ll_dy_rr - 2./3. * u1_dx_l_dy_rr + 2./3. * u1_dx_r_dy_rr - 1./12. * u1_dx_rr_dy_rr);

		odu1dxdy[gidx] = 1./DY * (1./12. * du1dx_dy_ll - 2./3. * du1dx_dy_l + 2./3. * du1dx_dy_r - 1./12. * du1dx_dy_rr);


		// Calculate idx with periodic boundary condition
		int32_t dy_ll_dz_ll, dy_ll_dz_l, dy_ll_dz_r, dy_ll_dz_rr;
		IDX((NY+Y-2)%NY, (NZ+Z-2)%NZ, dy_ll_dz_ll, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z-1)%NZ, dy_ll_dz_l, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+1)%NZ, dy_ll_dz_r, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+2)%NZ, dy_ll_dz_rr, NZ);

		double u1_dz_ll_dy_ll = irhou1_ll[dy_ll_dz_ll] / irho_ll[dy_ll_dz_ll];
		double u1_dz_l_dy_ll = irhou1_ll[dy_ll_dz_l] / irho_ll[dy_ll_dz_l];
		double u1_dz_r_dy_ll = irhou1_ll[dy_ll_dz_r] / irho_ll[dy_ll_dz_r];
		double u1_dz_rr_dy_ll = irhou1_ll[dy_ll_dz_rr] / irho_ll[dy_ll_dz_rr];

		double du1dz_dy_ll = 1./DZ * (1./12. * u1_dz_ll_dy_ll - 2./3. * u1_dz_l_dy_ll + 2./3. * u1_dz_r_dy_ll - 1./12. * u1_dz_rr_dy_ll);


		int32_t dy_l_dz_ll, dy_l_dz_l, dy_l_dz_r, dy_l_dz_rr;
		IDX((NY+Y-1)%NY, (NZ+Z-2)%NZ, dy_l_dz_ll, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z-1)%NZ, dy_l_dz_l, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+1)%NZ, dy_l_dz_r, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+2)%NZ, dy_l_dz_rr, NZ);

		double u1_dz_ll_dy_l = irhou1_ll[dy_l_dz_ll] / irho_ll[dy_l_dz_ll];
		double u1_dz_l_dy_l = irhou1_ll[dy_l_dz_l] / irho_ll[dy_l_dz_l];
		double u1_dz_r_dy_l = irhou1_ll[dy_l_dz_r] / irho_ll[dy_l_dz_r];
		double u1_dz_rr_dy_l = irhou1_ll[dy_l_dz_rr] / irho_ll[dy_l_dz_rr];

		double du1dz_dy_l = 1./DZ * (1./12. * u1_dz_ll_dy_l - 2./3. * u1_dz_l_dy_l + 2./3. * u1_dz_r_dy_l - 1./12. * u1_dz_rr_dy_l);


		int32_t dy_r_dz_ll, dy_r_dz_l, dy_r_dz_r, dy_r_dz_rr;
		IDX((NY+Y+1)%NY, (NZ+Z-2)%NZ, dy_r_dz_ll, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z-1)%NZ, dy_r_dz_l, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+1)%NZ, dy_r_dz_r, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+2)%NZ, dy_r_dz_rr, NZ);

		double u1_dz_ll_dy_r = irhou1_ll[dy_r_dz_ll] / irho_ll[dy_r_dz_ll];
		double u1_dz_l_dy_r = irhou1_ll[dy_r_dz_l] / irho_ll[dy_r_dz_l];
		double u1_dz_r_dy_r = irhou1_ll[dy_r_dz_r] / irho_ll[dy_r_dz_r];
		double u1_dz_rr_dy_r = irhou1_ll[dy_r_dz_rr] / irho_ll[dy_r_dz_rr];

		double du1dz_dy_r = 1./DZ * (1./12. * u1_dz_ll_dy_r - 2./3. * u1_dz_l_dy_r + 2./3. * u1_dz_r_dy_r - 1./12. * u1_dz_rr_dy_r);


		int32_t dy_rr_dz_ll, dy_rr_dz_l, dy_rr_dz_r, dy_rr_dz_rr;
		IDX((NY+Y+2)%NY, (NZ+Z-2)%NZ, dy_rr_dz_ll, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z-1)%NZ, dy_rr_dz_l, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+1)%NZ, dy_rr_dz_r, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+2)%NZ, dy_rr_dz_rr, NZ);

		double u1_dz_ll_dy_rr = irhou1_ll[dy_rr_dz_ll] / irho_ll[dy_rr_dz_ll];
		double u1_dz_l_dy_rr = irhou1_ll[dy_rr_dz_l] / irho_ll[dy_rr_dz_l];
		double u1_dz_r_dy_rr = irhou1_ll[dy_rr_dz_r] / irho_ll[dy_rr_dz_r];
		double u1_dz_rr_dy_rr = irhou1_ll[dy_rr_dz_rr] / irho_ll[dy_rr_dz_rr];

		double du1dz_dy_rr = 1./DZ * (1./12. * u1_dz_ll_dy_rr - 2./3. * u1_dz_l_dy_rr + 2./3. * u1_dz_r_dy_rr - 1./12. * u1_dz_rr_dy_rr);

		odu1dydz[gidx] = 1./DY * (1./12. * du1dz_dy_ll - 2./3. * du1dz_dy_l + 2./3. * du1dz_dy_r - 1./12. * du1dz_dy_rr);

	}
}

__global__ void dns_du2dxyz(int32_t i_worker, int32_t order_in, int32_t order_out, int32_t problemsize,
						int32_t block_size_z, int32_t block_size_y, int32_t warp_size_z, int32_t warp_size_y,
						/*order 0*/ double * __restrict__ irho_c,
						/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
						/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
						/*order 0*/ double * __restrict__ irhou2_c,
						/*order 1*/ double * __restrict__ irhou2_l, double * __restrict__ irhou2_r,
						/*order 2*/ double * __restrict__ irhou2_ll, double * __restrict__ irhou2_rr,
						/*order 0*/ double * __restrict__ odrhou2dx,
						/*order 0*/ double * __restrict__ odrhou2dy,
						/*order 0*/ double * __restrict__ odrhou2dz,
						/*order 0*/ double * __restrict__ odu2dx,
						/*order 0*/ double * __restrict__ odu2dy,
						/*order 0*/ double * __restrict__ odu2dz,
						/*order 0*/ double * __restrict__ odu2d2x,
						/*order 0*/ double * __restrict__ odu2d2y,
						/*order 0*/ double * __restrict__ odu2d2z,
						/*order 0*/ double * __restrict__ odrhou2u2dz,
						/*order 0*/ double * __restrict__ odu2dxdz,
						/*order 0*/ double * __restrict__ odu2dydz) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);


		double rhou2_dy_ll, rhou2_dy_l, rhou2_dy_c, rhou2_dy_r, rhou2_dy_rr;
		rhou2_dy_ll = irhou2_c[dy_ll];
		rhou2_dy_l = irhou2_c[dy_l];
		rhou2_dy_c = irhou2_c[gidx];
		rhou2_dy_r = irhou2_c[dy_r];
		rhou2_dy_rr = irhou2_c[dy_rr];

		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double u2_dy_ll, u2_dy_l, u2_dy_c, u2_dy_r, u2_dy_rr;
		u2_dy_ll = rhou2_dy_ll / rho_dy_ll;
		u2_dy_l = rhou2_dy_l / rho_dy_l;
		u2_dy_c = rhou2_dy_c / rho_dy_c;
		u2_dy_r = rhou2_dy_r / rho_dy_r;
		u2_dy_rr = rhou2_dy_rr / rho_dy_rr;

		odrhou2dy[gidx] = 1./DY * (1./12. * rhou2_dy_ll - 2./3. * rhou2_dy_l + 2./3. * rhou2_dy_r - 1./12. * rhou2_dy_rr);
		odu2dy[gidx] = 1./DY * (1./12. * u2_dy_ll - 2./3. * u2_dy_l + 2./3. * u2_dy_r - 1./12. * u2_dy_rr);
		odu2d2y[gidx] =1./(DY*DY) * (-1./12. * u2_dy_ll + 4./3. * u2_dy_l - 5./2. * u2_dy_c + 4./3. * u2_dy_r - 1./12. * u2_dy_rr);


		double rhou2_dz_ll, rhou2_dz_l, rhou2_dz_c, rhou2_dz_r, rhou2_dz_rr;
		rhou2_dz_ll = irhou2_c[dz_ll];
		rhou2_dz_l = irhou2_c[dz_l];
		rhou2_dz_c = irhou2_c[gidx];
		rhou2_dz_r = irhou2_c[dz_r];
		rhou2_dz_rr = irhou2_c[dz_rr];

		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double u2_dz_ll, u2_dz_l, u2_dz_c, u2_dz_r, u2_dz_rr;
		u2_dz_ll = rhou2_dz_ll / rho_dz_ll;
		u2_dz_l = rhou2_dz_l / rho_dz_l;
		u2_dz_c = rhou2_dz_c / rho_dz_c;
		u2_dz_r = rhou2_dz_r / rho_dz_r;
		u2_dz_rr = rhou2_dz_rr / rho_dz_rr;
		
		odrhou2dz[gidx] = 1./DZ * (1./12. * rhou2_dz_ll - 2./3. * rhou2_dz_l + 2./3. * rhou2_dz_r - 1./12. * rhou2_dz_rr);
		odu2dz[gidx] = 1./DZ * (1./12. * u2_dz_ll - 2./3. * u2_dz_l + 2./3. * u2_dz_r - 1./12. * u2_dz_rr);
		odu2d2z[gidx] =1./(DZ*DZ) * (-1./12. * u2_dz_ll + 4./3. * u2_dz_l - 5./2. * u2_dz_c + 4./3. * u2_dz_r - 1./12. * u2_dz_rr);
		odrhou2u2dz[gidx] = 1./DZ * (1./12. * (rhou2_dz_ll * u2_dz_ll) - 2./3. * (rhou2_dz_l * u2_dz_l) 
											 + 2./3. * (rhou2_dz_r * u2_dz_r) - 1./12. * (rhou2_dz_rr * u2_dz_rr));


		double rhou2_dx_ll, rhou2_dx_l, rhou2_dx_c, rhou2_dx_r, rhou2_dx_rr;
		rhou2_dx_ll = irhou2_ll[gidx];
		rhou2_dx_l = irhou2_l[gidx];
		rhou2_dx_c = irhou2_c[gidx];
		rhou2_dx_r = irhou2_r[gidx];
		rhou2_dx_rr = irhou2_rr[gidx];

		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double u2_dx_ll, u2_dx_l, u2_dx_c, u2_dx_r, u2_dx_rr;
		u2_dx_ll = rhou2_dx_ll / rho_dx_ll;
		u2_dx_l = rhou2_dx_l / rho_dx_l;
		u2_dx_c = rhou2_dx_c / rho_dx_c;
		u2_dx_r = rhou2_dx_r / rho_dx_r;
		u2_dx_rr = rhou2_dx_rr / rho_dx_rr;


		odrhou2dx[gidx] = 1./DX * (1./12. *  rhou2_dx_ll - 2./3. * rhou2_dx_l + 2./3. * rhou2_dx_r - 1./12. * rhou2_dx_rr);
		odu2dx[gidx] = 1./DX * (1./12. *  u2_dx_ll - 2./3. * u2_dx_l + 2./3. * u2_dx_r - 1./12. * u2_dx_rr);
		odu2d2x[gidx] = 1./(DX*DX) * (-1./12. * u2_dx_ll + 4./3. * u2_dx_l - 5./2. * u2_dx_c + 4./3. * u2_dx_r - 1./12. * u2_dx_rr); 


		double u2_dx_ll_dz_ll = irhou2_ll[dz_ll] / irho_ll[dz_ll];
		double u2_dx_l_dz_ll = irhou2_l[dz_ll] / irho_l[dz_ll];
		double u2_dx_r_dz_ll = irhou2_r[dz_ll] / irho_r[dz_ll];
		double u2_dx_rr_dz_ll = irhou2_rr[dz_ll] / irho_r[dz_ll];

		double du2dx_dz_ll = 1./DX * (1./12. *  u2_dx_ll_dz_ll - 2./3. * u2_dx_l_dz_ll + 2./3. * u2_dx_r_dz_ll - 1./12. * u2_dx_rr_dz_ll);

		double u2_dx_ll_dz_l = irhou2_ll[dz_l] / irho_ll[dz_l];
		double u2_dx_l_dz_l = irhou2_l[dz_l] / irho_l[dz_l];
		double u2_dx_r_dz_l = irhou2_r[dz_l] / irho_r[dz_l];
		double u2_dx_rr_dz_l = irhou2_rr[dz_l] / irho_r[dz_l];

		double du2dx_dz_l = 1./DX * (1./12. *  u2_dx_ll_dz_l - 2./3. * u2_dx_l_dz_l + 2./3. * u2_dx_r_dz_l - 1./12. * u2_dx_rr_dz_l);

		double u2_dx_ll_dz_r = irhou2_ll[dz_r] / irho_ll[dz_r];
		double u2_dx_l_dz_r = irhou2_l[dz_r] / irho_l[dz_r];
		double u2_dx_r_dz_r = irhou2_r[dz_r] / irho_r[dz_r];
		double u2_dx_rr_dz_r = irhou2_rr[dz_r] / irho_r[dz_r];

		double du2dx_dz_r = 1./DX * (1./12. *  u2_dx_ll_dz_r - 2./3. * u2_dx_l_dz_r + 2./3. * u2_dx_r_dz_r - 1./12. * u2_dx_rr_dz_r);

		double u2_dx_ll_dz_rr = irhou2_ll[dz_rr] / irho_ll[dz_rr];
		double u2_dx_l_dz_rr = irhou2_l[dz_rr] / irho_l[dz_rr];
		double u2_dx_r_dz_rr = irhou2_r[dz_rr] / irho_r[dz_rr];
		double u2_dx_rr_dz_rr = irhou2_rr[dz_rr] / irho_r[dz_rr];

		double du2dx_dz_rr = 1./DX * (1./12. *  u2_dx_ll_dz_rr - 2./3. * u2_dx_l_dz_rr + 2./3. * u2_dx_r_dz_rr - 1./12. * u2_dx_rr_dz_rr);

		odu2dxdz[gidx] = 1./DZ * (1./12. * du2dx_dz_ll - 2./3. * du2dx_dz_l + 2./3. * du2dx_dz_r - 1./12. * du2dx_dz_rr);


		// Calculate idx with periodic boundary condition
		int32_t dy_ll_dz_ll, dy_ll_dz_l, dy_ll_dz_r, dy_ll_dz_rr;
		IDX((NY+Y-2)%NY, (NZ+Z-2)%NZ, dy_ll_dz_ll, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z-1)%NZ, dy_ll_dz_l, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+1)%NZ, dy_ll_dz_r, NZ);
		IDX((NY+Y-2)%NY, (NZ+Z+2)%NZ, dy_ll_dz_rr, NZ);

		double u2_dz_ll_dy_ll = irhou2_ll[dy_ll_dz_ll] / irho_ll[dy_ll_dz_ll];
		double u2_dz_l_dy_ll = irhou2_ll[dy_ll_dz_l] / irho_ll[dy_ll_dz_l];
		double u2_dz_r_dy_ll = irhou2_ll[dy_ll_dz_r] / irho_ll[dy_ll_dz_r];
		double u2_dz_rr_dy_ll = irhou2_ll[dy_ll_dz_rr] / irho_ll[dy_ll_dz_rr];

		double du2dz_dy_ll = 1./DZ * (1./12. * u2_dz_ll_dy_ll - 2./3. * u2_dz_l_dy_ll + 2./3. * u2_dz_r_dy_ll - 1./12. * u2_dz_rr_dy_ll);


		int32_t dy_l_dz_ll, dy_l_dz_l, dy_l_dz_r, dy_l_dz_rr;
		IDX((NY+Y-1)%NY, (NZ+Z-2)%NZ, dy_l_dz_ll, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z-1)%NZ, dy_l_dz_l, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+1)%NZ, dy_l_dz_r, NZ);
		IDX((NY+Y-1)%NY, (NZ+Z+2)%NZ, dy_l_dz_rr, NZ);

		double u2_dz_ll_dy_l = irhou2_ll[dy_l_dz_ll] / irho_ll[dy_l_dz_ll];
		double u2_dz_l_dy_l = irhou2_ll[dy_l_dz_l] / irho_ll[dy_l_dz_l];
		double u2_dz_r_dy_l = irhou2_ll[dy_l_dz_r] / irho_ll[dy_l_dz_r];
		double u2_dz_rr_dy_l = irhou2_ll[dy_l_dz_rr] / irho_ll[dy_l_dz_rr];

		double du2dz_dy_l = 1./DZ * (1./12. * u2_dz_ll_dy_l - 2./3. * u2_dz_l_dy_l + 2./3. * u2_dz_r_dy_l - 1./12. * u2_dz_rr_dy_l);


		int32_t dy_r_dz_ll, dy_r_dz_l, dy_r_dz_r, dy_r_dz_rr;
		IDX((NY+Y+1)%NY, (NZ+Z-2)%NZ, dy_r_dz_ll, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z-1)%NZ, dy_r_dz_l, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+1)%NZ, dy_r_dz_r, NZ);
		IDX((NY+Y+1)%NY, (NZ+Z+2)%NZ, dy_r_dz_rr, NZ);

		double u2_dz_ll_dy_r = irhou2_ll[dy_r_dz_ll] / irho_ll[dy_r_dz_ll];
		double u2_dz_l_dy_r = irhou2_ll[dy_r_dz_l] / irho_ll[dy_r_dz_l];
		double u2_dz_r_dy_r = irhou2_ll[dy_r_dz_r] / irho_ll[dy_r_dz_r];
		double u2_dz_rr_dy_r = irhou2_ll[dy_r_dz_rr] / irho_ll[dy_r_dz_rr];

		double du2dz_dy_r = 1./DZ * (1./12. * u2_dz_ll_dy_r - 2./3. * u2_dz_l_dy_r + 2./3. * u2_dz_r_dy_r - 1./12. * u2_dz_rr_dy_r);


		int32_t dy_rr_dz_ll, dy_rr_dz_l, dy_rr_dz_r, dy_rr_dz_rr;
		IDX((NY+Y+2)%NY, (NZ+Z-2)%NZ, dy_rr_dz_ll, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z-1)%NZ, dy_rr_dz_l, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+1)%NZ, dy_rr_dz_r, NZ);
		IDX((NY+Y+2)%NY, (NZ+Z+2)%NZ, dy_rr_dz_rr, NZ);

		double u2_dz_ll_dy_rr = irhou2_ll[dy_rr_dz_ll] / irho_ll[dy_rr_dz_ll];
		double u2_dz_l_dy_rr = irhou2_ll[dy_rr_dz_l] / irho_ll[dy_rr_dz_l];
		double u2_dz_r_dy_rr = irhou2_ll[dy_rr_dz_r] / irho_ll[dy_rr_dz_r];
		double u2_dz_rr_dy_rr = irhou2_ll[dy_rr_dz_rr] / irho_ll[dy_rr_dz_rr];

		double du2dz_dy_rr = 1./DZ * (1./12. * u2_dz_ll_dy_rr - 2./3. * u2_dz_l_dy_rr + 2./3. * u2_dz_r_dy_rr - 1./12. * u2_dz_rr_dy_rr);

		odu2dydz[gidx] = 1./DY * (1./12. * du2dz_dy_ll - 2./3. * du2dz_dy_l + 2./3. * du2dz_dy_r - 1./12. * du2dz_dy_rr);

	}
}

__device__ double calp(double irhoE, double irho, double irhou0, double irhou1, double irhou2) {
	//return ((GAMA - 1) * (rhoE - 0.5 * rhou0 * rhou0 / rho - 0.5 * rhou1 * rhou1 / rho - 0.5 rhou2 * rhou2 / rho));
	double tmp = (irhoE - 0.5 * irhou0 * irhou0 / irho - 0.5 * irhou1 * irhou1 / irho - 0.5 * irhou2 * irhou2 / irho);
	return (GAMA - 1) * tmp;
}

__device__ double calT(double ip, double irho) {
	return MINF * MINF * GAMA * ip / irho;
}

__global__ void dns_drhoETpdxyz(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ irho_c,
						/*order 1*/ double * __restrict__ irho_l, double * __restrict__ irho_r,
						/*order 2*/ double * __restrict__ irho_ll, double * __restrict__ irho_rr,
						/*order 0*/ double * __restrict__ irhou0_c,
						/*order 1*/ double * __restrict__ irhou0_l, double * __restrict__ irhou0_r,
						/*order 2*/ double * __restrict__ irhou0_ll, double * __restrict__ irhou0_rr,
						/*order 0*/ double * __restrict__ irhou1_c,
						/*order 1*/ double * __restrict__ irhou1_l, double * __restrict__ irhou1_r,
						/*order 2*/ double * __restrict__ irhou1_ll, double * __restrict__ irhou1_rr,
						/*order 0*/ double * __restrict__ irhou2_c,
						/*order 1*/ double * __restrict__ irhou2_l, double * __restrict__ irhou2_r,
						/*order 2*/ double * __restrict__ irhou2_ll, double * __restrict__ irhou2_rr,
						/*order 0*/ double * __restrict__ irhoE_c,
						/*order 1*/ double * __restrict__ irhoE_l, double * __restrict__ irhoE_r,
						/*order 2*/ double * __restrict__ irhoE_ll, double * __restrict__ irhoE_rr,
						int sy_bc_ll, int sy_bc_l, int sy_bc_r, int sy_bc_rr,
						/*order 0*/ double * __restrict__ odrhodx,
						/*order 0*/ double * __restrict__ odrhody,
						/*order 0*/ double * __restrict__ odrhodz,
						/*order 0*/ double * __restrict__ odrhoEdx,
						/*order 0*/ double * __restrict__ odrhoEdy,
						/*order 0*/ double * __restrict__ odrhoEdz,
						/*order 0*/ double * __restrict__ odrhoEu0dx,
						/*order 0*/ double * __restrict__ odrhoEu1dy,
						/*order 0*/ double * __restrict__ odrhoEu2dz,
						/*order 0*/ double * __restrict__ odTd2x,
						/*order 0*/ double * __restrict__ odTd2y,
						/*order 0*/ double * __restrict__ odTd2z,
						/*order 0*/ double * __restrict__ odpdx,
						/*order 0*/ double * __restrict__ odpdy,
						/*order 0*/ double * __restrict__ odpdz,
						/*order 0*/ double * __restrict__ odpu0dx,
						/*order 0*/ double * __restrict__ odpu1dy,
						/*order 0*/ double * __restrict__ odpu2dz) {
	// Thread Idx
	int32_t tidx = blockIdx.x*blockDim.x+threadIdx.x;
	// Global Idx
	int32_t gidx;
	// Idx in Block
	int32_t col_in_block, row_in_block;
	// Global Idx
	//gidx = thread_to_global_idx(problemsize, tidx, block_size_z, block_size_y, warp_size_z, warp_size_y, &col_in_block, &row_in_block);
	gidx = thread_to_global_idx(my_n_part, tidx, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y, &col_in_block, &row_in_block);


	if (gidx<block_ncc) {
		//dfdx[gidx] = tidx;

		int32_t Y, Z;
		int32_t dy_ll, dy_l, dy_r, dy_rr;
		int32_t dz_ll, dz_l, dz_r, dz_rr;
		COORDS(gidx, Y, Z, NZ);
		
		// Calculate idx with periodic boundary condition
		IDX((NY+Y-2)%NY, Z, dy_ll, NZ);
		IDX((NY+Y-1)%NY, Z, dy_l, NZ);
		IDX((NY+Y+1)%NY, Z, dy_r, NZ);
		IDX((NY+Y+2)%NY, Z, dy_rr, NZ);

		// Calculate idx with periodic boundary condition
		IDX(Y, (NZ+Z-2)%NZ, dz_ll, NZ);
		IDX(Y, (NZ+Z-1)%NZ, dz_l, NZ);
		IDX(Y, (NZ+Z+1)%NZ, dz_r, NZ);
		IDX(Y, (NZ+Z+2)%NZ, dz_rr, NZ);


		double rho_dy_ll, rho_dy_l, rho_dy_c, rho_dy_r, rho_dy_rr;
		rho_dy_ll = irho_c[dy_ll];
		rho_dy_l = irho_c[dy_l];
		rho_dy_c = irho_c[gidx];
		rho_dy_r = irho_c[dy_r];
		rho_dy_rr = irho_c[dy_rr];

		double rhou0_dy_ll, rhou0_dy_l, rhou0_dy_c, rhou0_dy_r, rhou0_dy_rr;
		rhou0_dy_ll = irhou0_c[dy_ll];
		rhou0_dy_l = irhou0_c[dy_l];
		rhou0_dy_c = irhou0_c[gidx];
		rhou0_dy_r = irhou0_c[dy_r];
		rhou0_dy_rr = irhou0_c[dy_rr];

		double rhou1_dy_ll, rhou1_dy_l, rhou1_dy_c, rhou1_dy_r, rhou1_dy_rr;
		rhou1_dy_ll = irhou1_c[dy_ll];
		rhou1_dy_l = irhou1_c[dy_l];
		rhou1_dy_c = irhou1_c[gidx];
		rhou1_dy_r = irhou1_c[dy_r];
		rhou1_dy_rr = irhou1_c[dy_rr];

		double rhou2_dy_ll, rhou2_dy_l, rhou2_dy_c, rhou2_dy_r, rhou2_dy_rr;
		rhou2_dy_ll = irhou2_c[dy_ll];
		rhou2_dy_l = irhou2_c[dy_l];
		rhou2_dy_c = irhou2_c[gidx];
		rhou2_dy_r = irhou2_c[dy_r];
		rhou2_dy_rr = irhou2_c[dy_rr];

		double rhoE_dy_ll, rhoE_dy_l, rhoE_dy_c, rhoE_dy_r, rhoE_dy_rr;
		rhoE_dy_ll = irhoE_c[dy_ll];
		rhoE_dy_l = irhoE_c[dy_l];
		rhoE_dy_c = irhoE_c[gidx];
		rhoE_dy_r = irhoE_c[dy_r];
		rhoE_dy_rr = irhoE_c[dy_rr];

		double p_dy_ll, p_dy_l, p_dy_c, p_dy_r, p_dy_rr;
		p_dy_ll = calp(rhoE_dy_ll, rho_dy_ll, rhou0_dy_ll, rhou1_dy_ll, rhou2_dy_ll);
		p_dy_l = calp(rhoE_dy_l, rho_dy_l, rhou0_dy_l, rhou1_dy_l, rhou2_dy_l);
		p_dy_c = calp(rhoE_dy_c, rho_dy_c, rhou0_dy_c, rhou1_dy_c, rhou2_dy_c);
		p_dy_r = calp(rhoE_dy_r, rho_dy_r, rhou0_dy_r, rhou1_dy_r, rhou2_dy_r);
		p_dy_rr = calp(rhoE_dy_rr, rho_dy_rr, rhou0_dy_rr, rhou1_dy_rr, rhou2_dy_rr);

		odrhody[gidx] = 1./DY * (1./12. * rho_dy_ll - 2./3. * rho_dy_l + 2./3. * rho_dy_r - 1./12. * rho_dy_rr);
		odpdy[gidx] = 1./DY * (1./12. * p_dy_ll - 2./3. * p_dy_l + 2./3. * p_dy_r - 1./12. * p_dy_rr);
		odpu1dy[gidx] = 1./DY * (1./12. * p_dy_ll * rhou1_dy_ll / rho_dy_ll 
														- 2./3. * p_dy_l * rhou1_dy_l / rho_dy_l 
														+ 2./3. * p_dy_r * rhou1_dy_r / rho_dy_r 
														- 1./12. * p_dy_rr * rhou1_dy_rr / rho_dy_rr);
		odrhoEdy[gidx] = 1./DY * (1./12. * rhoE_dy_ll - 2./3. * rhoE_dy_l + 2./3. * rhoE_dy_r - 1./12. * rhoE_dy_rr);
		odrhoEu1dy[gidx] = 1./DY * (1./12. * rhoE_dy_ll * rhou1_dy_ll / rho_dy_ll 
														- 2./3. * rhoE_dy_l * rhou1_dy_l / rho_dy_l 
														+ 2./3. * rhoE_dy_r * rhou1_dy_r / rho_dy_r 
														- 1./12. * rhoE_dy_rr * rhou1_dy_rr / rho_dy_rr);
		odTd2y[gidx] = 1./(DY*DY) * (-1./12. * calT(p_dy_ll, rho_dy_ll) 
																+ 4./3. * calT(p_dy_l, rho_dy_l) 
																- 5./2. * calT(p_dy_c, rho_dy_c) 
																+ 4./3. * calT(p_dy_r, rho_dy_r)
																- 1./12. * calT(p_dy_rr, rho_dy_rr));


		double rho_dz_ll, rho_dz_l, rho_dz_c, rho_dz_r, rho_dz_rr;
		rho_dz_ll = irho_c[dz_ll];
		rho_dz_l = irho_c[dz_l];
		rho_dz_c = irho_c[gidx];
		rho_dz_r = irho_c[dz_r];
		rho_dz_rr = irho_c[dz_rr];

		double rhou0_dz_ll, rhou0_dz_l, rhou0_dz_c, rhou0_dz_r, rhou0_dz_rr;
		rhou0_dz_ll = irhou0_c[dz_ll];
		rhou0_dz_l = irhou0_c[dz_l];
		rhou0_dz_c = irhou0_c[gidx];
		rhou0_dz_r = irhou0_c[dz_r];
		rhou0_dz_rr = irhou0_c[dz_rr];

		double rhou1_dz_ll, rhou1_dz_l, rhou1_dz_c, rhou1_dz_r, rhou1_dz_rr;
		rhou1_dz_ll = irhou1_c[dz_ll];
		rhou1_dz_l = irhou1_c[dz_l];
		rhou1_dz_c = irhou1_c[gidx];
		rhou1_dz_r = irhou1_c[dz_r];
		rhou1_dz_rr = irhou1_c[dz_rr];

		double rhou2_dz_ll, rhou2_dz_l, rhou2_dz_c, rhou2_dz_r, rhou2_dz_rr;
		rhou2_dz_ll = irhou2_c[dz_ll];
		rhou2_dz_l = irhou2_c[dz_l];
		rhou2_dz_c = irhou2_c[gidx];
		rhou2_dz_r = irhou2_c[dz_r];
		rhou2_dz_rr = irhou2_c[dz_rr];

		double rhoE_dz_ll, rhoE_dz_l, rhoE_dz_c, rhoE_dz_r, rhoE_dz_rr;
		rhoE_dz_ll = irhoE_c[dz_ll];
		rhoE_dz_l = irhoE_c[dz_l];
		rhoE_dz_c = irhoE_c[gidx];
		rhoE_dz_r = irhoE_c[dz_r];
		rhoE_dz_rr = irhoE_c[dz_rr];

		double p_dz_ll, p_dz_l, p_dz_c, p_dz_r, p_dz_rr;
		p_dz_ll = calp(rhoE_dz_ll, rho_dz_ll, rhou0_dz_ll, rhou1_dz_ll, rhou2_dz_ll);
		p_dz_l = calp(rhoE_dz_l, rho_dz_l, rhou0_dz_l, rhou1_dz_l, rhou2_dz_l);
		p_dz_c = calp(rhoE_dz_c, rho_dz_c, rhou0_dz_c, rhou1_dz_c, rhou2_dz_c);
		p_dz_r = calp(rhoE_dz_r, rho_dz_r, rhou0_dz_r, rhou1_dz_r, rhou2_dz_r);
		p_dz_rr = calp(rhoE_dz_rr, rho_dz_rr, rhou0_dz_rr, rhou1_dz_rr, rhou2_dz_rr);
		
		odrhodz[gidx] = 1./DZ * (1./12. * rho_dz_ll - 2./3. * rho_dz_l + 2./3. * rho_dz_r - 1./12. * rho_dz_rr);
		odpdz[gidx] = 1./DZ * (1./12. * p_dz_ll - 2./3. * p_dz_l + 2./3. * p_dz_r - 1./12. * p_dz_rr);
		odpu2dz[gidx] = 1./DZ * (1./12. * p_dz_ll * rhou2_dz_ll / rho_dz_ll 
												- 2./3. * p_dz_l * rhou2_dz_l / rho_dz_l 
												+ 2./3. * p_dz_r * rhou2_dz_r / rho_dz_r 
												- 1./12. * p_dz_rr * rhou2_dz_rr / rho_dz_rr);
		odrhoEdz[gidx] = 1./DZ * (1./12. * rhoE_dz_ll - 2./3. * rhoE_dz_l + 2./3. * rhoE_dz_r - 1./12. * rhoE_dz_rr);
		odrhoEu2dz[gidx] = 1./DZ * (1./12. * rhoE_dz_ll * rhou2_dz_ll / rho_dz_ll 
													- 2./3. * rhoE_dz_l * rhou2_dz_l / rho_dz_l 
													+ 2./3. * rhoE_dz_r * rhou2_dz_r / rho_dz_r 
													- 1./12. * rhoE_dz_rr * rhou2_dz_rr / rho_dz_rr);
		odTd2z[gidx] = 1./(DZ*DZ) * (-1./12. * calT(p_dz_ll, rho_dz_ll) 
														+ 4./3. * calT(p_dz_l, rho_dz_l) 
														- 5./2. * calT(p_dz_c, rho_dz_c) 
														+ 4./3. * calT(p_dz_r, rho_dz_r)
														- 1./12. * calT(p_dz_rr, rho_dz_rr));


		double rho_dx_ll, rho_dx_l, rho_dx_c, rho_dx_r, rho_dx_rr;
		rho_dx_ll = irho_ll[gidx];
		rho_dx_l = irho_l[gidx];
		rho_dx_c = irho_c[gidx];
		rho_dx_r = irho_r[gidx];
		rho_dx_rr = irho_rr[gidx];

		double rhou0_dx_ll, rhou0_dx_l, rhou0_dx_c, rhou0_dx_r, rhou0_dx_rr;
		rhou0_dx_ll = irhou0_ll[gidx];
		rhou0_dx_l = irhou0_l[gidx];
		rhou0_dx_c = irhou0_c[gidx];
		rhou0_dx_r = irhou0_r[gidx];
		rhou0_dx_rr = irhou0_rr[gidx];

		double rhou1_dx_ll, rhou1_dx_l, rhou1_dx_c, rhou1_dx_r, rhou1_dx_rr;
		rhou1_dx_ll = irhou1_ll[gidx];
		rhou1_dx_l = irhou1_l[gidx];
		rhou1_dx_c = irhou1_c[gidx];
		rhou1_dx_r = irhou1_r[gidx];
		rhou1_dx_rr = irhou1_rr[gidx];

		double rhou2_dx_ll, rhou2_dx_l, rhou2_dx_c, rhou2_dx_r, rhou2_dx_rr;
		rhou2_dx_ll = irhou2_ll[gidx];
		rhou2_dx_l = irhou2_l[gidx];
		rhou2_dx_c = irhou2_c[gidx];
		rhou2_dx_r = irhou2_r[gidx];
		rhou2_dx_rr = irhou2_rr[gidx];

		double rhoE_dx_ll, rhoE_dx_l, rhoE_dx_c, rhoE_dx_r, rhoE_dx_rr;
		rhoE_dx_ll = irhoE_ll[gidx];
		rhoE_dx_l = irhoE_l[gidx];
		rhoE_dx_c = irhoE_c[gidx];
		rhoE_dx_r = irhoE_r[gidx];
		rhoE_dx_rr = irhoE_rr[gidx];

		double p_dx_ll, p_dx_l, p_dx_c, p_dx_r, p_dx_rr;
		p_dx_ll = calp(rhoE_dx_ll, rho_dx_ll, rhou0_dx_ll, rhou1_dx_ll, rhou2_dx_ll);
		p_dx_l = calp(rhoE_dx_l, rho_dx_l, rhou0_dx_l, rhou1_dx_l, rhou2_dx_l);
		p_dx_c = calp(rhoE_dx_c, rho_dx_c, rhou0_dx_c, rhou1_dx_c, rhou2_dx_c);
		p_dx_r = calp(rhoE_dx_r, rho_dx_r, rhou0_dx_r, rhou1_dx_r, rhou2_dx_r);
		p_dx_rr = calp(rhoE_dx_rr, rho_dx_rr, rhou0_dx_rr, rhou1_dx_rr, rhou2_dx_rr);

		odrhodx[gidx] = 1./DX * (1./12. * rho_dx_ll - 2./3. * rho_dx_l + 2./3. * rho_dx_r - 1./12. * rho_dx_rr);
		odpdx[gidx] = 1./DX * (1./12. * p_dx_ll - 2./3. * p_dx_l + 2./3. * p_dx_r - 1./12. * p_dx_rr);
		odpu0dx[gidx] = 1./DX * (1./12. * (p_dx_ll * sy_bc_ll * rhou0_dx_ll / rho_dx_ll) - 2./3. * (p_dx_l * sy_bc_l * rhou0_dx_l / rho_dx_l) 
											 + 2./3. * (p_dx_r * sy_bc_r * rhou0_dx_r / rho_dx_r) - 1./12. * (p_dx_rr * sy_bc_rr * rhou0_dx_rr / rho_dx_rr));
		odrhoEdx[gidx] = 1./DX * (1./12. * rhoE_dx_ll - 2./3. * rhoE_dx_l + 2./3. * rhoE_dx_r - 1./12. * rhoE_dx_rr);
		odrhoEu0dx[gidx] = 1./DX * (1./12. * (rhoE_dx_ll * sy_bc_ll * rhou0_dx_ll / rho_dx_ll) - 2./3. * (rhoE_dx_l * sy_bc_l * rhou0_dx_l / rho_dx_l) 
											 + 2./3. * (rhoE_dx_r * sy_bc_r * rhou0_dx_r / rho_dx_r) - 1./12. * (rhoE_dx_rr * sy_bc_rr * rhou0_dx_rr / rho_dx_rr));
		odTd2z[gidx] = 1./(DX*DX) * (-1./12. * calT(p_dx_ll, rho_dx_ll) 
														+ 4./3. * calT(p_dx_l, rho_dx_l) 
														- 5./2. * calT(p_dx_c, rho_dx_c) 
														+ 4./3. * calT(p_dx_r, rho_dx_r)
														- 1./12. * calT(p_dx_rr, rho_dx_rr));

	}

}


__global__ void dns_Res_v1(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ rho,
						/*order 0*/ double * __restrict__ u0,
						/*order 0*/ double * __restrict__ u1,
						/*order 0*/ double * __restrict__ u2,
						/*order 0*/ double * __restrict__ rhou0,
						/*order 0*/ double * __restrict__ rhou1,
						/*order 0*/ double * __restrict__ rhou2,
						/*order 0*/ double * __restrict__ rhoE,
						/*order 0*/ double * __restrict__ p,
						/*order 0*/ double * __restrict__ T,

						/*order 0*/ double * __restrict__ du0dx,
						/*order 0*/ double * __restrict__ du0dy,
						/*order 0*/ double * __restrict__ du0dz,

						/*order 0*/ double * __restrict__ du1dx,
						/*order 0*/ double * __restrict__ du1dy,
						/*order 0*/ double * __restrict__ du1dz,

						/*order 0*/ double * __restrict__ du2dx,
						/*order 0*/ double * __restrict__ du2dy,
						/*order 0*/ double * __restrict__ du2dz,

						/*order 0*/ double * __restrict__ drhodx,
						/*order 0*/ double * __restrict__ drhody,
						/*order 0*/ double * __restrict__ drhodz,

						/*order 0*/ double * __restrict__ drhou0dx,
						/*order 0*/ double * __restrict__ drhou0dy,
						/*order 0*/ double * __restrict__ drhou0dz,

						/*order 0*/ double * __restrict__ drhou1dx,
						/*order 0*/ double * __restrict__ drhou1dy,
						/*order 0*/ double * __restrict__ drhou1dz,

						/*order 0*/ double * __restrict__ drhou2dx,
						/*order 0*/ double * __restrict__ drhou2dy,
						/*order 0*/ double * __restrict__ drhou2dz,

						/*order 0*/ double * __restrict__ dpdx,
						/*order 0*/ double * __restrict__ dpdy,
						/*order 0*/ double * __restrict__ dpdz,

						/*order 0*/ double * __restrict__ dpu0dx,
						/*order 0*/ double * __restrict__ dpu1dy,
						/*order 0*/ double * __restrict__ dpu2dz,

						/*order 0*/ double * __restrict__ drhou0u0dx,
						/*order 0*/ double * __restrict__ drhou0u1dy,
						/*order 0*/ double * __restrict__ drhou0u2dz,

						/*order 0*/ double * __restrict__ drhou1u0dx,
						/*order 0*/ double * __restrict__ drhou1u1dy,
						/*order 0*/ double * __restrict__ drhou1u2dz,

						/*order 0*/ double * __restrict__ drhou2u0dx,
						/*order 0*/ double * __restrict__ drhou2u1dy,
						/*order 0*/ double * __restrict__ drhou2u2dz,

						/*order 0*/ double * __restrict__ du0d2x,
						/*order 0*/ double * __restrict__ du0d2y,
						/*order 0*/ double * __restrict__ du0d2z,

						/*order 0*/ double * __restrict__ du1d2x,
						/*order 0*/ double * __restrict__ du1d2y,
						/*order 0*/ double * __restrict__ du1d2z,

						/*order 0*/ double * __restrict__ du2d2x,
						/*order 0*/ double * __restrict__ du2d2y,
						/*order 0*/ double * __restrict__ du2d2z,

						/*order 0*/ double * __restrict__ du0dxdy,
						/*order 0*/ double * __restrict__ du0dxdz,
 
						/*order 0*/ double * __restrict__ du1dxdy,
						/*order 0*/ double * __restrict__ du1dydz,
 
						/*order 0*/ double * __restrict__ du2dxdz,
						/*order 0*/ double * __restrict__ du2dydz,

						/*order 0*/ double * __restrict__ dTd2x,
						/*order 0*/ double * __restrict__ dTd2y,
						/*order 0*/ double * __restrict__ dTd2z,

						/*order 0*/ double * __restrict__ drhoEdx,
						/*order 0*/ double * __restrict__ drhoEdy,
						/*order 0*/ double * __restrict__ drhoEdz,

						/*order 0*/ double * __restrict__ drhoEu0dx,
						/*order 0*/ double * __restrict__ drhoEu1dy,
						/*order 0*/ double * __restrict__ drhoEu2dz,
						
						/*order 0*/ double * __restrict__ Res_rho,
						/*order 0*/ double * __restrict__ Res_rhou0,
						/*order 0*/ double * __restrict__ Res_rhou1,
						/*order 0*/ double * __restrict__ Res_rhou2,
						/*order 0*/ double * __restrict__ Res_rhoE) {
	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x ;
	if (idx<block_ncc) {

		double lRes_rho = 0;
		double lRes_rhou0 = 0;
		double lRes_rhou1 = 0;
		double lRes_rhou2 = 0;
		double lRes_rhoE = 0;

		double ldu0dx = du0dx[idx];
		double ldu1dy = du1dy[idx];
		double ldu2dz = du2dz[idx];

		double tmp0 = -0.5 * (ldu0dx + ldu1dy + ldu2dz);

		lRes_rho += tmp0 * rho[idx];
		lRes_rhou0 += tmp0 * rhou0[idx];
		lRes_rhou1 += tmp0 * rhou1[idx];
		lRes_rhou2 += tmp0 * rhou2[idx];
		lRes_rhoE += tmp0 * rhoE[idx];

		double frac0 = 1./RE;
		double frac1 = 2./3.;
		double frac2 = 4./3.;
		double frac3 = 1./3.;
		lRes_rhoE += frac0 * (-frac1 * ldu0dx - frac1 * ldu1dy + frac2 * ldu2dz) * ldu2dz;
		lRes_rhoE += frac0 * (-frac1 * ldu0dx + frac2 * ldu1dy - frac1 * ldu2dz) * ldu1dy;
		lRes_rhoE += frac0 * ( frac2 * ldu0dx - frac1 * ldu1dy - frac1 * ldu2dz) * ldu0dx;

		double lu0 = u0[idx];
		lRes_rho +=   -0.5 * lu0 * drhodx[idx];
		lRes_rhou1 += -0.5 * lu0 * drhou1dx[idx];
		lRes_rhou2 += -0.5 * lu0 * drhou2dx[idx];
		lRes_rhoE +=  -0.5 * drhoEdx[idx] * lu0;

		double ldrhou0dx = drhou0dx[idx];
		lRes_rhou0 += -0.5 * lu0 * ldrhou0dx;
		lRes_rho +=   -0.5 * ldrhou0dx;

		tmp0 = frac0 * (frac2 * du0d2x[idx] + du0d2y[idx] + du0d2z[idx]
						+ frac3 * du1dxdy[idx]
						+ frac3 * du2dxdz[idx]);

		lRes_rhou0 += tmp0;
		lRes_rhoE += lu0 * tmp0;
		


		double lu1 = u1[idx];
		lRes_rho +=   -0.5 * lu1 * drhody[idx];
		lRes_rhou0 += -0.5 * lu1 * drhou0dy[idx];
		lRes_rhou2 += -0.5 * lu1 * drhou2dy[idx];
		lRes_rhoE +=  -0.5 * drhoEdy[idx] * lu1;

		double ldrhou1dy = drhou1dy[idx];
		lRes_rhou1 += -0.5 * lu1 * ldrhou1dy;
		lRes_rho +=   -0.5 * ldrhou1dy;

		tmp0 = frac0 * (frac2 * du1d2y[idx] + du1d2x[idx] + du1d2z[idx]
						+ frac3 * du0dxdy[idx]
						+ frac3 * du2dydz[idx]);

		lRes_rhou1 += tmp0;
		lRes_rhoE += lu1 * tmp0;



		double lu2 = u2[idx];
		lRes_rho +=   -0.5 * lu2 * drhodz[idx];
		lRes_rhou0 += -0.5 * lu2 * drhou0dz[idx];
		lRes_rhou1 += -0.5 * lu2 * drhou1dz[idx];
		lRes_rhoE +=  -0.5 * drhoEdz[idx] * lu2;

		double ldrhou2dz = drhou2dz[idx];
		lRes_rhou2 += -0.5 * lu2 * ldrhou2dz;
		lRes_rho +=   -0.5 * ldrhou2dz;

		Res_rho[idx] = lRes_rho;

		tmp0 = frac0 * (frac2 * du2d2z[idx] + du2d2x[idx] + du2d2y[idx]
						+ frac3 * du0dxdz[idx]
						+ frac3 * du1dydz[idx]);

		lRes_rhou2 += tmp0;
		lRes_rhoE += lu2 * tmp0;


		lRes_rhou0 += -0.5 * drhou0u0dx[idx] - 0.5 * drhou0u1dy[idx] - 0.5 * drhou0u2dz[idx];
		lRes_rhou0 += -dpdx[idx];

		Res_rhou0[idx] = lRes_rhou0;


		lRes_rhou1 += -0.5 * drhou1u0dx[idx] - 0.5 * drhou1u1dy[idx] - 0.5 * drhou1u2dz[idx];
		lRes_rhou1 += -dpdy[idx];

		Res_rhou1[idx] = lRes_rhou1;


		lRes_rhou2 += -0.5 * drhou2u0dx[idx] - 0.5 * drhou2u1dy[idx] - 0.5 * drhou2u2dz[idx];
		lRes_rhou2 += -dpdz[idx];

		Res_rhou2[idx] = lRes_rhou2;


		lRes_rhoE += -dpu0dx[idx] - dpu1dy[idx] - dpu2dz[idx];
		lRes_rhoE += -0.5 * drhoEu0dx[idx] - 0.5 * drhoEu1dy[idx] - 0.5 * drhoEu2dz[idx];


		lRes_rhoE += 1./RE *	(du0dy[idx] + du1dx[idx]) * du0dy[idx]
								 + 1./RE *	(du0dy[idx] + du1dx[idx]) * du1dx[idx];

		lRes_rhoE += 1./RE *	(du0dz[idx] + du2dx[idx]) * du0dz[idx]
								 + 1./RE *	(du0dz[idx] + du2dx[idx]) * du2dx[idx];

		lRes_rhoE += 1./RE *	(du1dz[idx] + du2dy[idx]) * du1dz[idx]
								 + 1./RE *	(du1dz[idx] + du2dy[idx]) * du2dy[idx];
		
		lRes_rhoE += (dTd2x[idx] + dTd2y[idx] + dTd2z[idx]) / (MINF * MINF * PR * RE * (GAMA - 1));

		Res_rhoE[idx] = lRes_rhoE;

	}



}

__global__ void dns_Res(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ rho,
						/*order 0*/ double * __restrict__ u0,
						/*order 0*/ double * __restrict__ u1,
						/*order 0*/ double * __restrict__ u2,
						/*order 0*/ double * __restrict__ rhou0,
						/*order 0*/ double * __restrict__ rhou1,
						/*order 0*/ double * __restrict__ rhou2,
						/*order 0*/ double * __restrict__ rhoE,
						/*order 0*/ double * __restrict__ p,
						/*order 0*/ double * __restrict__ T,

						/*order 0*/ double * __restrict__ du0dx,
						/*order 0*/ double * __restrict__ du0dy,
						/*order 0*/ double * __restrict__ du0dz,

						/*order 0*/ double * __restrict__ du1dx,
						/*order 0*/ double * __restrict__ du1dy,
						/*order 0*/ double * __restrict__ du1dz,

						/*order 0*/ double * __restrict__ du2dx,
						/*order 0*/ double * __restrict__ du2dy,
						/*order 0*/ double * __restrict__ du2dz,

						/*order 0*/ double * __restrict__ drhodx,
						/*order 0*/ double * __restrict__ drhody,
						/*order 0*/ double * __restrict__ drhodz,

						/*order 0*/ double * __restrict__ drhou0dx,
						/*order 0*/ double * __restrict__ drhou0dy,
						/*order 0*/ double * __restrict__ drhou0dz,

						/*order 0*/ double * __restrict__ drhou1dx,
						/*order 0*/ double * __restrict__ drhou1dy,
						/*order 0*/ double * __restrict__ drhou1dz,

						/*order 0*/ double * __restrict__ drhou2dx,
						/*order 0*/ double * __restrict__ drhou2dy,
						/*order 0*/ double * __restrict__ drhou2dz,

						/*order 0*/ double * __restrict__ dpdx,
						/*order 0*/ double * __restrict__ dpdy,
						/*order 0*/ double * __restrict__ dpdz,

						/*order 0*/ double * __restrict__ dpu0dx,
						/*order 0*/ double * __restrict__ dpu1dy,
						/*order 0*/ double * __restrict__ dpu2dz,

						/*order 0*/ double * __restrict__ drhou0u0dx,
						/*order 0*/ double * __restrict__ drhou0u1dy,
						/*order 0*/ double * __restrict__ drhou0u2dz,

						/*order 0*/ double * __restrict__ drhou1u0dx,
						/*order 0*/ double * __restrict__ drhou1u1dy,
						/*order 0*/ double * __restrict__ drhou1u2dz,

						/*order 0*/ double * __restrict__ drhou2u0dx,
						/*order 0*/ double * __restrict__ drhou2u1dy,
						/*order 0*/ double * __restrict__ drhou2u2dz,

						/*order 0*/ double * __restrict__ du0d2x,
						/*order 0*/ double * __restrict__ du0d2y,
						/*order 0*/ double * __restrict__ du0d2z,

						/*order 0*/ double * __restrict__ du1d2x,
						/*order 0*/ double * __restrict__ du1d2y,
						/*order 0*/ double * __restrict__ du1d2z,

						/*order 0*/ double * __restrict__ du2d2x,
						/*order 0*/ double * __restrict__ du2d2y,
						/*order 0*/ double * __restrict__ du2d2z,

						/*order 0*/ double * __restrict__ du0dxdy,
						/*order 0*/ double * __restrict__ du0dxdz,
 
						/*order 0*/ double * __restrict__ du1dxdy,
						/*order 0*/ double * __restrict__ du1dydz,
 
						/*order 0*/ double * __restrict__ du2dxdz,
						/*order 0*/ double * __restrict__ du2dydz,

						/*order 0*/ double * __restrict__ dTd2x,
						/*order 0*/ double * __restrict__ dTd2y,
						/*order 0*/ double * __restrict__ dTd2z,

						/*order 0*/ double * __restrict__ drhoEdx,
						/*order 0*/ double * __restrict__ drhoEdy,
						/*order 0*/ double * __restrict__ drhoEdz,

						/*order 0*/ double * __restrict__ drhoEu0dx,
						/*order 0*/ double * __restrict__ drhoEu1dy,
						/*order 0*/ double * __restrict__ drhoEu2dz,
						
						/*order 0*/ double * __restrict__ Res_rho,
						/*order 0*/ double * __restrict__ Res_rhou0,
						/*order 0*/ double * __restrict__ Res_rhou1,
						/*order 0*/ double * __restrict__ Res_rhou2,
						/*order 0*/ double * __restrict__ Res_rhoE) {
	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		/*
		Res_rho[idx] = 0.5 * (- rho[idx] * (du0dx[idx] + du1dy[idx] + du2dz[idx]) 
						- drhodx[idx] * u0[idx] - drhody[idx] * u1[idx] - drhodz[idx] * u2[idx] 
						- drhou0dx[idx] - drhou1dy[idx] - drhou2dz[idx]);
		*/
		Res_rho[idx] = -0.5 * (du1dy[idx] + du2dz[idx] + du0dx[idx]) * rho[idx] 
									- 0.5 * u0[idx] * drhodx[idx] - 0.5 * u1[idx] * drhody[idx] - 0.5 * u2[idx] * drhodz[idx] 
									- 0.5 * drhou1dy[idx] - 0.5 * drhou2dz[idx] - 0.5 * drhou0dx[idx];
		

		// -0.5 * (du1/dy + du2/dz + du0dx) * rho - 0.5 * u0 * drhodx - 0.5 * u1 * drhody - 0.5 * u2 * drhodz - 0.5 * drhou1dy - 0.5 * drhou2dz - 0.5 * drhou0dx
		
		Res_rhou0[idx] = -0.5 * (du0dx[idx] + du1dy[idx] + du2dz[idx]) * rhou0[idx]
										 -0.5 * u0[idx] * drhou0dx[idx] - 0.5 * u1[idx] * drhou0dy[idx] - 0.5 * u2[idx] * drhou0dz[idx] 
										 -0.5 * drhou0u0dx[idx] - 0.5 * drhou0u1dy[idx] - 0.5 * drhou0u2dz[idx]
										 - dpdx[idx]
										+ 1. / RE * (4./3. * du0d2x[idx] + du0d2y[idx] + du0d2z[idx] 
										+ 1./3. * du1dxdy[idx] 
										+ 1./3. * du2dxdz[idx]);

		Res_rhou1[idx] = -0.5 * (du0dx[idx] + du1dy[idx] + du2dz[idx]) * rhou1[idx]
										 -0.5 * u0[idx] * drhou1dx[idx] - 0.5 * u1[idx] * drhou1dy[idx] - 0.5 * u2[idx] * drhou1dz[idx] 
										 -0.5 * drhou1u0dx[idx] - 0.5 * drhou1u1dy[idx] - 0.5 * drhou1u2dz[idx]
										 - dpdy[idx]
										+ 1. / RE * (4./3. * du1d2y[idx] + du1d2x[idx] + du1d2z[idx] 
										+ 1./3. * du0dxdy[idx] 
										+ 1./3. * du2dydz[idx]);

		Res_rhou2[idx] = -0.5 * (du0dx[idx] + du1dy[idx] + du2dz[idx]) * rhou2[idx]
										 -0.5 * u0[idx] * drhou2dx[idx] - 0.5 * u1[idx] * drhou2dy[idx] - 0.5 * u2[idx] * drhou2dz[idx] 
										 -0.5 * drhou2u0dx[idx] - 0.5 * drhou2u1dy[idx] - 0.5 * drhou2u2dz[idx]
										 - dpdz[idx]
										+ 1. / RE * (4./3. * du2d2z[idx] + du2d2x[idx] + du2d2y[idx] 
										+ 1./3. * du0dxdz[idx] 
										+ 1./3. * du1dydz[idx]);

		Res_rhoE[idx] = -0.5 * (du0dx[idx] + du1dy[idx] + du2dz[idx]) * rhoE[idx]
										-0.5 * drhoEdx[idx] * u0[idx] - 0.5 * drhoEdy[idx] * u1[idx] - 0.5 * drhoEdz[idx] * u2[idx]
										-dpu0dx[idx] - dpu1dy[idx] - dpu2dz[idx] 
										-0.5 * drhoEu0dx[idx] - 0.5 * drhoEu1dy[idx] - 0.5 * drhoEu2dz[idx]
										+ u0[idx] / RE * (4./3. * du0d2x[idx] + du0d2y[idx] + du0d2z[idx]
											+ 1./3. * du1dxdy[idx] 
											+ 1./3. * du2dxdz[idx])
										+ u1[idx] / RE * (1./3. * du0dxdy[idx] 
											+ du1d2x[idx] + 4./3. * du1d2y[idx] + du1d2z[idx]
											+ 1./3. * du2dydz[idx])
										+ u2[idx] / RE * (1./3. * du0dxdz[idx]
											+ 1./3. * du1dydz[idx]
											+ du2d2x[idx] + du2d2y[idx] + 4./3. * du2d2z[idx])
										+ 1./RE *	(du0dy[idx] + du1dx[idx]) * du0dy[idx]
										+ 1./RE *	(du0dy[idx] + du1dx[idx]) * du1dx[idx]

										+ 1./RE *	(du0dz[idx] + du2dx[idx]) * du0dz[idx]
										+ 1./RE *	(du0dz[idx] + du2dx[idx]) * du2dx[idx]

										+ 1./RE *	(du1dz[idx] + du2dy[idx]) * du1dz[idx]
										+ 1./RE *	(du1dz[idx] + du2dy[idx]) * du2dy[idx]

										+ 1./RE *	(-2./3. * du0dx[idx] - 2./3. * du1dy[idx] + 4./3. * du2dz[idx]) * du2dz[idx]
										+ 1./RE *	(-2./3. * du0dx[idx] + 4./3. * du1dy[idx] - 2./3. * du2dz[idx]) * du1dy[idx]
										+ 1./RE *	( 4./3. * du0dx[idx] - 2./3. * du1dy[idx] - 2./3. * du2dz[idx]) * du0dx[idx]
									
										+ (dTd2x[idx] + dTd2y[idx] + dTd2z[idx]) / (MINF * MINF * PR * RE * (GAMA - 1));

	}
	


}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Utility
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void dns_copy(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ g) {
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		g[idx] = f[idx];
	}

}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Constituent relations
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void dns_fdivg(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f,
						/*order 0*/ double * __restrict__ g,
						/*order 0*/ double * __restrict__ res) {
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		res[idx] = f[idx] / g[idx];
	}
}

__global__ void dns_p(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ rho,
						/*order 0*/ double * __restrict__ u0,
						/*order 0*/ double * __restrict__ u1,
						/*order 0*/ double * __restrict__ u2,
						/*order 0*/ double * __restrict__ rhoE,
						/*order 0*/ double * __restrict__ p) {
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		p[idx] = (GAMA - 1) * (rhoE[idx] - 
							0.5 * rho[idx]  * u0[idx] * u0[idx] -
							0.5 * rho[idx]  * u1[idx] * u1[idx] -
							0.5 * rho[idx]  * u2[idx] * u2[idx]);
	}

}

__global__ void dns_T(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ rho,
						/*order 0*/ double * __restrict__ p,
						/*order 0*/ double * __restrict__ T) {
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		T[idx] = MINF * MINF * GAMA * p[idx] / rho[idx];
	}

}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Time advancement
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void dns_RKsubStage(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f0,
						/*order 0*/ double * __restrict__ f1,
						/*order 0*/ double * __restrict__ f2,
						/*order 0*/ double * __restrict__ f3,
						/*order 0*/ double * __restrict__ f4,
						/*order 0*/ double * __restrict__ f0_old,
						/*order 0*/ double * __restrict__ f1_old,
						/*order 0*/ double * __restrict__ f2_old,
						/*order 0*/ double * __restrict__ f3_old,
						/*order 0*/ double * __restrict__ f4_old,
						/*order 0*/ double * __restrict__ Residual0,
						/*order 0*/ double * __restrict__ Residual1,
						/*order 0*/ double * __restrict__ Residual2,
						/*order 0*/ double * __restrict__ Residual3,
						/*order 0*/ double * __restrict__ Residual4,
						/*order 0*/ double rk) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		f0[idx] = DT * rk * Residual0[idx] + f0_old[idx];
		f1[idx] = DT * rk * Residual1[idx] + f1_old[idx];
		f2[idx] = DT * rk * Residual2[idx] + f2_old[idx];
		f3[idx] = DT * rk * Residual3[idx] + f3_old[idx];
		f4[idx] = DT * rk * Residual4[idx] + f4_old[idx];
	}
}

__global__ void dns_RKtmpAdvance(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f0_old_out,
						/*order 0*/ double * __restrict__ f1_old_out,
						/*order 0*/ double * __restrict__ f2_old_out,
						/*order 0*/ double * __restrict__ f3_old_out,
						/*order 0*/ double * __restrict__ f4_old_out,
						/*order 0*/ double * __restrict__ f0_old,
						/*order 0*/ double * __restrict__ f1_old,
						/*order 0*/ double * __restrict__ f2_old,
						/*order 0*/ double * __restrict__ f3_old,
						/*order 0*/ double * __restrict__ f4_old,
						/*order 0*/ double * __restrict__ Residual0,
						/*order 0*/ double * __restrict__ Residual1,
						/*order 0*/ double * __restrict__ Residual2,
						/*order 0*/ double * __restrict__ Residual3,
						/*order 0*/ double * __restrict__ Residual4,
						/*order 0*/ double rk) {

	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		f0_old_out[idx] = DT * rk * Residual0[idx] + f0_old[idx];
		f1_old_out[idx] = DT * rk * Residual1[idx] + f1_old[idx];
		f2_old_out[idx] = DT * rk * Residual2[idx] + f2_old[idx];
		f3_old_out[idx] = DT * rk * Residual3[idx] + f3_old[idx];
		f4_old_out[idx] = DT * rk * Residual4[idx] + f4_old[idx];
	}
}

__global__ void dns_init(const double * __restrict__ p_in, double * __restrict__ p_out) {
	// Calculate position in part
	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_header_size) {
 		p_out[idx]=p_in[idx];
	}


}


__global__ void dns_DebugAdvance(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f0_out,
						/*order 0*/ double * __restrict__ f0_in) {

	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<block_ncc) {
		f0_out[idx] = f0_in[idx] + 1.0;

	}
}

__global__ void dns_Debug(int32_t i_worker, int32_t order_in, int32_t order_out,
						/*order 0*/ double * __restrict__ f0_in, /*order 0*/ double * __restrict__ f0_out) {

	int32_t idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx<2) {
		printf("In: f0[%d] = %lf    ", idx, f0_in[idx]);
		if (idx == 0) {
			f0_out[idx] = f0_in[idx];
		}
		if (idx == 1) {
			printf("Out old: f0[%d] = %lf    ", idx, f0_out[idx]);
			f0_out[idx] = f0_in[idx] + 1;
			printf("Out new: f0[%d] = %lf    ", idx, f0_out[idx]);
		}

	}
}




void DS::caller_worker (double ** p_in, double ** p_out, int32_t i_part, int32_t i_super_cycle,
						int32_t order_in, int32_t order_out, int32_t iworker, int32_t nworker,
						hipStream_t * stream, int32_t threads_per_block, int32_t blockSize, int32_t myID) {

	//cout << "in:" << p_in[0] << " " << p_in[1] << " " << p_in[2] << " " << p_in[3] << " " << p_in[4] << " " << p_in[5] << endl;
	//cout << "out:" << p_out[0] << " " << p_out[1] << " " << p_out[2] << " " << p_out[3] << " " << p_out[4] << " " << p_out[5] << endl;

	// the order of arrays in p_in and p_out is:
	// center, left, right, left-left, right-right, left-left-left, right-right-right, and so on
	// entries can be NULL when invalid

	//cout << NX << ", " << NY << ", " << NZ << ", " << DX << ", " << DY << ", " << DZ <<  ", " <<  DT << ", " <<  GAMA <<  ", " <<  MINF <<  ", " <<  RE <<  ", " <<  PR << endl;


	int32_t global_worker_id = nworker * myID + iworker;
	int32_t n_global_worker = n_procs * n_worker;
	int32_t stage = (global_worker_id + n_global_worker * i_super_cycle) % 3;

	//cout << "Working on stage " << stage << endl;

	double rkold = RKOLD[stage];
	double rknew = RKNEW[stage];

	//cout << "rkold " << rkold << " rknew " << rknew << endl;

	
	// Sort out parts
	double* p_c = p_in[0];
	double* p_l = p_in[1];
	double* p_r = p_in[2];
	double* p_ll = p_in[3];
	double* p_rr = p_in[4];

	// Sort out parts
	double* p_c_out = p_out[0];

	// Symmetry BC u0 = -u0
	int sy_bc_ll = 1;
	int sy_bc_l  = 1;
	int sy_bc_r  = 1;
	int sy_bc_rr = 1;
	if (i_part == 0) {
		p_ll = p_rr;
		p_l = p_r;
		sy_bc_ll = -1;
		sy_bc_l  = -1;
	} else if(i_part == 1) {
		p_ll = p_c;
		sy_bc_ll = -1;
	} else if (i_part == my_n_part - 2) {
		p_rr = p_c;
		sy_bc_rr = -1;
	} else if (i_part == my_n_part - 1) {
		p_rr = p_ll;
		p_r = p_l;
		sy_bc_rr = -1;
		sy_bc_r  = -1;
	}

	// Offsets in pages for each field
	size_t offset_rho   = block_header_size + 0*block_ncc;
	size_t offset_rhou0 = block_header_size + 1*block_ncc;
	size_t offset_rhou1 = block_header_size + 2*block_ncc;
	size_t offset_rhou2 = block_header_size + 3*block_ncc;
	size_t offset_rhoE = block_header_size + 4*block_ncc;
	size_t offset_rho_old   = block_header_size + 5*block_ncc;
	size_t offset_rhou0_old = block_header_size + 6*block_ncc;
	size_t offset_rhou1_old = block_header_size + 7*block_ncc;
	size_t offset_rhou2_old = block_header_size + 8*block_ncc;
	size_t offset_rhoE_old = block_header_size + 9*block_ncc;
	size_t offset_tmp0 = block_header_size + 10*block_ncc;
	size_t offset_tmp1 = block_header_size + 11*block_ncc;
	size_t offset_tmp2 = block_header_size + 12*block_ncc;
	size_t offset_tmp3 = block_header_size + 13*block_ncc;
	size_t offset_tmp4 = block_header_size + 14*block_ncc;



	//cout << "Blocksize: " << blockSize << ", threads_per_block: " << threads_per_block << endl;

	threads_per_block = 128;
	int32_t gridSize = (blockSize + threads_per_block - 1) / threads_per_block;

	//cout << "Slice Size: " << blockSize << ", gridSize: " << gridSize << ", started Threads: " << gridSize * threads_per_block << endl;




	if (stage == 0) {
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], &p_c[offset_rho_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], &p_c[offset_rhou0_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], &p_c[offset_rhou1_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], &p_c[offset_rhou2_old]);
		dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhoE], &p_c[offset_rhoE_old]);
	}

	// Constituent relations
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], &p_c[offset_rho], (double*) d_u0_c);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou0], &p_l[offset_rho], (double*) d_u0_l);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_r[offset_rhou0], &p_r[offset_rho], (double*) d_u0_r);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_ll[offset_rhou0], &p_ll[offset_rho], (double*) d_u0_ll);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_rr[offset_rhou0], &p_rr[offset_rho], (double*) d_u0_rr);

	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], &p_c[offset_rho], (double*) d_u1_c);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou1], &p_l[offset_rho], (double*) d_u1_l);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_r[offset_rhou1], &p_r[offset_rho], (double*) d_u1_r);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_ll[offset_rhou1], &p_ll[offset_rho], (double*) d_u1_ll);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_rr[offset_rhou1], &p_rr[offset_rho], (double*) d_u1_rr);

	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], &p_c[offset_rho], (double*) d_u2_c);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou2], &p_l[offset_rho], (double*) d_u2_l);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_r[offset_rhou2], &p_r[offset_rho], (double*) d_u2_r);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_ll[offset_rhou2], &p_ll[offset_rho], (double*) d_u2_ll);
	dns_fdivg <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_rr[offset_rhou2], &p_rr[offset_rho], (double*) d_u2_rr);

	dns_p <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], (double*) d_u0_c, (double*) d_u1_c, (double*) d_u2_c, &p_c[offset_rhoE], (double*) d_p_c);
	dns_T <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], (double*) d_p_c, (double*) d_T_c);

	//30
	#ifdef DOPTI
	int32_t threads_per_block_opti = BLOCKSIZE_Z * BLOCKSIZE_Y;
	int32_t gridSize_opti = (blockSize + threads_per_block_opti - 1) / threads_per_block_opti;
	#endif


	dns_du0dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
						&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
						&p_c[offset_rhou0], &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
						sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
						(double*) d_drhou0dx, (double*) d_drhou0dy, (double*) d_drhou0dz,
						(double*) d_du0dx, (double*) d_du0dy, (double*) d_du0dz, 
						(double*) d_du0d2x, (double*) d_du0d2y, (double*) d_du0d2z,
						(double*) d_drhou0u0dx,
						(double*) d_du0dxdy, (double*) d_du0dxdz);

	dns_du1dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
						&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
						&p_c[offset_rhou1], &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
						(double*) d_drhou1dx, (double*) d_drhou1dy, (double*) d_drhou1dz,
						(double*) d_du1dx, (double*) d_du1dy, (double*) d_du1dz, 
						(double*) d_du1d2x, (double*) d_du1d2y, (double*) d_du1d2z,
						(double*) d_drhou1u1dy,
						(double*) d_du1dxdy, (double*) d_du1dydz);

	dns_du2dxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, my_n_part, BLOCKSIZE_Z, BLOCKSIZE_Y, WARPSIZE_Z, WARPSIZE_Y,
						&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
						&p_c[offset_rhou2], &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
						(double*) d_drhou2dx, (double*) d_drhou2dy, (double*) d_drhou2dz,
						(double*) d_du2dx, (double*) d_du2dy, (double*) d_du2dz, 
						(double*) d_du2d2x, (double*) d_du2d2y, (double*) d_du2d2z,
						(double*) d_drhou2u2dz,
						(double*) d_du2dxdz, (double*) d_du2dydz);

	dns_drhoETpdxyz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
						&p_c[offset_rho], &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho],
						&p_c[offset_rhou0], &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
						&p_c[offset_rhou1], &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
						&p_c[offset_rhou2], &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
						&p_c[offset_rhoE], &p_l[offset_rhoE], &p_r[offset_rhoE], &p_ll[offset_rhoE], &p_rr[offset_rhoE],
						sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr,
						(double*) d_drhodx, (double*) d_drhody, (double*) d_drhodz,
						(double*) d_drhoEdx, (double*) d_drhoEdy, (double*) d_drhoEdz,
						(double*) d_drhoEu0dx, (double*) d_drhoEu1dy, (double*) d_drhoEu2dz,
						(double*) d_dTd2x, (double*) d_dTd2y, (double*) d_dTd2z,
						(double*) d_dpdx, (double*) d_dpdy, (double*) d_dpdz,
						(double*) d_dpu0dx, (double*) d_dpu1dy, (double*) d_dpu2dz);

	// drhou0u0dx, drhou0u1dy, drhou0u2dz
	dns_dfgdy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], (double*) d_u1_c, (double*) d_drhou0u1dy);
	dns_dfgdz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], (double*) d_u2_c, (double*) d_drhou0u2dz);

	// drhou1u0dx, drhou1u1dy, drhou1u2dz
	dns_dfgdx <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
																												(double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_drhou1u0dx, 1, 1, 1, 1, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfgdz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], (double*) d_u2_c, (double*) d_drhou1u2dz);

	// drhou2u0dx, drhou2u1dy, drhou2u2dz
	dns_dfgdx <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
																												(double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_drhou2u0dx, 1, 1, 1, 1, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfgdy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], (double*) d_u1_c, (double*) d_drhou2u1dy);

	//14 *3		
	// du0dxdy, du0dxdz
	//dns_dfdy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du0dx, (double*) d_du0dxdy);
	//dns_dfdz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du0dx, (double*) d_du0dxdz);

	// du1dxdy, du1dydz
	//dns_dfdy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du1dx, (double*) d_du1dxdy);
	//dns_dfdz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du1dy, (double*) d_du1dydz);

	// du2dxdz, du2dydz
	//dns_dfdy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du2dx, (double*) d_du2dxdz);
	//dns_dfdz <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du2dy, (double*) d_du2dydz);

	// 4 * 3

	int32_t threads_per_block_dns_Res = 128;
	int32_t gridSize_dns_Res = (blockSize + threads_per_block_dns_Res - 1) / threads_per_block_dns_Res;

	dns_Res <<<gridSize_dns_Res,threads_per_block_dns_Res,0,*stream>>>(0, 0, 0,
						(double*) &p_c[offset_rho],
						(double*) d_u0_c, (double*) d_u1_c, (double*) d_u2_c,
						&p_c[offset_rhou0], &p_c[offset_rhou1], &p_c[offset_rhou2], &p_c[offset_rhoE],
						(double*) d_p_c, (double*) d_T_c,
						(double*) d_du0dx, (double*) d_du0dy, (double*) d_du0dz,
						(double*) d_du1dx, (double*) d_du1dy, (double*) d_du1dz,
						(double*) d_du2dx, (double*) d_du2dy, (double*) d_du2dz,
						(double*) d_drhodx, (double*) d_drhody, (double*) d_drhodz,
						(double*) d_drhou0dx, (double*) d_drhou0dy, (double*) d_drhou0dz,
						(double*) d_drhou1dx, (double*) d_drhou1dy, (double*) d_drhou1dz,
						(double*) d_drhou2dx, (double*) d_drhou2dy, (double*) d_drhou2dz,
						(double*) d_dpdx, (double*) d_dpdy, (double*) d_dpdz,
						(double*) d_dpu0dx, (double*) d_dpu1dy, (double*) d_dpu2dz,
						(double*) d_drhou0u0dx, (double*) d_drhou0u1dy, (double*) d_drhou0u2dz,
						(double*) d_drhou1u0dx, (double*) d_drhou1u1dy, (double*) d_drhou1u2dz,
						(double*) d_drhou2u0dx, (double*) d_drhou2u1dy, (double*) d_drhou2u2dz,
						(double*) d_du0d2x, (double*) d_du0d2y, (double*) d_du0d2z,
						(double*) d_du1d2x, (double*) d_du1d2y, (double*) d_du1d2z,
						(double*) d_du2d2x, (double*) d_du2d2y, (double*) d_du2d2z,
						(double*) d_du0dxdy, (double*) d_du0dxdz,
 						(double*) d_du1dxdy, (double*) d_du1dydz,
 						(double*) d_du2dxdz, (double*) d_du2dydz,
						(double*) d_dTd2x, (double*) d_dTd2y, (double*) d_dTd2z,
						(double*) d_drhoEdx, (double*) d_drhoEdy, (double*) d_drhoEdz,
						(double*) d_drhoEu0dx, (double*) d_drhoEu1dy, (double*) d_drhoEu2dz,
						(double*) d_Res_rho,
						(double*) d_Res_rhou0,
						(double*) d_Res_rhou1,
						(double*) d_Res_rhou2,
						(double*) d_Res_rhoE);

		dns_RKsubStage <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
					&p_c_out[offset_rho], 
					&p_c_out[offset_rhou0], &p_c_out[offset_rhou1], &p_c_out[offset_rhou2],
					&p_c_out[offset_rhoE],
					&p_c[offset_rho_old],
					&p_c[offset_rhou0_old], &p_c[offset_rhou1_old], &p_c[offset_rhou2_old],
					&p_c[offset_rhoE_old],
					(double*) d_Res_rho,
					(double*) d_Res_rhou0, (double*) d_Res_rhou1, (double*) d_Res_rhou2, 
					(double*) d_Res_rhoE,
					rknew);
	
	dns_RKtmpAdvance <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0,
					&p_c_out[offset_rho_old],
					&p_c_out[offset_rhou0_old], &p_c_out[offset_rhou1_old], &p_c_out[offset_rhou2_old],
					&p_c_out[offset_rhoE_old],
					&p_c[offset_rho_old],
					&p_c[offset_rhou0_old], &p_c[offset_rhou1_old], &p_c[offset_rhou2_old],
					&p_c[offset_rhoE_old],
					(double*) d_Res_rho,
					(double*) d_Res_rhou0, (double*) d_Res_rhou1, (double*) d_Res_rhou2, 
					(double*) d_Res_rhoE,
					rkold);

	//dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_Res_rho, &p_c[offset_tmp0]);
	//dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_Res_rhou0, &p_c[offset_tmp1]);
	//dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_Res_rhou1, &p_c[offset_tmp2]);
	//dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_Res_rhou2, &p_c[offset_tmp3]);
	//dns_copy <<<gridSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_Res_rhoE, &p_c[offset_tmp4]);

	// Copy Header
	dns_init <<<gridSize,threads_per_block,0,*stream>>>((const double*)p_c, (double*)p_c_out);
	//hipMemcpy((void*)p_c_out,(const void*)p_c,block_header_size * sizeof(double),hipMemcpyDeviceToDevice); cudaCheckError(__LINE__,__FILE__);
	// 4

}


__global__ void prepare_visual_rectilinear(double * __restrict__ p_in, double * __restrict__ p_out) {

	int32_t global_id = blockIdx.x*blockDim.x+threadIdx.x;
	// int32_t n_threads = blockDim.x*gridDim.x;

	// int32_t * p_in_i32 = (int32_t *)p_in;
	int64_t * p_in_i64 = (int64_t*)p_in;
	double * p_in_d = (double*)p_in;


	int64_t i_part=p_in_i64[0];

	// if (global_id==0) {
	// 	printf("part:%i\n",i_part);
	// }

	float * p_out_float=(float*)p_out;


	// if (global_id==0) {
	// 	p_out_i32[0]=n_mol;
	// 	p_out_i32[1]=i_part;
	// }
	if (global_id<block_ncc) {
		int32_t i_cell=global_id;
		int32_t i_x=i_part;
		int32_t i_y=i_cell/my_n_part;
		int32_t i_z=i_cell-i_y*my_n_part;

		//printf("i_part_%i_%i_%i_%i_\n",i_x,i_y,i_z,i_part);

		for (int32_t i_field=0;i_field<block_n_fields;i_field++) {
			double dtmp=p_in_d[block_header_size+i_field*block_ncc+i_cell];

			/*
			if (i_field == 0) {
			if (i_cell < 2) {
				printf(" \n i_cell = %d, dtmp: %lf", i_cell, dtmp);
			}
			if (i_cell == 0) {
				printf("\nI write my part index to %d", i_field*my_n_part*block_ncc+i_z*block_ncc+i_y*my_n_part+i_x);
				printf("\nCalculated from: i_field %d, my_n_part %d, block_ncc %d, i_z %d, i_y %d, i_x %d\n", i_field, my_n_part, block_ncc, i_z, i_y, i_x);
			}
			}
			*/
			

			
			//double dtmp = 5.0;

			p_out_float[i_field*my_n_part*block_ncc+i_z*block_ncc+i_y*my_n_part+i_x]=dtmp;
		}
	}

}

void DS::write_vtr (float * p_data, int32_t i_part, int32_t i_cycle) {
	string FileName;
	FileName.append("visual/visual_");
	FileName+=to_string(my_n_part);
	FileName.append("_");
	FileName+=to_string(i_cycle);
	// FileName.append("/visual_");
	// FileName+=to_string(i_part);
	FileName.append(".vtr");

	/*
	printf("\np_data: ");
	
	for (int i = 0; i < 27; ++i) {
		if (i % 3 == 0) printf("\n");
		if (i % 9 == 0) printf("\n");
		printf("%f, ", p_data[i]);
		
	}
	printf("\n");
	*/
	
	

	ofstream ofs;
	ofs.open(FileName, ios::out | ios::binary);
	if (ofs) {
		int64_t append_offset=0;
		ofs << "<VTKFile type=\"RectilinearGrid\" version=\"1.0\" byte_order=\"LittleEndian\" header_type=\"UInt64\">" << endl;
		ofs << "<RectilinearGrid WholeExtent=\"" << "0 " << my_n_part-1 << " 0 " << my_n_part-1 << " 0 " << my_n_part-1 << "\">" << endl;
		ofs << "<Piece Extent=\"" << "0 " << my_n_part-1 << " 0 " << my_n_part-1 << " 0 " << my_n_part-1 << "\">" << endl;

		ofs << "<PointData Scalars=\"\" Name=\"a\">" << endl;
		ofs << "<DataArray type=\"Float32\" Name=\"rho\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhou0\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhou1\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhou2\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhoE\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rho_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhou0_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhou1_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);
		
		ofs << "<DataArray type=\"Float32\" Name=\"rhou2_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"rhoE_old\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);

		// ============================ TMP Output ============================
		//ofs << "<DataArray type=\"Float32\" Name=\"tmp0\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float32\" Name=\"tmp1\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float32\" Name=\"tmp2\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float32\" Name=\"tmp3\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);
//
		//ofs << "<DataArray type=\"Float32\" Name=\"tmp4\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		//ofs << append_offset;
		//ofs << "\">";
		//ofs << "</DataArray>" << endl;
		//append_offset+=(my_n_part*block_ncc)*sizeof(float)+sizeof(int64_t);
//
		// ============================ TMP Output ============================

		ofs << "</PointData>" << endl;

		ofs << "<Coordinates>" << endl;
		ofs << "<DataArray type=\"Float32\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(float)+sizeof(int64_t);

		ofs << "<DataArray type=\"Float32\" Name=\"Points\" NumberOfComponents=\"1\" format=\"appended\" offset=\"";
		ofs << append_offset;
		ofs << "\">";
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		ofs << "</DataArray>" << endl;
		append_offset+=(my_n_part+1)*sizeof(float)+sizeof(int64_t);

		ofs << "</Coordinates>" << endl;

		// ofs << "\" NumberOfCells=\"0\">" << endl;
		// ofs << "<PointData Scalars=\"species\">" << endl;
		// ofs << "<DataArray type=\"Float32\" Name=\"species\" format=\"appended\" offset=\"0\" RangeMin=\"0\" RangeMax=\"6\">" << endl;
		// ofs << "</DataArray>" << endl;
		// ofs << "</PointData>" << endl;
		// ofs << "<Points>" << endl;
		// ofs << "<DataArray type=\"Float32\" Name=\"Points\" NumberOfComponents=\"3\" format=\"appended\" offset=\"";
		// ofs << n_mol*sizeof(float)+8;
		// ofs << "\" RangeMin=\"0\" RangeMax=\"1.0\">" << endl;
		// ofs << "</DataArray>" << endl;
		// ofs << "</Points>" << endl;
		// ofs << "<Cells>" << endl;
		// ofs << "<DataArray type=\"Int32\" Name=\"connectivity\"></DataArray>" << endl;
		// ofs << "<DataArray type=\"Int32\" Name=\"offsets\"></DataArray>" << endl;
		// ofs << "<DataArray type=\"UInt8\" Name=\"types\"></DataArray>" << endl;
		// ofs << "</Cells>" << endl;
		ofs << "</Piece>" << endl;
		ofs << "</RectilinearGrid>" << endl;
		ofs << "<AppendedData encoding=\"raw\">" << endl;
		ofs << "_";	// mark start of appended data
		ofs.close();
	}

	// write appended data
	int64_t size_append=0;

	// cell data
	for (int32_t i_field=0;i_field<block_n_fields;i_field++) {
		size_append=(my_n_part*block_ncc)*sizeof(float);
		MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
		MemToFile((int64_t*)&p_data[i_field*my_n_part*block_ncc],size_append,(char*)FileName.c_str(),0);
	}

	// coordinates - same for x,y,z
	float * x_coordinates=new float [my_n_part+1];
	for (int i=0;i<my_n_part+1;i++) {
		x_coordinates[i]=i;
	}
	size_append=(my_n_part+1)*sizeof(float);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);

	size_append=(my_n_part+1)*sizeof(float);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);

	size_append=(my_n_part+1)*sizeof(float);
	MemToFile(&size_append,sizeof(int64_t),(char*)FileName.c_str(),0);
	MemToFile((int64_t*)x_coordinates,size_append,(char*)FileName.c_str(),0);
	delete [] x_coordinates;

	// write closing tags
	ofs.open(FileName, ios::out | ios::binary | ios_base::app);
	if (ofs) {
		ofs << "</AppendedData>" << endl;
		ofs << "</VTKFile>" << endl;
		ofs.close();
	}
}

void DS::caller_output_vtk_rectilinear (double * p_in, double * p_out, hipStream_t * stream, int32_t threads_per_block, int32_t blockSize, int32_t myID, int32_t i_cycle, int32_t i_part) {

	int32_t n_blocks=block_ncc/threads_per_block;
	n_blocks++;

	prepare_visual_rectilinear <<<n_blocks,threads_per_block,0,*stream>>> (p_in,p_out);
	// int32_t * p_my_vis_i32=(int32_t*)p_my_vis;
	// float * p_my_vis_float=(float*)p_my_vis;
	if (i_part==(my_n_part-1)) {
		// last part
		float * p_my_vis_float=new float[block_n_fields*my_n_part*block_ncc];

		hipDeviceSynchronize();        cudaCheckError(__LINE__,__FILE__);

		size_t copy_size=1;
		copy_size*=block_n_fields;
		copy_size*=my_n_part;
		copy_size*=block_ncc;
		copy_size*=sizeof(float);
		// cout << copy_size << endl;
		hipMemcpy((void*)p_my_vis_float,(const void*)p_out,copy_size,hipMemcpyDeviceToHost);

		// for (int i=0;i<block_n_fields*my_n_part*block_ncc;i++) cout << p_my_vis_float[i] << endl;

		// string new_dir;
		// new_dir.append("visual/visual_");
		// new_dir+=to_string(i_cycle);

		// boost::filesystem::create_directory(new_dir.c_str());
		write_vtr(p_my_vis_float,0,i_cycle);
		delete [] p_my_vis_float;
	}
}


/*
	// du0dx, du0dy, du0dz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_du0dx, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u0_c, (double*) d_du0dy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u0_c, (double*) d_du0dz);

	// du1dx, du1dy, du1dz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u1_l, (double*) d_u1_r, (double*) d_u1_ll, (double*) d_u1_rr, (double*) d_du1dx);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u1_c, (double*) d_du1dy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u1_c, (double*) d_du1dz);

	// du2dx, du2dy, du2dz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u2_l, (double*) d_u2_r, (double*) d_u2_ll, (double*) d_u2_rr, (double*) d_du2dx);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u2_c, (double*) d_du2dy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u2_c, (double*) d_du2dz);

	// drhodx, drhody, drhodz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rho], &p_r[offset_rho], &p_ll[offset_rho], &p_rr[offset_rho], (double*) d_drhodx);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], (double*) d_drhody);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rho], (double*) d_drhodz);

	// drhou0dx, drhou0dy, drhou0dz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0], (double*) d_drhou0dx, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], (double*) d_drhou0dy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], (double*) d_drhou0dz);

	// drhou1dx, drhou1dy, drhou1dz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1], (double*) d_drhou1dx);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], (double*) d_drhou1dy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], (double*) d_drhou1dz);

	// drhou2dx, drhou2dy, drhou2dz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2], (double*) d_drhou2dx);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], (double*) d_drhou2dy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], (double*) d_drhou2dz);

	// dpdx, dpdy, dpdz
	dns_dfdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_p_l, (double*) d_p_r, (double*) d_p_ll, (double*) d_p_rr, (double*) d_dpdx);
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_p_c, (double*) d_dpdy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_p_c, (double*) d_dpdz);

	// drhou0u0dx, drhou0u1dy, drhou0u2dz
	dns_dfgdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou0], &p_r[offset_rhou0], &p_ll[offset_rhou0], &p_rr[offset_rhou0],
																												(double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_drhou0u0dx, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfgdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], (double*) d_u1_c, (double*) d_drhou0u1dy);
	dns_dfgdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou0], (double*) d_u2_c, (double*) d_drhou0u2dz);

	// drhou1u0dx, drhou1u1dy, drhou1u2dz
	dns_dfgdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou1], &p_r[offset_rhou1], &p_ll[offset_rhou1], &p_rr[offset_rhou1],
																												(double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_drhou1u0dx, 1, 1, 1, 1, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfgdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], (double*) d_u1_c, (double*) d_drhou1u1dy);
	dns_dfgdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou1], (double*) d_u2_c, (double*) d_drhou1u2dz);

	// drhou2u0dx, drhou2u1dy, drhou2u2dz
	dns_dfgdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_l[offset_rhou2], &p_r[offset_rhou2], &p_ll[offset_rhou2], &p_rr[offset_rhou2],
																												(double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_drhou2u0dx, 1, 1, 1, 1, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfgdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], (double*) d_u1_c, (double*) d_drhou2u1dy);
	dns_dfgdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, &p_c[offset_rhou2], (double*) d_u2_c, (double*) d_drhou2u2dz);

	// du0d2x, du0d2y, du0d2z
	dns_dfd2x <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u0_c, (double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_du0d2x, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfd2y <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u0_c, (double*) d_du0d2y);
	dns_dfd2z <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_u0_c, (double*) d_du0d2z);
						
	// du0dxdy, du0dxdz
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du0dx, (double*) d_du0dxdy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du0dx, (double*) d_du0dxdz);

	// du1dxdy, du1dydz
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du1dx, (double*) d_du1dxdy);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du1dy, (double*) d_du1dydz);

	// du2dxdz, du2dydz
	dns_dfdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du2dx, (double*) d_du2dxdz);
	dns_dfdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_du2dy, (double*) d_du2dydz);

	// dpu0dx, dpu1dy, dpu2dz
	dns_dfgdx <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_p_l, (double*) d_p_r, (double*) d_p_ll, (double*) d_p_rr,
																												(double*) d_u0_l, (double*) d_u0_r, (double*) d_u0_ll, (double*) d_u0_rr, (double*) d_dpu0dx, 1, 1, 1, 1, sy_bc_ll, sy_bc_l, sy_bc_r, sy_bc_rr);
	dns_dfgdy <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_p_c, (double*) d_u1_c, (double*) d_dpu1dy);
	dns_dfgdz <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_p_c, (double*) d_u2_c, (double*) d_dpu2dz);

	// dTd2x, dTd2y, dTd2z
	dns_dfd2x <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_T_c, (double*) d_T_l, (double*) d_T_r, (double*) d_T_ll, (double*) d_T_rr, (double*) d_dTd2x);
	dns_dfd2y <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_T_c, (double*) d_dTd2y);
	dns_dfd2z <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0, (double*) d_T_c, (double*) d_dTd2z);
	
	dns_Res <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0,
						(double*) &p_c[offset_rho],
						(double*) d_u0_c, (double*) d_u1_c, (double*) d_u2_c,
						&p_c[offset_rhou0], &p_c[offset_rhou1], &p_c[offset_rhou2], &p_c[offset_rhoE],
						(double*) d_p_c, (double*) d_T_c,
						(double*) d_du0dx, (double*) d_du0dy, (double*) d_du0dz,
						(double*) d_du1dx, (double*) d_du1dy, (double*) d_du1dz,
						(double*) d_du2dx, (double*) d_du2dy, (double*) d_du2dz,
						(double*) d_drhodx, (double*) d_drhody, (double*) d_drhodz,
						(double*) d_drhou0dx, (double*) d_drhou0dy, (double*) d_drhou0dz,
						(double*) d_drhou1dx, (double*) d_drhou1dy, (double*) d_drhou1dz,
						(double*) d_drhou2dx, (double*) d_drhou2dy, (double*) d_drhou2dz,
						(double*) d_dpdx, (double*) d_dpdy, (double*) d_dpdz,
						(double*) d_dpu0dx, (double*) d_dpu1dy, (double*) d_dpu2dz,
						(double*) d_drhou0u0dx, (double*) d_drhou0u1dy, (double*) d_drhou0u2dz,
						(double*) d_drhou1u0dx, (double*) d_drhou1u1dy, (double*) d_drhou1u2dz,
						(double*) d_drhou2u0dx, (double*) d_drhou2u1dy, (double*) d_drhou2u2dz,
						(double*) d_du0d2x, (double*) d_du0d2y, (double*) d_du0d2z,
						(double*) d_du1d2x, (double*) d_du1d2y, (double*) d_du1d2z,
						(double*) d_du2d2x, (double*) d_du2d2y, (double*) d_du2d2z,
						(double*) d_du0dxdy, (double*) d_du0dxdz,
 						(double*) d_du1dxdy, (double*) d_du1dydz,
 						(double*) d_du2dxdz, (double*) d_du2dydz,
						(double*) d_dTd2x, (double*) d_dTd2y, (double*) d_dTd2z,
						(double*) d_drhoEdx, (double*) d_drhoEdy, (double*) d_drhoEdz,
						(double*) d_drhoEu0dx, (double*) d_drhoEu1dy, (double*) d_drhoEu2dz,
						(double*) d_Res_rho,
						(double*) d_Res_rhou0,
						(double*) d_Res_rhou1,
						(double*) d_Res_rhou2,
						(double*) d_Res_rhoE);

	dns_RKsubStage <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0,
					&p_c_out[offset_rho], 
					&p_c_out[offset_rhou0], &p_c_out[offset_rhou1], &p_c_out[offset_rhou2],
					&p_c_out[offset_rhoE],
					&p_c[offset_rho_old],
					&p_c[offset_rhou0_old], &p_c[offset_rhou1_old], &p_c[offset_rhou2_old],
					&p_c[offset_rhoE_old],
					(double*) d_Res_rho,
					(double*) d_Res_rhou0, (double*) d_Res_rhou1, (double*) d_Res_rhou2, 
					(double*) d_Res_rhoE,
					rknew);
	
	dns_RKtmpAdvance <<<blockSize,threads_per_block,0,*stream>>>(0, 0, 0,
					&p_c_out[offset_rho_old],
					&p_c_out[offset_rhou0_old], &p_c_out[offset_rhou1_old], &p_c_out[offset_rhou2_old],
					&p_c_out[offset_rhoE_old],
					&p_c[offset_rho_old],
					&p_c[offset_rhou0_old], &p_c[offset_rhou1_old], &p_c[offset_rhou2_old],
					&p_c[offset_rhoE_old],
					(double*) d_Res_rho,
					(double*) d_Res_rhou0, (double*) d_Res_rhou1, (double*) d_Res_rhou2, 
					(double*) d_Res_rhoE,
					rkold);


*/
